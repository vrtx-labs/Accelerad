#include "hip/hip_runtime.h"
/*
 *  klems_full.cu - program for Klems bin identification on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

rtDeclareVariable(float3, normal, , );	/* Normal direction */
rtDeclareVariable(float3, up, , );		/* Up direction */
rtDeclareVariable(int, RHS, , ) = 1;	/* Coordinate system handedness: set to -1 for left-handed system */

// Calculate the Klems bin based on klems_full.cal.
RT_CALLABLE_PROGRAM int kbin(const float3 direction)
{
	const float DdotN = dot(direction, normal);
	const float DdotU = dot(direction, up);
	const float NdotU = dot(normal, up);

	if (DdotN > 0) return -1; // Wrong-side ray, probably bad
	if (DdotN <= -1) return 0;
	float pol = acosf(-DdotN) * 180 * M_1_PIf;
	float azi = atan2f(-DdotU + DdotN * NdotU, -RHS * dot(direction, cross(up, normal))) * 180 * M_1_PIf;
	if (azi < 0.0f) azi += 360.0f;

	const int kpola[] = { 5, 15, 25, 35, 45, 55, 65, 75, 90 };
	const int knaz[] = { 1, 8, 16, 20, 24, 24, 24, 16, 12 };	// Number of patches per row

	int row = 0;
	while (pol > kpola[row]) // This is kfindrow
		row++;

	float inc = 360.0f / knaz[row];
	int kaccum = ((360.0f - 0.5f * inc) > azi) ? (int)floor((azi + 0.5f * inc) / inc) : 0; // This is kazn

	for (int r = 0; r < row; r++)
		kaccum += knaz[r];

	return kaccum;
}

///* Calculate the Klems bin based on klems_full.cal for north bins. */
//RT_CALLABLE_PROGRAM int kbinN(const float3 direction)
//{
//	return kbin(direction, make_float3(0, -1, 0), make_float3(0, 0, 1));
//}
//
///* Calculate the Klems bin based on klems_full.cal for east bins. */
//RT_CALLABLE_PROGRAM int kbinE(const float3 direction)
//{
//	return kbin(direction, make_float3(-1, 0, 0), make_float3(0, 0, 1));
//}
//
///* Calculate the Klems bin based on klems_full.cal for south bins. */
//RT_CALLABLE_PROGRAM int kbinS(const float3 direction)
//{
//	return kbin(direction, make_float3(0, 1, 0), make_float3(0, 0, 1));
//}
//
///* Calculate the Klems bin based on klems_full.cal for west bins. */
//RT_CALLABLE_PROGRAM int kbinW(const float3 direction)
//{
//	return kbin(direction, make_float3(1, 0, 0), make_float3(0, 0, 1));
//}
//
///* Calculate the Klems bin based on klems_full.cal for skylight bins. */
//RT_CALLABLE_PROGRAM int kbinD(const float3 direction)
//{
//	return kbin(direction, make_float3(0, 0, -1), make_float3(0, 1, 0));
//}
