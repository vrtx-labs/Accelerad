#include "hip/hip_runtime.h"
/*
 *  material_normal.cu - hit programs for normal materials on GPUs.
 */

#include "accelerad_copyright.h"

#include "otypes.h"	/* For definition of MAT_METAL */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_ray.h"
#ifdef CONTRIB_DOUBLE
#include "optix_double.h"
#endif

using namespace optix;

#define  AMBIENT
#define  TRANSMISSION

#ifndef  MAXITER
#define  MAXITER	10		/* maximum # specular ray attempts */
#endif
#define  MAXSPART	64		/* maximum partitions per source */
//#define frandom()	(rnd( prd.seed )/float(RAND_MAX))
//#define frandom()	(rnd( prd.seed ))

				/* specularity flags */
#define  SP_REFL	01		/* has reflected specular component */
#define  SP_TRAN	02		/* has transmitted specular */
#define  SP_PURE	04		/* purely specular (zero roughness) */
#define  SP_FLAT	010		/* flat reflecting surface */
#define  SP_RBLT	020		/* reflection below sample threshold */
#define  SP_TBLT	040		/* transmission below threshold */

typedef struct {
	unsigned int specfl;		/* specularity flags, defined above */
	float3 mcolor;		/* color of this material */
	float3 scolor;		/* color of specular component */
	//float3 vrefl;		/* vector in direction of reflected ray */
	float3 prdir;		/* vector in transmitted direction */
	float3 normal;
	float3 hit;
	float  alpha2;		/* roughness squared */
	float  rdiff, rspec;	/* reflected specular, diffuse */
	float  trans;		/* transmissivity */
	float  tdiff, tspec;	/* transmitted specular, diffuse */
	float3 pnorm;		/* perturbed surface normal */
	float  pdot;		/* perturbed dot product */
}  NORMDAT;		/* normal material data */

/* Context variables */
rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(rtObject,     top_ambient, , );

rtDeclareVariable(float,        specthresh, , );	/* This is the minimum fraction of reflection or transmission, under which no specular sampling is performed */
rtDeclareVariable(float,        specjitter, , );	/* specular sampling (ss) */

#ifdef LIGHTS
rtDeclareVariable(float,        dstrsrc, , ); /* direct jitter (dj) */
rtDeclareVariable(float,        srcsizerat, , );	/* direct sampling ratio (ds) */
//rtDeclareVariable(float,        shadthresh, , );	/* direct threshold (dt) */
//rtDeclareVariable(float,        shadcert, , );	/* direct certainty (dc) */
//rtDeclareVariable(int,          directrelay, , );	/* direct relays for secondary sources (dr) */
//rtDeclareVariable(int,          vspretest, , );	/* direct presampling density for secondary sources (dp) */
#endif /* LIGHTS */

#ifdef AMBIENT
rtDeclareVariable(float3,       ambval, , );	/* This is the final value used in place of an indirect light calculation */
rtDeclareVariable(int,          ambvwt, , );	/* As new indirect irradiances are computed, they will modify the default ambient value in a moving average, with the specified weight assigned to the initial value given on the command and all other weights set to 1 */
rtDeclareVariable(int,          ambounce, , );	/* Ambient bounces (ab) */
//rtDeclareVariable(int,          ambres, , );	/* Ambient resolution (ar) */
rtDeclareVariable(float,        ambacc, , );	/* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(int,          ambdiv, , );	/* Ambient divisions (ad) */
rtDeclareVariable(int,          ambdiv_final, , ); /* Number of ambient divisions for final-pass fill (ag) */
//rtDeclareVariable(int,          ambssamp, , );	/* Ambient super-samples (as) */
rtDeclareVariable(float,        avsum, , );		/* computed ambient value sum (log) */
rtDeclareVariable(unsigned int, navsum, , );	/* number of values in avsum */
#endif /* AMBIENT */

rtDeclareVariable(float,        exposure, , ) = 0.0f; /* Current exposure (-pe), zero unless called from rvu */

rtBuffer<DistantLight> lights;

/* Geometry instance variables */
#ifdef LIGHTS
rtBuffer<float3> vertex_buffer;
rtBuffer<uint3>  lindex_buffer;    // position indices
#endif


RT_METHOD float3 dirnorm(Ray *shadow_ray, PerRayData_shadow *shadow_prd, const NORMDAT *nd, const float& omega, const float3& ray_dir, PerRayData_radiance &prd);
RT_METHOD float3 gaussamp(const NORMDAT *nd, const float3& ray_dir, PerRayData_radiance &prd);
#ifdef AMBIENT
RT_METHOD float3 multambient(float3 aval, const float3& normal, const float3& pnormal, const float3& hit, const unsigned int& ambincl, PerRayData_radiance &prd);
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, PerRayData_radiance &prd, DaysimCoef dc);
#else
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, PerRayData_radiance &prd);
#endif
//RT_METHOD int ambsample( AMBHEMI *hp, const int& i, const int& j, const float3 normal, const float3 hit );
#endif /* AMBIENT */
#ifdef LIGHTS
RT_METHOD unsigned int flatpart( const float3& v, const float3& r0, const float3& r1, const float3& r2, const float& weight );
RT_METHOD float solid_angle( const float3& r0, const float3& r1, const float3& r2 );
RT_METHOD float3 barycentric( float2& lambda, const float3& r0, const float3& r1, const float3& r2, const int flip );
#endif /* LIGHTS */
//RT_METHOD float2 multisamp2(float r);
//RT_METHOD int ilhash(int3 d);


RT_CALLABLE_PROGRAM PerRayData_shadow closest_hit_normal_shadow(IntersectData const&data, PerRayData_shadow prd_shadow)
{
	NORMDAT nd;

	/* check for back side */
	nd.pnorm = faceforward(data.world_shading_normal, -data.ray_direction, data.world_geometric_normal);
	nd.normal = faceforward(data.world_geometric_normal, -data.ray_direction, data.world_geometric_normal);

	nd.hit = data.hit;
	nd.mcolor = data.mat.color;
	nd.rspec = data.mat.params.n.spec;
	nd.alpha2 = data.mat.params.n.rough * data.mat.params.n.rough;
	nd.specfl = 0u; /* specularity flags */

#ifdef TRANSMISSION
	if (data.mat.params.n.trans > 0.0f) { // type == MAT_TRANS
		/* get roughness */
		if (nd.alpha2 <= FTINY) {
			nd.specfl |= SP_PURE; // label this as a purely specular reflection
		}

		/* perturb normal */
		float3 pert = nd.normal - nd.pnorm;
		int hastexture = dot(pert, pert) > FTINY * FTINY;
		nd.pdot = -dot(data.ray_direction, nd.pnorm);
		if (nd.pdot < 0.0f) {		/* fix orientation from raynormal in raytrace.c */
			nd.pnorm += 2.0f * nd.pdot * data.ray_direction;
			nd.pdot = -nd.pdot;
		}
		if (nd.pdot < 0.001f)
			nd.pdot = 0.001f;			/* non-zero for dirnorm() */

		// if it's a face or a ring label as flat (currently we only support triangles, so everything is flat)
		nd.specfl |= SP_FLAT;

		/* modify material color */
		//nd.mcolor *= rtTex3D(rtTextureId id, texcoord.x, texcoord.y, texcoord.z).xyz;

		/* compute Fresnel approx. */
		float fest = 0.0f;
		if (nd.specfl & SP_PURE && nd.rspec >= FRESTHRESH) {
			fest = FRESNE(nd.pdot);
			nd.rspec += fest * (1.0f - nd.rspec);
		}

		/* compute transmission */
		nd.prdir = data.ray_direction;
		nd.trans = data.mat.params.n.trans * (1.0f - nd.rspec);
		nd.tspec = nd.trans * data.mat.params.n.tspec;
		if (nd.tspec > FTINY) {
			nd.specfl |= SP_TRAN;

			/* check threshold */
			if (!(nd.specfl & SP_PURE) && specthresh >= nd.tspec - FTINY)
				nd.specfl |= SP_TBLT;
			if (hastexture) { //TODO only if ambient depth == 0
				if (dot(nd.prdir - pert, nd.normal) < -FTINY)
					nd.prdir = normalize(nd.prdir - pert);	/* OK */
			}
		}
	}

	/* transmitted ray */
	if ((nd.specfl&(SP_TRAN | SP_PURE | SP_TBLT)) == (SP_TRAN | SP_PURE)) {
#ifdef CONTRIB
		prd_shadow.rcoef *= nd.mcolor * nd.tspec;
#endif
		Ray trans_ray = make_Ray(nd.hit, nd.prdir, SHADOW_RAY, ray_start(nd.hit, nd.prdir, nd.normal, RAY_START), RAY_END);
		rtTrace(top_object, trans_ray, prd_shadow);
		prd_shadow.result *= nd.mcolor * nd.tspec;
#ifdef DAYSIM_COMPATIBLE
		daysimScale(prd_shadow.dc, nd.mcolor.x * nd.tspec);
#endif
	}
#endif /* TRANSMISSION */
	return prd_shadow;
}


RT_CALLABLE_PROGRAM PerRayData_radiance closest_hit_normal_radiance(IntersectData const&data, PerRayData_radiance prd)
{
	NORMDAT nd;

	/* check for back side */
	nd.pnorm = faceforward(data.world_shading_normal, -data.ray_direction, data.world_geometric_normal);
	nd.normal = faceforward(data.world_geometric_normal, -data.ray_direction, data.world_geometric_normal);
	nd.hit = data.hit;

	PerRayData_radiance new_prd;
	float3 result = prd.mirror = make_float3(0.0f);
	nd.mcolor = data.mat.color;
	nd.scolor = make_float3(0.0f);
	nd.rspec = data.mat.params.n.spec;
	nd.alpha2 = data.mat.params.n.rough * data.mat.params.n.rough;
	nd.specfl = 0u; /* specularity flags */

	/* get roughness */
	if (nd.alpha2 <= FTINY) {
		nd.specfl |= SP_PURE; // label this as a purely specular reflection
	}

	/* perturb normal */
	float3 pert = nd.normal - nd.pnorm;
	int hastexture = dot(pert, pert) > FTINY * FTINY;
	nd.pdot = -dot(data.ray_direction, nd.pnorm);
	if (nd.pdot < 0.0f) {		/* fix orientation from raynormal in raytrace.c */
		nd.pnorm += 2.0f * nd.pdot * data.ray_direction;
		nd.pdot = -nd.pdot;
	}
	if (nd.pdot < 0.001f)
		nd.pdot = 0.001f;			/* non-zero for dirnorm() */

	// if it's a face or a ring label as flat (currently we only support triangles, so everything is flat)
	nd.specfl |= SP_FLAT;

	/* modify material color */
	//nd.mcolor *= rtTex3D(rtTextureId id, texcoord.x, texcoord.y, texcoord.z).xyz;

	/* compute Fresnel approx. */
	float fest = 0.0f;
	if (nd.specfl & SP_PURE && nd.rspec >= FRESTHRESH) {
		fest = FRESNE(nd.pdot);
		nd.rspec += fest * (1.0f - nd.rspec);
	}

	/* compute transmission */
	nd.tdiff = nd.tspec = nd.trans = 0.0f; // because it's opaque
#ifdef TRANSMISSION
	nd.prdir = data.ray_direction;
	if (data.mat.params.n.trans > 0.0f) { // data.mat.type == MAT_TRANS
		nd.trans = data.mat.params.n.trans * (1.0f - nd.rspec);
		nd.tspec = nd.trans * data.mat.params.n.tspec;
		nd.tdiff = nd.trans - nd.tspec;
		if (nd.tspec > FTINY) {
			nd.specfl |= SP_TRAN;

							/* check threshold */
			if (!(nd.specfl & SP_PURE) && specthresh >= nd.tspec - FTINY)
				nd.specfl |= SP_TBLT;
			if (!prd.ambient_depth && hastexture) {
				if (dot(nd.prdir - pert, nd.normal) < -FTINY)
					nd.prdir = normalize(nd.prdir - pert);	/* OK */
			}
		}
	}

	/* diffuse reflection */
	nd.rdiff = 1.0f - nd.trans - nd.rspec;

	/* transmitted ray */
	if ((nd.specfl&(SP_TRAN | SP_PURE | SP_TBLT)) == (SP_TRAN | SP_PURE) && rayorigin(new_prd, prd, nd.mcolor * nd.tspec, 0, 0)) {
#ifdef DAYSIM_COMPATIBLE
		new_prd.dc = daysimNext(prd.dc);
#endif
		setupPayload(new_prd);
		Ray trans_ray = make_Ray(nd.hit, nd.prdir, RADIANCE_RAY, ray_start(nd.hit, nd.prdir, nd.normal, RAY_START), new_prd.tmax);
		rtTrace(top_object, trans_ray, new_prd);
		new_prd.result *= nd.mcolor * nd.tspec;
		result += new_prd.result;
#ifdef DAYSIM_COMPATIBLE
		daysimAddScaled(prd.dc, new_prd.dc, nd.mcolor.x * nd.tspec);
#endif
		if (nd.tspec >= 1.0f - FTINY) {
			/* completely transparent */
			prd.mirror = new_prd.mirror * nd.mcolor * nd.tspec;
			prd.mirror_distance = data.t + new_prd.mirror_distance;
			prd.distance = data.t + new_prd.distance;
		}
		else if (nd.tspec > nd.tdiff + nd.rdiff)
			prd.distance = data.t + rayDistance(new_prd);
		resolvePayload(prd, new_prd);
	}
#endif

	// return if it's a shadow ray, which it isn't

	/* get specular reflection */
	if (nd.rspec > FTINY) {
		nd.specfl |= SP_REFL;

		/* compute specular color */
		if (data.mat.type != MAT_METAL) {
			nd.scolor = make_float3(nd.rspec);
		} else {
			if (fest > FTINY) {
				float d = data.mat.params.n.spec * (1.0f - fest);
				nd.scolor = fest + nd.mcolor * d;
			} else {
				nd.scolor = nd.mcolor * nd.rspec;
			}
		}

		/* check threshold */
		if (!(nd.specfl & SP_PURE) && specthresh >= nd.rspec - FTINY) {
			nd.specfl |= SP_RBLT;
		}
	}

	/* reflected ray */
	if ((nd.specfl&(SP_REFL | SP_PURE | SP_RBLT)) == (SP_REFL | SP_PURE) && rayorigin(new_prd, prd, nd.scolor, 1, 0)) {
#ifdef DAYSIM_COMPATIBLE
		new_prd.dc = daysimNext(prd.dc);
#endif
		setupPayload(new_prd);
		float3 vrefl = reflect(data.ray_direction, nd.pnorm);
		Ray refl_ray = make_Ray(nd.hit, vrefl, RADIANCE_RAY, ray_start(nd.hit, vrefl, nd.normal, RAY_START), new_prd.tmax);
		rtTrace(top_object, refl_ray, new_prd);
		new_prd.result *= nd.scolor;
		prd.mirror = new_prd.result;
		result += new_prd.result;
		prd.mirror_distance = data.t;
#ifdef DAYSIM_COMPATIBLE
		daysimAddScaled(prd.dc, new_prd.dc, nd.scolor.x);
#endif
		if (nd.specfl & SP_FLAT && (prd.ambient_depth || !hastexture))
			prd.mirror_distance += rayDistance(new_prd);
		resolvePayload(prd, new_prd);
	}

	if (!(nd.specfl & SP_PURE && nd.rdiff <= FTINY && nd.tdiff <= FTINY)) { /* not 100% pure specular */
		/* checks *BLT flags */
		if (!(nd.specfl & SP_PURE))
			result += gaussamp(&nd, data.ray_direction, prd);

#ifdef AMBIENT
		/* ambient from this side */
		if (nd.rdiff > FTINY) {
			float3 aval = nd.mcolor * nd.rdiff;	/* modified by material color */
			if (nd.specfl & SP_RBLT)	/* add in specular as well? */
				aval += nd.scolor;
			result += multambient(aval, nd.normal, nd.pnorm, nd.hit, data.mat.params.n.ambincl, prd);	/* add to returned color */
		}

#ifdef TRANSMISSION
		/* ambient from other side */
		if (nd.tdiff > FTINY) {
			float3 aval = nd.mcolor;	/* modified by material color */
			if (nd.specfl & SP_TBLT)
				aval *= nd.trans;
			else
				aval *= nd.tdiff;
			result += multambient(aval, -nd.normal, -nd.pnorm, nd.hit, data.mat.params.n.ambincl, prd);	/* add to returned color */
		}
#endif /* TRANSMISSION */
#endif /* AMBIENT */

		/* add direct component */
		// This is the call to direct() in source.c
		// Let's start at line 447, and not bother with sorting for now

		// compute direct lighting
		PerRayData_shadow shadow_prd;
#ifdef DAYSIM_COMPATIBLE
		shadow_prd.dc = daysimNext(prd.dc);
#endif
		Ray shadow_ray = make_Ray(nd.hit, nd.pnorm, SHADOW_RAY, RAY_START, RAY_END);

		/* contributions from distant lights (mainly the sun) */
		unsigned int num_lights = lights.size();
		for (unsigned int i = 0u; i < num_lights; i++) {
			const DistantLight light = lights[i];
			if ( light.casts_shadow ) {
				shadow_prd.target = i;
				shadow_ray.direction = normalize( light.pos ); //TODO implement direct jitter for distant light sources
				shadow_ray.tmin = ray_start(nd.hit, shadow_ray.direction, nd.normal, RAY_START);
				shadow_ray.tmax = RAY_END;
				result += dirnorm(&shadow_ray, &shadow_prd, &nd, light.solid_angle, data.ray_direction, prd);
			}
		}

#ifdef LIGHTS
		/* contributions from nearby lights */
		num_lights = lindex_buffer.size();
		for (unsigned int i = 0u; i < num_lights; i++) {
			const uint3 v_idx = lindex_buffer[i];

			const float3 r0 = vertex_buffer[v_idx.x] - nd.hit;
			const float3 r1 = vertex_buffer[v_idx.y] - nd.hit;
			const float3 r2 = vertex_buffer[v_idx.z] - nd.hit;
			float3 rdir = ( r0 + r1 + r2 ) / 3.0f;

			const unsigned int divs = flatpart( rdir, r0, r1, r2, prd.weight ); //TODO divisions should be smaller closer to the light source
			const float step = 1.0f / divs;

			for ( int j = 0; j < divs; j++ )
				for ( int k = 0; k < divs; k++ ) {
					float2 lambda = make_float2( step * j, step * k );
					const float3 p0 = barycentric( lambda, r0, r1, r2, k + j >= divs );

					lambda = make_float2( step * ( j + 1 ), step * k );
					const float3 p1 = barycentric( lambda, r0, r1, r2, k + j >= divs );

					lambda = make_float2( step * j, step * ( k + 1 ) );
					const float3 p2 = barycentric( lambda, r0, r1, r2, k + j >= divs );

					const float omega = solid_angle( p0, p1, p2 );

					if ( omega > FTINY ) {
						/* from nextssamp in srcsamp.c */
						rdir = ( p0 + p1 + p2 ) / 3.0f;
						if ( dstrsrc > FTINY ) {
							/* jitter sample using random barycentric coordinates */
							lambda = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
							float3 vpos = barycentric( lambda, p0, p1, p2, lambda.x + lambda.y >= 1.0f );
							rdir += dstrsrc * ( vpos - rdir );
						}

						shadow_prd.target = -v_idx.x - 1; //TODO find a better way to identify surface
						shadow_ray.direction = normalize( rdir );
						shadow_ray.tmin = ray_start(nd.hit, shadow_ray.direction, nd.normal, RAY_START);
						shadow_ray.tmax = length(rdir) * 1.0001f;
						result += dirnorm(&shadow_ray, &shadow_prd, &nd, omega, data.ray_direction, prd);
					}
				}
		}
#endif /* LIGHTS */
	}

	// pass the color back up the tree
	prd.result = result;
	return prd;
}

/* compute source contribution */
RT_METHOD float3 dirnorm(Ray *shadow_ray, PerRayData_shadow *shadow_prd, const NORMDAT *nd, const float& omega, const float3& ray_dir, PerRayData_radiance &prd)
{
	float3 cval = make_float3( 0.0f );
	float ldot = dot(nd->pnorm, shadow_ray->direction);

#ifdef TRANSMISSION
	if (ldot < 0.0f ? nd->trans <= FTINY : nd->trans >= 1.0f - FTINY)
#else
	if ( ldot <= FTINY )
#endif
		return cval;
	
	/* Fresnel estimate */
	float lrdiff = nd->rdiff;
	float ltdiff = nd->tdiff;
	if (nd->specfl & SP_PURE && nd->rspec >= FRESTHRESH && (lrdiff > FTINY) | (ltdiff > FTINY)) {
		float dtmp = 1.0f - FRESNE(fabs(ldot));
		lrdiff *= dtmp;
		ltdiff *= dtmp;
	}

	if (ldot > FTINY && lrdiff > FTINY) {
		/*
		 *  Compute and add diffuse reflected component to returned
		 *  color.  The diffuse reflected component will always be
		 *  modified by the color of the material.
		 */
		float dtmp = ldot * omega * lrdiff * M_1_PIf;
		cval += nd->mcolor * dtmp;
	}
#ifdef TRANSMISSION
	if (ldot < -FTINY && ltdiff > FTINY) {
		/*
		 *  Compute diffuse transmission.
		 */
		float dtmp = -ldot * omega * ltdiff * M_1_PIf;
		cval += nd->mcolor * dtmp;
	}
#endif
	if (ldot > FTINY && (nd->specfl&(SP_REFL | SP_PURE)) == SP_REFL) {
		/*
		 *  Compute specular reflection coefficient using
		 *  Gaussian distribution model.
		 */
		/* roughness */
		float dtmp = nd->alpha2;
		/* + source if flat */
		if (nd->specfl & SP_FLAT)
			dtmp += omega * 0.25f * M_1_PIf;
		/* half vector */
		float3 vtmp = shadow_ray->direction - ray_dir;
		float d2 = dot(vtmp, nd->pnorm);
		d2 *= d2;
		float d3 = dot( vtmp, vtmp );
		float d4 = (d3 - d2) / d2;
		/* new W-G-M-D model */
		dtmp = expf(-d4/dtmp) * d3 / (M_PIf * d2*d2 * dtmp);
		/* worth using? */
		if (dtmp > FTINY) {
			dtmp *= ldot * omega;
			cval += nd->scolor * dtmp;
		}
	}
#ifdef TRANSMISSION
	if (ldot < -FTINY && (nd->specfl&(SP_TRAN | SP_PURE)) == SP_TRAN) {
		/*
		 *  Compute specular transmission.  Specular transmission
		 *  is always modified by material color.
		 */
						/* roughness + source */
		float dtmp = nd->alpha2 + omega * M_1_PIf;
						/* Gaussian */
		dtmp = expf((2.0f * dot(nd->prdir, shadow_ray->direction) - 2.0f) / dtmp) / (M_PIf * dtmp); // may need to perturb direction
						/* worth using? */
		if (dtmp > FTINY) {
			dtmp *= nd->tspec * omega * sqrtf(-ldot / nd->pdot);
			cval += nd->mcolor * dtmp;
		}
	}
#endif

	/* from direct() in source.c */
	if (fmaxf(cval) <= 0.0f)
		return cval;

	// cast shadow ray
	shadow_prd->result = make_float3(0.0f);
#ifdef CONTRIB
	shadow_prd->rcoef = prd.rcoef * cval;
#endif
#ifdef ANTIMATTER
	shadow_prd->mask = prd.mask;
	shadow_prd->inside = prd.inside;
#endif
#ifdef DAYSIM_COMPATIBLE
	daysimSet(shadow_prd->dc, 0.0f);
#endif
	rtTrace(top_object, *shadow_ray, *shadow_prd);

#ifdef DAYSIM_COMPATIBLE
	daysimAddScaled(prd.dc, shadow_prd->dc, cval.x);
#endif
	return cval * shadow_prd->result;
}

// sample Gaussian specular
RT_METHOD float3 gaussamp(const NORMDAT *nd, const float3& ray_dir, PerRayData_radiance &prd)
{
	float3 rcol = make_float3( 0.0f );

	/* This section is based on the gaussamp method in normal.c */
	if ((nd->specfl & (SP_REFL | SP_RBLT)) != SP_REFL && (nd->specfl & (SP_TRAN | SP_TBLT)) != SP_TRAN)
		return rcol;

	PerRayData_radiance gaus_prd;
	Ray gaus_ray = make_Ray(nd->hit, nd->pnorm, RADIANCE_RAY, RAY_START, RAY_END);

	float d;

	/* set up sample coordinates */
	float3 u = getperpendicular(nd->pnorm); // prd.state?
	float3 v = cross(nd->pnorm, u);

	unsigned int nstarget, nstaken, ntrials;

	/* compute reflection */
	if ((nd->specfl & (SP_REFL | SP_RBLT)) == SP_REFL && rayorigin(gaus_prd, prd, nd->scolor, 1, 1)) { //TODO the ambient depth increment is a hack to prevent the sun from affecting specular values
		float3 scolor = nd->scolor;
		nstarget = 1;
		if (specjitter > 1.5f) {	/* multiple samples? */ // By default it's 1.0
			nstarget = specjitter * prd.weight + 0.5f;
			if ( gaus_prd.weight <= minweight * nstarget )
				nstarget = gaus_prd.weight / minweight;
			if ( nstarget > 1 ) {
				d = 1.0f / nstarget;
				scolor *= d; //scolor, stored as ray rcoef
#ifdef CONTRIB
				gaus_prd.rcoef *= d;
#endif
				gaus_prd.weight *= d; // TODO make sure weight isn't changed by hit programs
			} else
				nstarget = 1;
		}
		float3 scol = make_float3( 0.0f );
#ifdef DAYSIM_COMPATIBLE
		DaysimCoef dc = daysimNext(prd.dc);
		if (nstarget > 1) {
			daysimSet(dc, 0.0f);
			gaus_prd.dc = daysimNext(dc);
		} else
			gaus_prd.dc = dc;
#endif
		//dimlist[ndims++] = (int)(size_t)np->mp;
		unsigned int maxiter = MAXITER * nstarget;
		for (nstaken = ntrials = 0; nstaken < nstarget && ntrials < maxiter; ntrials++) {
			float2 rv = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) ); // should be evenly distributed in both dimensions
			d = 2.0f * M_PIf * rv.x;
			float cosp = cosf( d );
			float sinp = sinf( d );
			if ( ( 0.0f <= specjitter ) && ( specjitter < 1.0f ) )
				rv.y = 1.0f - specjitter * rv.y;
			if ( rv.y <= FTINY )
				d = 1.0f;
			else
				d = sqrtf(nd->alpha2 * -logf(rv.y));
			float3 h = nd->pnorm + d * (cosp * u + sinp * v);
			d = -2.0f * dot( h, ray_dir ) / ( 1.0f + d*d );
			gaus_ray.direction = ray_dir + h * d;

			/* sample rejection test */
			if ((d = dot(gaus_ray.direction, nd->normal)) <= FTINY)
				continue;

			gaus_ray.direction = normalize( gaus_ray.direction );
			gaus_ray.tmin = ray_start(nd->hit, gaus_ray.direction, nd->normal, RAY_START);
			gaus_ray.tmax = gaus_prd.tmax;

			setupPayload(gaus_prd);
			//if (nstaken) // check for prd data that needs to be cleared
			rtTrace(top_object, gaus_ray, gaus_prd);
			resolvePayload(prd, gaus_prd);

			/* W-G-M-D adjustment */
			if (nstarget > 1) {	
				d = 2.0f / (1.0f - dot(ray_dir, nd->normal) / d);
				scol += gaus_prd.result * d;
#ifdef DAYSIM_COMPATIBLE
				daysimAddScaled(dc, gaus_prd.dc, d);
#endif
			} else {
				rcol += gaus_prd.result * scolor;
#ifdef DAYSIM_COMPATIBLE
				daysimAddScaled(prd.dc, gaus_prd.dc, scolor.x);
#endif
			}

			++nstaken;
		}
		/* final W-G-M-D weighting */
		if (nstarget > 1) {
			scol *= scolor;
			d = (float)nstarget / ntrials;
			rcol += scol * d;
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, dc, scolor.x * d);
#endif
		}
		//ndims--;
	}

#ifdef TRANSMISSION
	/* compute transmission */
	float3 mcolor = nd->mcolor * nd->tspec;	/* modified by color */
	if ((nd->specfl & (SP_TRAN | SP_TBLT)) == SP_TRAN && rayorigin(gaus_prd, prd, mcolor, 0, 0)) {
		nstarget = 1;
		if (specjitter > 1.5f) {	/* multiple samples? */ // By default it's 1.0
			nstarget = specjitter * prd.weight + 0.5f;
			if ( gaus_prd.weight <= minweight * nstarget )
				nstarget = gaus_prd.weight / minweight;
			if ( nstarget > 1 ) {
				d = 1.0f / nstarget;
				mcolor *= d; //mcolor, stored as ray rcoef
#ifdef CONTRIB
				gaus_prd.rcoef *= d;
#endif
				gaus_prd.weight *= d; // TODO make sure weight isn't changed by hit programs
			} else
				nstarget = 1;
		}
		//dimlist[ndims++] = (int)(size_t)np->mp;
		unsigned int maxiter = MAXITER * nstarget;
		for (nstaken = ntrials = 0; nstaken < nstarget && ntrials < maxiter; ntrials++) {
			float2 rv = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) ); // should be evenly distributed in both dimensions
			d = 2.0f * M_PIf * rv.x;
			float cosp = cosf( d );
			float sinp = sinf( d );
			if ( ( 0.0f <= specjitter ) && ( specjitter < 1.0f ) )
				rv.y = 1.0f - specjitter * rv.y;
			if ( rv.y <= FTINY )
				d = 1.0f;
			else
				d = sqrtf(nd->alpha2 * -logf(rv.y));
			gaus_ray.direction = nd->prdir + d * (cosp * u + sinp * v); // ray direction is perturbed

			/* sample rejection test */
			if (dot(gaus_ray.direction, nd->normal) >= -FTINY)
				continue;

			gaus_ray.direction = normalize( gaus_ray.direction );
			gaus_ray.tmin = ray_start(nd->hit, gaus_ray.direction, nd->normal, RAY_START);
			gaus_ray.tmax = gaus_prd.tmax;

#ifdef DAYSIM_COMPATIBLE
			gaus_prd.dc = daysimNext(prd.dc);
#endif
			setupPayload(gaus_prd);
			//if (nstaken) // check for prd data that needs to be cleared
			rtTrace(top_object, gaus_ray, gaus_prd);
			resolvePayload(prd, gaus_prd);
			rcol += gaus_prd.result * mcolor;
			++nstaken;
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, gaus_prd.dc, mcolor.x);
#endif
		}
		//ndims--;
	}
#endif
	//return make_float3(0.0f);
	return rcol;
}

#ifdef AMBIENT
// Compute the ambient component and multiply by the coefficient.
RT_METHOD float3 multambient(float3 aval, const float3& normal, const float3& pnormal, const float3& hit, const unsigned int& ambincl, PerRayData_radiance &prd)
{
	if (exposure && !prd.ambient_depth) // TODO exposure is hack to check if we are running rvu
		return make_float3(0.0f);

	int do_ambient = 1;
	float 	d;

	if (ambdiv <= 0)			/* no ambient calculation */
		goto dumbamb;
						/* check number of bounces */
	if (prd.ambient_depth >= ambounce)
		goto dumbamb;
						/* check ambient list */
	if (!ambincl)
		goto dumbamb;

	if (ambacc > FTINY && navsum != 0) {			/* ambient storage */
		//if (tracktime)				/* sort to minimize thrashing */
		//	sortambvals(0);

		/* interpolate ambient value */
		//acol = make_float3( 0.0f );
		//d = sumambient(acol, r, normal, rdepth, &atrunk, thescene.cuorg, thescene.cusize);
		PerRayData_ambient ambient_prd;
		ambient_prd.result = make_float3( 0.0f );
		ambient_prd.surface_point = hit;
		ambient_prd.surface_normal = pnormal;
		ambient_prd.ambient_depth = prd.ambient_depth;
		ambient_prd.wsum = 0.0f;
		ambient_prd.weight = prd.weight;
#ifdef DAYSIM_COMPATIBLE
		ambient_prd.dc = daysimNext(prd.dc);
		daysimSet(ambient_prd.dc, 0.0f);
#endif
#ifdef HIT_COUNT
		ambient_prd.hit_count = 0;
#endif
		const float tmax = ray_start(hit, AMBIENT_RAY_LENGTH);
		Ray ambient_ray = make_Ray(hit - normal * tmax, normal, AMBIENT_RAY, 0.0f, 2.0f * tmax);
		rtTrace(top_ambient, ambient_ray, ambient_prd, RT_VISIBILITY_ALL, RT_RAY_FLAG_DISABLE_CLOSESTHIT);
#ifdef HIT_COUNT
		prd.hit_count += ambient_prd.hit_count;
#endif
		if (ambient_prd.wsum > FTINY) { // TODO if miss program is called, set wsum = 1.0f or place this before ambacc == 0.0f
			ambient_prd.result *= 1.0f / ambient_prd.wsum;
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, ambient_prd.dc, aval.x / ambient_prd.wsum);
#endif
			return aval * ambient_prd.result;
		}
		//rdepth++;				/* need to cache new value */
		//d = makeambient(acol, r, normal, rdepth-1); //TODO implement as miss program for ambient ray
		//rdepth--;
		//if ( dot( ambient_prd.result, ambient_prd.result) > FTINY) { // quick check to see if a value was returned by miss program
		//	return aval * ambient_prd.result;		/* got new value */
		//}

		do_ambient = !prd.ambient_depth && ambdiv_final;
	}
	if (do_ambient) {			/* no ambient storage */
		/* Option to show error if nothing found */
		if (ambdiv_final < 0)
			rtThrow(RT_EXCEPTION_CUSTOM - ambdiv_final);

		float3 acol = aval;
#ifdef DAYSIM_COMPATIBLE
		DaysimCoef dc = daysimNext(prd.dc);
		daysimSet(dc, 0.0f);
		d = doambient(&acol, normal, pnormal, hit, prd, dc);
		if (d > FTINY)
			daysimAdd(prd.dc, dc);
#else
		d = doambient(&acol, normal, pnormal, hit, prd);
#endif
		if (d > FTINY)
			return acol;
	}
dumbamb:					/* return global value */
	if ((ambvwt <= 0) || (navsum == 0)) {
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * ambval.x);
#endif
		return aval * ambval;
	}
	float l = bright(ambval);			/* average in computations */
	if (l > FTINY) {
		d = (logf(l)*(float)ambvwt + avsum) / (float)(ambvwt + navsum);
		d = expf(d) / l;
		aval *= ambval;	/* apply color of ambval */
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * ambval.x * d);
#endif
	} else {
		d = expf( avsum / (float)navsum );
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * d);
#endif
	}
	return aval * d;
}

/* sample indirect hemisphere, based on samp_hemi in ambcomp.c */
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, PerRayData_radiance &prd, DaysimCoef dc)
#else
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, PerRayData_radiance &prd)
#endif
{
	float	d;
	float wt = prd.weight;

					/* set number of divisions */
	if (wt > (d = 0.8f * fmaxf(*rcol) * wt / (ambdiv_final * minweight))) // Ignore ambacc <= FTINY check because this is faking ambacc == 0 calc
		wt = d;			/* avoid ray termination */
	int n = sqrtf(ambdiv_final * wt) + 0.5f;
	int i = 1 + 5 * (ambacc > FTINY);	/* minimum number of samples */
	if (n < i)
		n = i;
	const int nn = n * n;
	float3 acol = make_float3( 0.0f );
	unsigned int sampOK = 0u;
					/* assign coefficient */
	float3 acoef = *rcol / nn;

	/* Setup from ambsample in ambcomp.c */
	PerRayData_radiance new_prd;
#ifdef DAYSIM_COMPATIBLE
	new_prd.dc = daysimNext(dc);
#endif

	Ray amb_ray = make_Ray( hit, pnormal, RADIANCE_RAY, RAY_START, RAY_END ); // Use normal point as temporary direction
	/* End ambsample setup */

					/* make tangent plane axes */
	float3 ux = getperpendicular( pnormal, prd.state );
	float3 uy = cross( pnormal, ux );
					/* sample divisions */
	for (i = n; i--; )
	    for (int j = n; j--; ) {
			if (!rayorigin(new_prd, prd, acoef, 1, 1))
				continue;

			//hp.sampOK += ambsample( &hp, i, j, normal, hit );
			/* ambsample in ambcomp.c */
			float2 spt = make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
			if (i > 0 && i < n - 1 && j > 0 && j < n - 1) /* avoid coincident samples */
				spt = 0.1f + 0.8f * spt;
			SDsquare2disk( spt, (j+spt.y) / n, (i+spt.x) / n );
			float zd = sqrtf( 1.0f - dot( spt, spt ) );
			amb_ray.direction = normalize( spt.x*ux + spt.y*uy + zd*pnormal );
			if (dot(amb_ray.direction, normal) <= 0) /* Prevent light leaks */
				continue;
			amb_ray.tmin = ray_start( hit, amb_ray.direction, normal, RAY_START );
			amb_ray.tmax = new_prd.tmax;
			//dimlist[ndims++] = AI(hp,i,j) + 90171;

			setupPayload(new_prd);
			//Ray amb_ray = make_Ray( hit, rdir, RADIANCE_RAY, RAY_START, new_prd.tmax );
			rtTrace(top_object, amb_ray, new_prd);
			resolvePayload(prd, new_prd);

			//ndims--;
			if ( isnan( new_prd.result ) ) // TODO How does this happen?
				continue;
			if ( new_prd.distance <= FTINY )
				continue;		/* should never happen */
			acol += new_prd.result * acoef;	/* add to our sum */
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(dc, new_prd.dc, acoef.x);
#endif
			sampOK++;
		}
	*rcol = acol;
	if ( !sampOK ) {		/* utter failure? */
		return( 0 );
	}
	if ( sampOK < nn ) {
		//hp.sampOK *= -1;	/* soft failure */
		return( 1 );
	}
	//n = ambssamp * wt + 0.5f;
	//if (n > 8) {			/* perform super-sampling? */
	//	ambsupersamp(hp, n);
	//	*rcol = hp.acol;
	//}
	return( 1 );			/* all is well */
}
#endif /* AMBIENT */

#ifdef LIGHTS
/* partition a flat source */
RT_METHOD unsigned int flatpart( const float3& v, const float3& r0, const float3& r1, const float3& r2, const float& weight )
{
	//float3 vp = source[si->sn].snorm;
	//if ( dot( v, vp ) <= 0.0f )		/* behind source */
	//	return 0u;

	if ( srcsizerat <= FTINY )
		return 1u;

	float d;

	/* Find longest edge */
	float3 vp = r1 - r0;
	float d2 = dot( vp, vp );
	vp = r2 - r1;
	if ( ( d = dot( vp, vp ) ) > d2 )
		d2 = d;
	vp = r2 - r0;
	if ( ( d = dot( vp, vp ) ) > d2 )
		d2 = d;

	/* Find minimum partition size */
	d = srcsizerat / weight;
	d *= d * dot( v, v );

	/* Find number of partions */
	d2 /= d;
	if ( d2 < 1.0f )
		return 1u;
	if ( d2 > ( d = MAXSPART >> 1 ) ) // Divide maximum partitions by two going from rectangle to triangle
		d2 = d;
	return (unsigned int)sqrtf( d2 );
}

/* Solid angle calculation from "The solid angle of a plane triangle", A van Oosterom and J Strackee */
RT_METHOD float solid_angle( const float3& r0, const float3& r1, const float3& r2 )
{
	const float l0 = length( r0 );
	const float l1 = length( r1 );
	const float l2 = length( r2 );

	const float numerator = dot( r0, cross( r1, r2 ) );
	const float denominator = l0 * l1 * l2 + dot( r0, r1 ) * l2 + dot( r0, r2 ) * l1 + dot( r1, r2 ) * l0;
	return 2.0f * fabsf( atan2( numerator, denominator ) );
}

/* Compute point from barycentric coordinates and flip if outside triangle */
RT_METHOD float3 barycentric( float2& lambda, const float3& r0, const float3& r1, const float3& r2, const int flip )
{
	if ( flip )
		lambda = 1.0f - lambda;
	return r0 * ( 1.0f - lambda.x - lambda.y ) + r1 * lambda.x + r2 * lambda.y;
}
#endif /* LIGHTS */

/* convert 1-dimensional sample to 2 dimensions, based on multisamp.c */
//RT_METHOD float2 multisamp2(float r)	/* 1-dimensional sample [0,1) */
//{
//	int	j;
//	register int	k;
//	int2	ti;
//	float	s;
//
//	ti = make_int2( 0 );
//	j = 8;
//	while (j--) {
//		k = s = r*(1<<2);
//		r = s - k;
//		ti += ti + make_int2( ((k>>2) & 1), ((k>>1) & 1) );
//	}
//	ti += make_int2( frandom() );
//	ti *= 1.0f/256.0f;
//}

/* hash a set of integer values */
//RT_METHOD int ilhash(int3 d)
//{
//	register int  hval;
//
//	hval = 0;
//	hval ^= d.x * 73771;
//	hval ^= d.y * 96289;
//	hval ^= d.z * 103699;
//	return(hval & 0x7fffffff);
//}
