#include "hip/hip_runtime.h"
/*
 *  fisheye.cu - program for removal of fisheye distortion to acheive equiangular projection on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>

using namespace optix;

struct Transform
{
	optix::Matrix<3, 3> m;
};

/* Program variables */
rtDeclareVariable(int, data, , ); /* texture ID */
rtDeclareVariable(int, type, , ); /* type of data (true for float) */
rtDeclareVariable(Transform, transform, , ); /* transformation matrix */

// Correct lens distortion for Sigma 4.5mm fisheye lens.
RT_CALLABLE_PROGRAM float3 fisheye(const float3 direction, const float3 ignore)
{
	const float3 dir = transform.m * direction;

	float phi = acosf(dir.z); // phi = 0 +z, phi = pi/2 xy equator, phi = pi -z
	float theta = atan2f(dir.x, -dir.y); // theta = 0 along -y axis

	float x = phi * M_2_PIf; // normalize
	float x2 = x * x;
	float x3 = x2 * x;
	float x4 = x3 * x;
	float y = 0.7617f * (x4 * x) - 1.9134f * x4 + 1.5577f * x3 - 0.6087f * x2 + 1.2056f * x; // fisheye correction

	if (y > 1.0f) // outside of image
		return make_float3(0.0f);

	y /= 2.0f; // radius
	float u = 0.5f + y * sin(theta);
	float v = 0.5f + y * cos(theta);

	/* Renormalize to remove edges */
	uint3 ne = rtTexSize(data);
	u = (u * (ne.x - 1) + 0.5f) / ne.x;
	v = (v * (ne.y - 1) + 0.5f) / ne.y;

	if (type)
		return make_float3(rtTex2D<float>(data, u, v)); // this is corr from source.cal
	float4 tex = rtTex2D<float4>(data, u, v);
	return make_float3(tex.y, tex.z, tex.w);
}
