#include "hip/hip_runtime.h"
/*
 *  isotropsky.cu - program for Tregenza sky patch identification on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>

// Calculate the Tregenza patch based on tregenza.cal.
RT_CALLABLE_PROGRAM int tbin(const float3 direction)
{
	if (direction.z >= 1.0f) return 145;
	if (direction.z < 0.0f) return 0;
	float alt = asinf(direction.z) * 180 * M_1_PIf;
	float azi = atan2f(direction.x, direction.y) * 180 * M_1_PIf;
	if (azi < 0.0f) azi += 360.0f;

	int x = 0, inc = 0;
	if (alt < 12.0f) {
		x = 1; inc = 12;
	}
	else if (alt < 24.0f) {
		x = 31; inc = 12;
	}
	else if (alt < 36.0f) {
		x = 61; inc = 15;
	}
	else if (alt < 48.0f) {
		x = 85; inc = 15;
	}
	else if (alt < 60.0f) {
		x = 109; inc = 20;
	}
	else if (alt < 72.0f) {
		x = 127; inc = 30;
	}
	else if (alt < 84.0f) {
		x = 139; inc = 60;
	}
	else
		return 145;

	int y = azi + inc / 2;
	if (y < 360)
		x += y / inc;

	return x;
}