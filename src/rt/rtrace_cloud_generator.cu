#include "hip/hip_runtime.h"
/*
 *  rtrace_cloud_generator.cu - entry point for geometry sampling for individual ray tracing on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_common.h"
#include "optix_point_common.h"

#define RING_BUFFER_SIZE	8

using namespace optix;

/* Contex variables */
rtBuffer<RayData, 2>             ray_buffer;
rtBuffer<PointDirection, 3>      seed_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(rtObject,      top_irrad, , );
rtDeclareVariable(unsigned int,  imm_irrad, , ) = 0u; /* Immediate irradiance (-I) */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

RT_PROGRAM void cloud_generator()
{
	PerRayData_point_cloud prd;

	// Init random state
	init_rand(&prd.state, launch_index.x + launch_dim.x * launch_index.y);

	prd.index = make_uint3(launch_index, 0u);
	prd.seeds = seed_buffer.size().z;
	unsigned int loop = 2u * prd.seeds; // Prevent infinite looping

	float3 point_ring[RING_BUFFER_SIZE];
	float3 dir_ring[RING_BUFFER_SIZE];
	unsigned int ring_start = 0, ring_end = 0, ring_full = 0;

	float tmin = ray_start(ray_buffer[launch_index].origin, RAY_START);
	float tmax;
	if ( imm_irrad ) {
		tmax = 2.0f * tmin;
		tmin = 0.0f;
	} else {
		// Zero or negative aft clipping distance indicates infinity
		tmax = ray_buffer[launch_index].max;
		if (tmax <= FTINY) {
			tmax = RAY_END;
		}
	}

	Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, POINT_CLOUD_RAY, tmin, tmax);

	while (prd.index.z < prd.seeds && loop--) {
		prd.forward = prd.reverse = make_float3(0.0f);
#ifdef ANTIMATTER
		prd.mask = 0u;
		prd.inside = 0;
#endif

		// Trace the current ray
		if ( imm_irrad )
			rtTrace(top_irrad, ray, prd);
		else
			rtTrace(top_object, ray, prd);

		// Add next forward ray to ring buffer
		if (isfinite(prd.point) && dot(prd.forward, prd.forward) > FTINY) { // NaN values will be false
			point_ring[ring_end] = prd.point;
			dir_ring[ring_end] = prd.forward;
			ring_end = (ring_end + 1) % RING_BUFFER_SIZE;
			ring_full = ring_start == ring_end;
		}

		// Add next reverse ray to ring buffer
		if (!ring_full && isfinite(prd.point) && dot(prd.reverse, prd.reverse) > FTINY) { // NaN values will be false
			point_ring[ring_end] = prd.point;
			dir_ring[ring_end] = prd.reverse;
			ring_end = (ring_end + 1) % RING_BUFFER_SIZE;
			ring_full = ring_start == ring_end;
		}

		if (!ring_full && ring_start == ring_end)
			break;

		// Prepare for next ray
		ray.origin = point_ring[ring_start];
		ray.direction = dir_ring[ring_start];
		ring_start = (ring_start + 1) % RING_BUFFER_SIZE;
		ring_full = 0;
		ray.tmin = ray_start(ray.origin, RAY_START);
		ray.tmax = RAY_END;
	}

	// If outdoors, there are no bounces, but we need to prevent junk data
	while (prd.index.z < prd.seeds) {
		clear(seed_buffer[prd.index]);
		prd.index.z++;
	}
}

RT_PROGRAM void exception()
{
#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
	uint3 index = make_uint3(launch_index, seed_buffer.size().z - 1u); // record error to last segment
	seed_buffer[index].pos = exceptionToFloat3(rtGetExceptionCode());
	seed_buffer[index].dir = make_float3( 0.0f );
#ifdef AMBIENT_CELL
	seed_buffer[index].cell = make_uint2(0);
#endif
}
