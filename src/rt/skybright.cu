#include "hip/hip_runtime.h"
/*
*  skybright.cu - program for CIE sky distribution on GPUs.
*/

#include "accelerad_copyright.h"

#include <optix_world.h>

struct Transform
{
	optix::Matrix<3,3> m;
};

/* Program variables */
rtDeclareVariable(unsigned int, type, , ); /* 1 for CIE clear, 2 for CIE overcast, 3 for uniform, 4 for CIE intermediate */
rtDeclareVariable(float,        zenith, , ); /* zenith brightness */
rtDeclareVariable(float,        ground, , ); /* ground plane brightness */
rtDeclareVariable(float,        factor, , ); /* normalization factor based on sun direction */
rtDeclareVariable(float3,       sun, , ); /* sun direction */
rtDeclareVariable(Transform,    transform, , ); /* transformation matrix */

// Calculate the sky brightness function for sunny and cloudy skies.
// This function replicates the algorithm in skybright.cal.
RT_CALLABLE_PROGRAM float3 skybr(const float3 direction, const float3 ignore)
{
	//rtPrintf("SkyBright Recieved (%f, %f, %f)\n", direction.x, direction.y, direction.z);

	const float3 dir = transform.m * direction;

	const float cosgamma = 0.999999f * optix::dot(dir, sun); // cosgamma = Dx*A8 + Dy*A9 + Dz*A10; // Adjusted to keep in range [-1,1]
	const float gamma = acosf(cosgamma); // gamma = Acos(cosgamma);		{ angle from sun to this point in sky }
	//float dz = dir.z;
	float sky = zenith; // unifsky = A2; select(A1, sunnysky, cloudysky, unifsky, intersky)

	if (type == 1u) { // CIE clear
		sky *= ( 0.91f + 10.0f * expf( -3.0f * gamma ) + 0.45f * cosgamma * cosgamma ) / factor;
		if ( dir.z > 0.01f ) {
			sky *= 1.0f - expf(-0.32f / dir.z );
		}
		//sunnysky = A2 * (.91 + 10*exp(-3*gamma) + .45*cosgamma*cosgamma)
	    //  *  if( Dz - .01, 1.0 - exp(-.32/Dz), 1.0) / A4;
	} else if (type == 2u) { // CIE overcast
		sky *= ( 1.0f + 2.0f * dir.z ) / 3.0f; // cloudysky = A2 * (1 + 2*Dz)/3;
	} else if (type == 4u) { // CIE intermediate
		float zt = acosf(sun.z); // zt = Acos(A10);			{ angle from zenith to sun }
		float eta = acosf(dir.z); // eta = Acos(Dz);			{ angle from zenith to this point in sky }
		sky *= ( ( 1.35f * sinf( 5.631f - 3.59f * eta ) + 3.12f ) * sinf( 4.396f - 2.6f * zt) + 6.37f - eta ) / 2.326f *
			expf( gamma * -0.563f * ( ( 2.629f - eta ) * ( 1.562f - zt ) + 0.812f ) ) / factor;
		//intersky = A2 * ( (1.35*sin(5.631-3.59*eta)+3.12)*sin(4.396-2.6*zt)
		//	+ 6.37 - eta ) / 2.326 *
		// exp(gamma*-.563*((2.629-eta)*(1.562-zt)+.812)) / A4;
	}

	const float a = powf(dir.z + 1.01f, 10.0f);
	const float b = powf(dir.z + 1.01f, -10.0f);

	const float skybr = (a * sky + b * ground) / (a + b); // wmean(a, x, b, y) = (a*x+b*y)/(a+b);
	return make_float3(skybr);
}