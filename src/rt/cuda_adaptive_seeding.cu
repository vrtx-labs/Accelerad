#include "hip/hip_runtime.h"
/*
 *  cuda_adaptive_seeding.cu - routines for adaptive seeding on GPUs.
 */

#include "accelerad_copyright.h"

#include <stdio.h>
#include <stdlib.h>
#define __STDC_FORMAT_MACROS
#include <inttypes.h>

#include "kmeans.h"

//#define PRINT_CUDA
#define MULTI_BLOCK

#define VALID_HORIZONTAL	0x01	/* Horizontal neighbor quad tree node is valid. */
#define VALID_VERTICAL		0x10	/* Vertical neighbor quad tree node is valid. */

#ifdef CAP_REGISTERS_PER_THREAD
#include "accelerad.h"
/* This is the maximum number of registers used by any cuda kernel in this in this file,
found by using the flag "-Xptxas -v" to compile in nvcc. This should be updated when
changes are made to the kernels. */
#ifdef RTX
#define REGISTERS_PER_THREAD	36	/* Registers per thread under CUDA 10.0 */
#else
#define REGISTERS_PER_THREAD	23	/* Registers per thread under CUDA 7.5 */
#endif
#endif

#ifdef __cplusplus
extern "C" {
#endif

// Ambient sample distribution based on Wang et al. (2009) "An efficient GPU-based approach for interactive global illumination"
__device__ inline static
PointDirection average_point_direction(const PointDirection& a, const PointDirection& b, const PointDirection& c, const PointDirection& d)
{
	PointDirection average;
	average.pos = (a.pos + b.pos + c.pos + d.pos) / 4.0f;
	//average.dir = optix::normalize(a.dir + b.dir + c.dir + d.dir);
	average.dir = a.dir + b.dir + c.dir + d.dir;
	const float length = optix::length(average.dir);
	if ( length > 0.0f )
		average.dir /= length;
	return average;
}

__device__ inline static
float geometric_error(const PointDirection& a, const PointDirection& b, const float alpha)
{
	return alpha * optix::length(a.pos - b.pos) + sqrtf(2.0f * fmaxf(1.0f - optix::dot(a.dir, b.dir), 0.0f));
}

__device__ inline static
unsigned int valid_hit(const PointDirection& hit)
{
	return optix::dot(hit.dir, hit.dir) > 0.0f && optix::dot(hit.pos, hit.pos) >= 0.0f;
}

static int CCALL isPowerOfTwo(unsigned int x)
{
  return ((x != 0) && !(x & (x - 1)));
}

static unsigned int CCALL calc_block_dim(const unsigned int maxThreadsPerBlock, const unsigned int levels)
{
	unsigned int blockDim = 1u;
	unsigned int size = maxThreadsPerBlock << 1;
	while ( size >>= 2 )
		blockDim <<= 1;
	if ( blockDim > (1u << levels) )
		blockDim = 1u << levels;
	return blockDim;
}

#ifndef MULTI_BLOCK
__device__ inline static
void reduce(float *error, const int level, const int idX, const int idY, const int width)
{
	int tid = idX + idY * width;
	unsigned int stride = 1u;
	float err = error[tid];

	for (int i = 0; i < level; i++) {
		unsigned int stride2 = stride << 1;
		if (!(idX % stride2) && !(idY % stride2)) {
			err += error[tid + stride];
			err += error[tid + stride * width];
			err += error[tid + stride * (width + 1)];
		
			error[tid] = err;
		}
		stride = stride2;
		__syncthreads();
	}
}

// Ambient sample distribution
__global__ static
void geometric_variation(PointDirection *deviceHits, int *seed,
				   const unsigned int width, const unsigned int height, const unsigned int levels, const float alpha)
{
	extern __shared__ PointDirection blockSharedMemory[];

	unsigned int idX = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idY = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tid = idX + idY * width;
	unsigned int sid = blockDim.x * threadIdx.y + threadIdx.x;

	float *err = (float*)malloc(levels * sizeof(float));
	unsigned int stride = 1u;

	PointDirection hit;
	unsigned int valid = idX < width && idY < height;
	if (valid) {
		hit = deviceHits[tid];
		valid = valid_hit(hit);
	}
	if (!valid)
		hit.pos.x = hit.pos.y = hit.pos.z = hit.dir.x = hit.dir.y = hit.dir.z = 0.0f;
	PointDirection accum = hit;
	blockSharedMemory[sid] = hit;
#ifdef PRINT_CUDA
		if (!tid)
			printf("mip_map_hits width=%i, height=%i, accum=%g,%g,%g, %g,%g,%g, valid=%i\n", width, height, accum.pos.x, accum.pos.y, accum.pos.z, accum.dir.x, accum.dir.y, accum.dir.z, valid);
#endif

	__syncthreads();

	/* Calculate geometric error for each hit point to each quad-tree node. */
	for (int i = 0; i < levels; i++) {
#ifdef PRINT_CUDA
		if (!tid)
			printf("mip_map_hits stride=%i, accum=%g,%g,%g\n", stride, accum.pos.x, accum.pos.y, accum.pos.z);
#endif
		unsigned int stride2 = stride << 1;

		if (!(idX % stride2) && !(idY % stride2)) {
			accum = average_point_direction(
				accum,
				blockSharedMemory[sid + stride],
				blockSharedMemory[sid + stride * width],
				blockSharedMemory[sid + stride * (width + 1)]
			);

			blockSharedMemory[sid] = accum;
		}

		__syncthreads();

		err[i] = valid ? geometric_error(hit, blockSharedMemory[sid - idX % stride2 - (idY % stride2) * blockDim.x], alpha) : 0.0f;
		stride = stride2;
	}

	__syncthreads();

	float *error = (float *)blockSharedMemory;

	for (int i = levels; i--; ) {
		unsigned int stride2 = stride >> 1;

		/* Calculate geometric error average at each quad-tree node. */
		error[tid] = err[i];

		__syncthreads();

		if (i) 
			reduce(error, i, idX, idY, width); // sum errors at this quad tree node

		/* Divide the pool proportinally to error at each quad-tree node. */
		if (idX < width && idY < height && !(idX % stride) && !(idY % stride)) {
			valid = 0u;
			if (idX + stride2 < width)
				valid |= VALID_HORIZONTAL;
			if (idY + stride2 < height)
				valid |= VALID_VERTICAL;
			float err[4];
			err[0] = error[tid];
			err[1] = (valid2 & VALID_HORIZONTAL) ? error[tid + stride2] : 0.0f;
			err[2] = (valid2 & VALID_VERTICAL) ? error[tid + stride2 * width] : 0.0f;
			err[3] = (valid2 & (VALID_HORIZONTAL | VALID_VERTICAL)) == (VALID_HORIZONTAL | VALID_VERTICAL) ? error[tid + stride2 * (width + 1)] : 0.0f;
			float errSum = err[0] + err[1] + err[2] + err[3];
			int seedSum = seed[tid];
			float scoreSum = errSum > 0.0f ? seedSum / errSum : 0.0f;

			int s[4];
			s[0] = scoreSum * err[0];
			s[1] = scoreSum * err[1];
			s[2] = scoreSum * err[2];
			s[3] = scoreSum * err[3];
			int diff = seedSum - s[0] - s[1] - s[2] - s[3];
#ifdef PRINT_CUDA
			if (!tid)
				printf("calc_score stride=%i, i=%i, errSum=%g, seedSum=%i, scoreSum=%g, diff=%i\n", stride, i, errSum, seedSum, scoreSum, diff);
#endif
			if (diff && errSum > 0.0f) {
				float max[3] = { 0.0f, 0.0f, 0.0f }; // Will store up to 3 maximum values in err[]
				int maxi[3] = { -1, -1, -1 }; // Will store the indices of up to 3 maximum values in err[]
				for (int j = 0; j < 4; j++) { // Find 3 largest values
					if (err[j] > max[0]) {
						max[2] = max[1]; maxi[2] = maxi[1];
						max[1] = max[0]; maxi[1] = maxi[0];
						max[0] = err[j]; maxi[0] = j;
					}
					else if (err[j] > max[1]) {
						max[2] = max[1]; maxi[2] = maxi[1];
						max[1] = err[j]; maxi[1] = j;
					}
					else if (err[j] > max[2]) {
						max[2] = err[j]; maxi[2] = j;
					}
				}
				if (diff > 2 && max[2] > 0.0f) {
					s[maxi[2]] += 1;
					diff -= 1;
				}
				if (diff > 1 && max[1] > 0.0f) {
					s[maxi[1]] += 1;
					diff -= 1;
				}
				if (diff && max[0] > 0.0f) {
					s[maxi[0]] += diff;
				}
			}

			seed[tid] = s[0];
			if (valid & VALID_HORIZONTAL)
				seed[tid + stride2] = s[1];
			if (valid & VALID_VERTICAL) {
				seed[tid + stride2 * width] = s[2];
				if (valid & VALID_HORIZONTAL)
					seed[tid + stride2 * (width + 1)] = s[3];
			}
		}

		__syncthreads();

		stride = stride2;
	}

	free(err);
}
#else /* MULTI_BLOCK */

__global__ static
void mip_map_hits(PointDirection *deviceHits, PointDirection *deviceMipMap,
				   const unsigned int width, const unsigned int height)
{
	extern __shared__ PointDirection blockSharedMemory[];

	unsigned int idX = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idY = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tid = idX + idY * width;
	unsigned int sid = blockDim.x * threadIdx.y + threadIdx.x;
	unsigned int offset = 0u;

	unsigned int stride = 1u;
	unsigned int levelWidth = width;
	unsigned int levelHeight = height;

	PointDirection hit;
	unsigned int valid = idX < width && idY < height;
	if (valid) {
		hit = deviceHits[tid];
		valid = valid_hit(hit);
	}
	if (!valid)
		hit.pos.x = hit.pos.y = hit.pos.z = hit.dir.x = hit.dir.y = hit.dir.z = 0.0f;
	PointDirection accum = hit;
	blockSharedMemory[sid] = hit;
#ifdef PRINT_CUDA
		if (!tid)
			printf("mip_map_hits width=%i, height=%i, accum=%g,%g,%g, %g,%g,%g, valid=%i\n", width, height, accum.pos.x, accum.pos.y, accum.pos.z, accum.dir.x, accum.dir.y, accum.dir.z, valid);
#endif

	__syncthreads();

	/* Calculate geometric error for each hit point to each quad-tree node. */
	while (stride < blockDim.x) {
#ifdef PRINT_CUDA
		if (!tid)
			printf("mip_map_hits stride=%i, offset=%i, accum=%g,%g,%g\n", stride, offset, accum.pos.x, accum.pos.y, accum.pos.z);
#endif
		unsigned int stride2 = stride << 1;
		levelWidth = (levelWidth - 1) / 2 + 1;
		levelHeight = (levelHeight - 1) / 2 + 1;

		if (!(idX % stride2) && !(idY % stride2)) {
			accum = average_point_direction(
				accum,
				blockSharedMemory[sid + stride],
				blockSharedMemory[sid + stride * blockDim.x],
				blockSharedMemory[sid + stride * (blockDim.x + 1)]
			);

			blockSharedMemory[sid] = accum;
			deviceMipMap[offset + (idX + idY * levelWidth) / stride2] = accum;
		}
#ifdef PRINT_CUDA
		if (!tid)
			printf("mip_map_hits width=%i, height=%i, accum=%g,%g,%g\n", levelWidth, levelHeight, accum.pos.x, accum.pos.y, accum.pos.z);
#endif

		__syncthreads();

		stride = stride2;
		offset += levelWidth * levelHeight;
	}
}

__global__ static
void calc_error(PointDirection *deviceHits, PointDirection *deviceMipMap, float *error,
				   const unsigned int width, const unsigned int height, const unsigned int levels, float alpha)
{
	unsigned int idX = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int idY = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int tid = idX + idY * width;

	unsigned int stride = 1u;
	unsigned int levelWidth = width;
	unsigned int levelHeight = height;

	if (idX < width && idY < height) {
		PointDirection hit = deviceHits[tid];
		unsigned int valid = valid_hit(hit);
		if (!valid)
			hit.pos.x = hit.pos.y = hit.pos.z = hit.dir.x = hit.dir.y = hit.dir.z = 0.0f;

		PointDirection *mipMapLevel = deviceMipMap;

		/* Calculate geometric error for each hit point to each quad-tree node. */
		for (unsigned int i = 0u; i < levels; i++) {
#ifdef PRINT_CUDA
			if (!tid)
				printf("calc_error stride=%i, i=%i, valid=%i\n", stride, i, valid);
#endif
			stride <<= 1;
			levelWidth = (levelWidth - 1) / 2 + 1;
			levelHeight = (levelHeight - 1) / 2 + 1;

			error[tid + i * width * height] = valid ? geometric_error(hit, mipMapLevel[idX / stride + idY / stride * levelWidth], alpha) : 0.0f;
			mipMapLevel += levelWidth * levelHeight;
		}
	}
}

__global__ static
void reduce_error(float *error, const unsigned int width, const unsigned int height, const unsigned int levels, const unsigned int scale)
{
	unsigned int idX = scale * (blockDim.x * blockIdx.x + threadIdx.x);
	unsigned int idY = scale * (blockDim.y * blockIdx.y + threadIdx.y);
	unsigned int tid = idX + idY * width;
	unsigned int valid = idX < width && idY < height; 

	for (unsigned int j = 1u; j < levels; j++) {
		tid += width * height;
		float err = valid ? error[tid] : 0.0f;

		unsigned int stride = scale;

		while (stride < (scale << j) && stride < blockDim.x * scale) {
#ifdef PRINT_CUDA
			if (!(tid % (width * height)))
				printf("reduce_error stride=%i, j=%i, scale=%i, err=%g\n", stride, j, scale, err);
#endif
			unsigned int stride2 = stride << 1;
			if (valid && !(idX % stride2) && !(idY % stride2)) {
				if (idX + stride < width)
					err += error[tid + stride];
				if (idY + stride < height) {
					err += error[tid + stride * width];
					if (idX + stride < width)
						err += error[tid + stride * (width + 1)];
				}
		
				error[tid] = err;
			}
			stride = stride2;
			__syncthreads();
		}
	}
}

__global__ static
void calc_score(float *error, int *seed, const unsigned int width, const unsigned int height, const unsigned int levels, const unsigned int scale)
{
	unsigned int idX = scale * (blockDim.x * blockIdx.x + threadIdx.x);
	unsigned int idY = scale * (blockDim.y * blockIdx.y + threadIdx.y);
	unsigned int tid = idX + idY * width;
	unsigned int valid = idX < width && idY < height; 

	unsigned int stride = scale << levels;

	for (int i = levels; i--; ) {
		unsigned int stride2 = stride >> 1;

		/* Divide the pool proportinally to error at each quad-tree node. */
		if (valid && !(idX % stride) && !(idY % stride)) {
			unsigned int valid2 = 0u;
			if (idX + stride2 < width)
				valid2 |= VALID_HORIZONTAL;
			if (idY + stride2 < height)
				valid2 |= VALID_VERTICAL;
			unsigned int lid = tid + width * height * i;
			float err[4];
			err[0] = error[lid];
			err[1] = (valid2 & VALID_HORIZONTAL) ? error[lid + stride2] : 0.0f;
			err[2] = (valid2 & VALID_VERTICAL) ? error[lid + stride2 * width] : 0.0f;
			err[3] = (valid2 & (VALID_HORIZONTAL | VALID_VERTICAL)) == (VALID_HORIZONTAL | VALID_VERTICAL) ? error[lid + stride2 * (width + 1)] : 0.0f;
			float errSum = err[0] + err[1] + err[2] + err[3];
			int seedSum = seed[tid];
			float scoreSum = errSum > 0.0f ? seedSum / errSum : 0.0f;

			int s[4];
			s[0] = scoreSum * err[0];
			s[1] = scoreSum * err[1];
			s[2] = scoreSum * err[2];
			s[3] = scoreSum * err[3];
			int diff = seedSum - s[0] - s[1] - s[2] - s[3];
#ifdef PRINT_CUDA
			if (!tid)
				printf("calc_score stride=%i, i=%i, tid=%i, lid=%i, scale=%i, errSum=%g, seedSum=%i, scoreSum=%g, diff=%i\n", stride, i, tid, lid, scale, errSum, seedSum, scoreSum, diff);
#endif
			if (diff && errSum > 0.0f) {
				float max[3] = { 0.0f, 0.0f, 0.0f }; // Will store up to 3 maximum values in err[]
				int maxi[3] = { -1, -1, -1 }; // Will store the indices of up to 3 maximum values in err[]
				for (int j = 0; j < 4; j++) { // Find 3 largest values
					if (err[j] > max[0]) {
						max[2] = max[1]; maxi[2] = maxi[1];
						max[1] = max[0]; maxi[1] = maxi[0];
						max[0] = err[j]; maxi[0] = j;
					}
					else if (err[j] > max[1]) {
						max[2] = max[1]; maxi[2] = maxi[1];
						max[1] = err[j]; maxi[1] = j;
					}
					else if (err[j] > max[2]) {
						max[2] = err[j]; maxi[2] = j;
					}
				}
				if (diff > 2 && max[2] > 0.0f) {
					s[maxi[2]] += 1;
					diff -= 1;
				}
				if (diff > 1 && max[1] > 0.0f) {
					s[maxi[1]] += 1;
					diff -= 1;
				}
				if (diff && max[0] > 0.0f) {
					s[maxi[0]] += diff;
				}
			}

			seed[tid] = s[0];
			if (valid2 & VALID_HORIZONTAL)
				seed[tid + stride2] = s[1];
			if (valid2 & VALID_VERTICAL) {
				seed[tid + stride2 * width] = s[2];
				if (valid2 & VALID_HORIZONTAL)
					seed[tid + stride2 * (width + 1)] = s[3];
			}
		}

		__syncthreads();

		stride = stride2;
	}
}

/* Calculate average of hits at each quad tree node */
static void CCALL cuda_mip_map_hits_recursive(PointDirection *deviceHits, PointDirection *deviceMipMap,
	const unsigned int width, const unsigned int height, const unsigned int levels, const unsigned int maxThreadsPerBlock, dim3 dimGrid, dim3 dimBlock, size_t blockSharedMemorySize)
{
	/* Calculate average of hits at each quad tree node */
	mip_map_hits <<< dimGrid, dimBlock, blockSharedMemorySize >>>
			(deviceHits, deviceMipMap, width, height);

	hipDeviceSynchronize(); checkLastCudaError();

	if ( dimBlock.x < (1u << levels) ) {
		unsigned int complete = 1u;
		unsigned int offset = 0u;
		unsigned int levelWidth = (width - 1) / 2 + 1;
		unsigned int levelHeight = (height - 1) / 2 + 1;
		for (unsigned int i = 1u; i < dimBlock.x / 2u; i <<= 1) {
			complete++;
			offset += levelWidth * levelHeight;
			levelWidth = (levelWidth - 1) / 2 + 1;
			levelHeight = (levelHeight - 1) / 2 + 1;
		}

		const unsigned int blockDim = calc_block_dim(maxThreadsPerBlock, levels - complete);
		const unsigned int blocksX = (levelWidth - 1) / blockDim + 1;
		const unsigned int blocksY = (levelHeight - 1) / blockDim + 1;
		const dim3 dimSuperGrid(blocksX, blocksY);
		const dim3 dimSuperBlock(blockDim, blockDim);

#ifdef PRINT_CUDA
		fprintf(stderr, "cuda_mip_map_hits_recursive: offset %i, width %i, height %i, levels %i\n", offset, levelWidth, levelHeight, levels - complete);
#endif

		cuda_mip_map_hits_recursive(deviceMipMap + offset, deviceMipMap + offset + levelWidth * levelHeight,
			levelWidth, levelHeight, levels - complete, maxThreadsPerBlock, dimSuperGrid, dimSuperBlock, dimSuperBlock.x * dimSuperBlock.y * sizeof(PointDirection));
	}
}

/* Calculate average geometric variation for each quad tree node */
static void CCALL cuda_score_hits_recursive(float *deviceError, int *deviceSeeds,
	const unsigned int width, const unsigned int height, unsigned int levels, const unsigned int scale, const unsigned int maxThreadsPerBlock, dim3 dimGrid, dim3 dimBlock)
{
	/* Perform reduction on error */
	reduce_error <<< dimGrid, dimBlock, 0 >>>
			(deviceError, width, height, levels, scale);

	hipDeviceSynchronize(); checkLastCudaError();

	/* Recruse if block not large enough for reduction */
	if ( dimBlock.x < (1u << levels) ) {
		unsigned int complete = 0u;
		for (unsigned int i = 1u; i < dimBlock.x; i <<= 1)
			complete++;

		const unsigned int blockDim = calc_block_dim(maxThreadsPerBlock, levels - complete);
		const unsigned int blocksX = (dimGrid.x - 1) / blockDim + 1;
		const unsigned int blocksY = (dimGrid.y - 1) / blockDim + 1;
		const dim3 dimSuperGrid(blocksX, blocksY);
		const dim3 dimSuperBlock(blockDim, blockDim);

		cuda_score_hits_recursive(deviceError + width * height * complete, deviceSeeds, width, height, levels - complete, scale * dimBlock.x, maxThreadsPerBlock, dimSuperGrid, dimSuperBlock);
		levels = complete;
	}

	/* Calculate score for each leaf node based on error */
	calc_score <<< dimGrid, dimBlock, 0 >>>
			(deviceError, deviceSeeds, width, height, levels, scale);

	hipDeviceSynchronize(); checkLastCudaError();
}
#endif /* MULTI_BLOCK */

/* Score the relative need for an irradiance cache entry at each hit point */
void CCALL cuda_score_hits(PointDirection *hits, int *seeds, const unsigned int width, const unsigned int height, const float weight, const unsigned int seed_count)
{
	PointDirection *deviceHits;
#ifdef MULTI_BLOCK
	PointDirection *deviceMipMap;
	float *deviceError;
#endif
	int *deviceSeeds;
	
	/* Calculate number of levels */
	unsigned int levels = 0;
	unsigned int size = width > height ? width : height;
	if ( !isPowerOfTwo(size) )
		levels++;
	while ( size >>= 1 )
		levels++;

	/* Determine block size */
	hipDeviceProp_t deviceProp;
	int deviceNum;
	hipGetDevice(&deviceNum);
	hipGetDeviceProperties(&deviceProp, deviceNum);

#ifdef CAP_REGISTERS_PER_THREAD
	const unsigned int registersPerBlock = deviceProp.regsPerBlock;
	unsigned int threadsPerBlock = deviceProp.maxThreadsPerBlock;
	while (registersPerBlock / threadsPerBlock < REGISTERS_PER_THREAD)
		threadsPerBlock >>= 1;
#else
	const unsigned int threadsPerBlock = deviceProp.maxThreadsPerBlock;
#endif

	/* To support reduction, blockDim *must* be a power of two. */
	const unsigned int blockDim = calc_block_dim(threadsPerBlock, levels);
	const unsigned int blocksX = (width - 1) / blockDim + 1;
	const unsigned int blocksY = (height - 1) / blockDim + 1;
	const size_t blockSharedMemorySize = blockDim * blockDim * sizeof(PointDirection);

#ifndef MULTI_BLOCK
	if (blocksX != 1u || blocksY != 1u)
		err("Your CUDA hardware has insufficient block size %u threads (%u x %u blocks needed). Recompile with MULTI_BLOCK flag.", deviceProp.maxThreadsPerBlock, blocksX, blocksY);
#endif
	if (blockSharedMemorySize > deviceProp.sharedMemPerBlock)
		err("Your CUDA hardware has insufficient block shared memory %" PRIu64 " (%" PRIu64 " needed).", deviceProp.sharedMemPerBlock, blockSharedMemorySize);

	const dim3 dimGrid(blocksX, blocksY);
	const dim3 dimBlock(blockDim, blockDim);
#ifdef PRINT_CUDA
	fprintf(stderr, "Adaptive sampling: Block %i x %i, Grid %i x %i, Shared %i, Levels %i, Weight %g\n", blockDim, blockDim, blocksX, blocksY, blockSharedMemorySize, levels, weight);
#endif

	/* Allocate memory and copy hits to the GPU */
	size = width * height;
	checkCuda(hipMalloc(&deviceHits, size * sizeof(PointDirection)));
	checkCuda(hipMemcpy(deviceHits, hits, size * sizeof(PointDirection), hipMemcpyHostToDevice));

#ifdef MULTI_BLOCK
	/* Allocate memory on the GPU */
	unsigned int mipMapSize = 0u;
	unsigned int levelWidth = width;
	unsigned int levelHeight = height;
	while (levelWidth > 1u || levelHeight > 1u) {
		levelWidth = (levelWidth - 1) / 2 + 1;
		levelHeight = (levelHeight - 1) / 2 + 1;
		mipMapSize += levelWidth * levelHeight;
	}
	checkCuda(hipMalloc(&deviceMipMap, mipMapSize * sizeof(PointDirection))); // Storage requirement for mip map is 1/3 or original data
	checkCuda(hipMalloc(&deviceError, size * levels * sizeof(float)));

	/* Calculate average of hits at each quad tree node */
	cuda_mip_map_hits_recursive(deviceHits, deviceMipMap, width, height, levels, threadsPerBlock, dimGrid, dimBlock, blockSharedMemorySize);

	/* Calculate geometric variation at each quad tree node */
	calc_error <<< dimGrid, dimBlock, 0 >>>
			(deviceHits, deviceMipMap, deviceError, width, height, levels, weight);

	hipDeviceSynchronize(); checkLastCudaError();

	/* Free memory on the GPU */
	checkCuda(hipFree(deviceHits));
	checkCuda(hipFree(deviceMipMap));
#endif /* MULTI_BLOCK */

	/* Allocate memory and copy first seed to the GPU */
	seeds[0] = seed_count;
#ifdef PRINT_CUDA
	fprintf(stderr, "Target total score: %i\n", seed_count);
#endif
	checkCuda(hipMalloc(&deviceSeeds, size * sizeof(int)));
	checkCuda(hipMemcpy(deviceSeeds, seeds, sizeof(int), hipMemcpyHostToDevice)); // transfer only first entry

#ifdef MULTI_BLOCK
	/* Calculate average geometric variation for each quad tree node */
	cuda_score_hits_recursive(deviceError, deviceSeeds, width, height, levels, 1u, threadsPerBlock, dimGrid, dimBlock);

	/* Free memory on the GPU */
	checkCuda(hipFree(deviceError));
#else /* MULTI_BLOCK */
	/* Run kernel */
	geometric_variation <<< dimGrid, dimBlock, blockSharedMemorySize >>>
			(deviceHits, deviceSeeds, width, height, levels, weight);
	
	hipDeviceSynchronize(); checkLastCudaError();

	/* Free memory on the GPU */
	checkCuda(hipFree(deviceHits));
#endif /* MULTI_BLOCK */

	/* Copy results from GPU and free memory */
	checkCuda(hipMemcpy(seeds, deviceSeeds, size * sizeof(int), hipMemcpyDeviceToHost));
	checkCuda(hipFree(deviceSeeds));
}

static void printDevProp(const hipDeviceProp_t *devProp)
{
	fprintf(stderr, "Revision number:                    %d.%d\n", devProp->major, devProp->minor);
	fprintf(stderr, "Name:                               %s\n", devProp->name);
	fprintf(stderr, "Total global memory:                %" PRIu64 " bytes\n", devProp->totalGlobalMem);
	fprintf(stderr, "Total constant memory:              %" PRIu64 " bytes\n", devProp->totalConstMem);
	fprintf(stderr, "L2 cache size:                      %u bytes\n", devProp->l2CacheSize);
	fprintf(stderr, "Maximum threads per block:          %d\n", devProp->maxThreadsPerBlock);
	fprintf(stderr, "Shared memory per block:            %" PRIu64 " bytes\n", devProp->sharedMemPerBlock);
	fprintf(stderr, "Registers per block:                %d\n", devProp->regsPerBlock);
	fprintf(stderr, "Maximum threads per multiprocessor: %d\n", devProp->maxThreadsPerMultiProcessor);
	fprintf(stderr, "Shared mem per multiprocessor:      %" PRIu64 " bytes\n", devProp->sharedMemPerMultiprocessor);
	fprintf(stderr, "Registers per multiprocessor:       %d\n", devProp->regsPerMultiprocessor);
	fprintf(stderr, "Warp size:                          %d\n", devProp->warpSize);
	fprintf(stderr, "Maximum memory pitch:               %" PRIu64 " bytes\n", devProp->memPitch);
	for (int i = 0; i < 3; ++i)
		fprintf(stderr, "Maximum dimension %d of block:       %d\n", i, devProp->maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		fprintf(stderr, "Maximum dimension %d of grid:        %d\n", i, devProp->maxGridSize[i]);
	fprintf(stderr, "Global memory bus width:            %d bits\n", devProp->memoryBusWidth);
	fprintf(stderr, "Peak memory clock frequency:        %d kHz\n", devProp->memoryClockRate);
	fprintf(stderr, "Clock rate:                         %d kHz\n", devProp->clockRate);
	fprintf(stderr, "Texture alignment:                  %" PRIu64 "\n", devProp->textureAlignment);
	fprintf(stderr, "Texture pitch alignment:            %" PRIu64 "\n", devProp->texturePitchAlignment);
	fprintf(stderr, "Concurrent kernels:                 %s\n", devProp->concurrentKernels ? "Yes" : "No");
	fprintf(stderr, "Concurrent copy and execution:      %s\n", devProp->deviceOverlap ? "Yes" : "No");
	fprintf(stderr, "Number of async engines:            %d\n", devProp->asyncEngineCount);
	fprintf(stderr, "Number of multiprocessors:          %d\n", devProp->multiProcessorCount);
	fprintf(stderr, "Kernel execution timeout:           %s\n", devProp->kernelExecTimeoutEnabled ? "Yes" : "No");
	fprintf(stderr, "Unified addressing with host:       %s\n", devProp->unifiedAddressing ? "Yes" : "No");
	fprintf(stderr, "Device can map host memory:         %s\n", devProp->canMapHostMemory ? "Yes" : "No");
	fprintf(stderr, "Device supports managed memory:     %s\n", devProp->managedMemory ? "Yes" : "No");
	return;
}
 
void printCUDAProp()
{
	// Number of CUDA devices
	int devCount;
	hipGetDeviceCount(&devCount);
	fprintf(stderr, "CUDA Device Query...\n");
	fprintf(stderr, "There are %d CUDA devices.\n", devCount);
 
	// Iterate through devices
	for (int i = 0; i < devCount; ++i)
	{
		// Get device properties
		fprintf(stderr, "\nCUDA Device #%d\n", i);
		hipDeviceProp_t devProp;
		hipGetDeviceProperties(&devProp, i);
		printDevProp(&devProp);
	}
}

#ifdef __cplusplus
}
#endif
