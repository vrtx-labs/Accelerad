#include "hip/hip_runtime.h"
/*
 *  point_cloud_normal.cu - hit programs for geometry sampling on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_ray.h"
#include "optix_point_common.h"

using namespace optix;

/* Context variables */
rtBuffer<PointDirection, 3>      seed_buffer; /* output */

rtDeclareVariable(float, specthresh, , );	/* This is the minimum fraction of reflection or transmission, under which no specular sampling is performed */
#ifdef AMBIENT_CELL
rtDeclareVariable(float, specjitter, , );	/* specular sampling (ss) */

rtDeclareVariable(float3, cuorg, , ); /* bounding box minimum */
rtDeclareVariable(float, cell_size, , ); /* cell side dimension */
rtDeclareVariable(unsigned int, level, , ) = 0u;

rtDeclareVariable(rtObject, top_ambient, , );


RT_METHOD uint2 cell_hash(const float3& pos, const float3& dir)
{
	uint2 cell;
	float3 absdir = make_float3(fabsf(dir.x), fabsf(dir.y), fabsf(dir.z));
	if (absdir.x > absdir.y) {
		if (absdir.x > absdir.z)
			cell.x = dir.x > 0 ? 0 : 0x10000;
		else
			cell.x = dir.z > 0 ? 0x40000 : 0x50000;
	}
	else {
		if (absdir.y > absdir.z)
			cell.x = dir.y > 0 ? 0x20000 : 0x30000;
		else
			cell.x = dir.z > 0 ? 0x40000 : 0x50000;
	}
	float3 cell_index = (pos - cuorg) / cell_size;
	cell.x += ((unsigned int)cell_index.x) & 0xffff;
	cell.y = (((unsigned int)cell_index.y) << 16) + (((unsigned int)cell_index.z) & 0xffff);
	return cell;
}

RT_METHOD int occupied(const float3& pos, const float3& dir, const float3& world)
{
	float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	PerRayData_ambient ambient_prd;
	ambient_prd.result = make_float3(0.0f);
	ambient_prd.surface_point = pos;
	ambient_prd.surface_normal = faceforward(world_shading_normal, -ray.direction, world);
	ambient_prd.ambient_depth = level;
	ambient_prd.wsum = 0.0f;
	ambient_prd.weight = 1.0f;
	for ( int i = level; i--; )
		ambient_prd.weight *= AVGREFL; // Compute weight as in makeambient() from ambient.c

#ifdef DAYSIM_COMPATIBLE
	ambient_prd.dc = make_uint3(0u); // Mark as null (TODO check this)
#endif
#ifdef HIT_COUNT
	ambient_prd.hit_count = 0;
#endif
	const float tmax = ray_start(pos, AMBIENT_RAY_LENGTH);
	Ray ambient_ray = make_Ray(pos - dir * tmax, dir, AMBIENT_RAY, 0.0f, 2.0f * tmax);
	rtTrace(top_ambient, ambient_ray, ambient_prd, RT_VISIBILITY_ALL, RT_RAY_FLAG_DISABLE_CLOSESTHIT);
#ifdef HIT_COUNT
	prd.hit_count += ambient_prd.hit_count;
#endif
	return ambient_prd.wsum > FTINY;
}
#else /* AMBIENT_CELL */
RT_METHOD float3 sample_hemisphere(const float3& uz, rand_state* state)
{
	const float3 ux = getperpendicular(uz);
	const float3 uy = normalize(cross(uz, ux));
	float zd = sqrtf(hiprand_uniform(state));
	const float phi = 2.0f*M_PIf * hiprand_uniform(state);
	const float xd = cosf(phi) * zd;
	const float yd = sinf(phi) * zd;
	zd = sqrtf(1.0f - zd*zd);
	return normalize(xd*ux + yd*uy + zd*uz);
}
#endif /* AMBIENT_CELL */

RT_CALLABLE_PROGRAM PerRayData_point_cloud closest_hit_glass_point_cloud(IntersectData const&data, PerRayData_point_cloud prd)
{
	float3 snormal = faceforward(data.world_geometric_normal, -data.ray_direction, data.world_geometric_normal);

	prd.point = data.hit;

	/* Transmission */
#ifdef AMBIENT_CELL
	prd.forward = data.ray_direction;
#else
	prd.forward = sample_hemisphere(-snormal, prd.state);
#endif

	/* Reflection */
#ifdef AMBIENT_CELL
	prd.reverse = reflect(data.ray_direction, snormal);
#else
	prd.reverse = sample_hemisphere(snormal, prd.state);
#endif

	return prd;
}

RT_CALLABLE_PROGRAM PerRayData_point_cloud closest_hit_normal_point_cloud(IntersectData const&data, PerRayData_point_cloud prd)
{
	float3 snormal = faceforward(data.world_geometric_normal, -data.ray_direction, data.world_geometric_normal);

	float trans = data.mat.params.n.trans * (1.0f - data.mat.params.n.spec);
	float tspec = trans * data.mat.params.n.tspec;
	float alpha2 = data.mat.params.n.rough * data.mat.params.n.rough;

	/* Record new origin */
	prd.point = data.hit;

	/* Transmitted ambient */
	if (data.mat.params.n.ambincl && trans - tspec > FTINY && prd.index.z < prd.seeds) {
#ifdef AMBIENT_CELL
		if (!occupied(prd.point, -snormal, data.world_geometric_normal)) {
			seed_buffer[prd.index].cell = cell_hash(prd.point, -snormal);
#endif

			/* Store seed point */
			seed_buffer[prd.index].pos = prd.point;
			seed_buffer[prd.index].dir = -snormal;
			prd.index.z++;

#ifdef AMBIENT_CELL
		}
#endif
	}

	/* Reflected ambient */
	if (data.mat.params.n.ambincl && 1.0f - trans - data.mat.params.n.spec > FTINY && prd.index.z < prd.seeds) {
#ifdef AMBIENT_CELL
		if (!occupied(prd.point, snormal, data.world_geometric_normal)) {
			seed_buffer[prd.index].cell = cell_hash(prd.point, snormal);
#endif

			/* Store seed point */
			seed_buffer[prd.index].pos = prd.point;
			seed_buffer[prd.index].dir = snormal;
			prd.index.z++;

#ifdef AMBIENT_CELL
		}
#endif
	}

	if (prd.index.z >= prd.seeds) return prd;

	/* Transmitted ray */
	if (tspec > FTINY && (alpha2 <= FTINY || specthresh < tspec - FTINY)) {
#ifdef AMBIENT_CELL
		prd.forward = ray.direction;

		if (alpha2 > FTINY) {
			float3 u = getperpendicular(-snormal); //TODO should be pnormal
			float3 v = cross(-snormal, u);
			float2 rv = make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state)); // should be evenly distributed in both dimensions
			float d = 2.0f * M_PIf * rv.x;
			float cosp = cosf(d);
			float sinp = sinf(d);
			if ((0.0f <= specjitter) && (specjitter < 1.0f))
				rv.y = 1.0f - specjitter * rv.y;
			if (rv.y <= FTINY)
				d = 1.0f;
			else
				d = sqrtf(alpha2 * -logf(rv.y));
			float3 h = d * (cosp * u + sinp * v) - snormal; //TODO should be pnormal
			d = -2.0f * dot(h, prd.forward) / (1.0f + d*d);
			h = prd.forward + h * d;

			/* sample rejection test */
			if (dot(h, snormal) < -FTINY)
				prd.forward = h;
		}
#else
		prd.forward = sample_hemisphere(-snormal, prd.state);
#endif
	}

	/* Reflected ray */
	if (data.mat.params.n.spec > FTINY && (alpha2 <= FTINY || specthresh < data.mat.params.n.spec - FTINY)) {
#ifdef AMBIENT_CELL
		prd.reverse = reflect(ray.direction, snormal);

		if (alpha2 > FTINY) {
			float3 u = getperpendicular(snormal); //TODO should be pnormal
			float3 v = cross(snormal, u);
			float2 rv = make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state)); // should be evenly distributed in both dimensions
			float d = 2.0f * M_PIf * rv.x;
			float cosp = cosf(d);
			float sinp = sinf(d);
			if ((0.0f <= specjitter) && (specjitter < 1.0f))
				rv.y = 1.0f - specjitter * rv.y;
			if (rv.y <= FTINY)
				d = 1.0f;
			else
				d = sqrtf(alpha2 * -logf(rv.y));
			float3 h = d * (cosp * u + sinp * v) + snormal; //TODO should be pnormal
			d = -2.0f * dot(h, prd.reverse) / (1.0f + d*d);
			h = prd.reverse + h * d;

			/* sample rejection test */
			if (dot(h, snormal) > FTINY)
				prd.reverse = h;
		}
#else
		prd.reverse = sample_hemisphere(snormal, prd.state);
#endif
	}

	return prd;
}
