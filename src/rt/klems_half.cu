#include "hip/hip_runtime.h"
/*
 *  klems_half.cu - program for Klems bin identification on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

rtDeclareVariable(float3, normal, , );	/* Normal direction */
rtDeclareVariable(float3, up, , );		/* Up direction */
rtDeclareVariable(int, RHS, , ) = 1;	/* Coordinate system handedness: set to -1 for left-handed system */

// Calculate the Klems bin based on klems_full.cal.
RT_CALLABLE_PROGRAM int kbin(const float3 direction)
{
	const float DdotN = dot(direction, normal);
	const float DdotU = dot(direction, up);
	const float NdotU = dot(normal, up);

	if (DdotN > 0) return -1; // Wrong-side ray, probably bad
	if (DdotN <= -1) return 0;
	float pol = acosf(-DdotN) * 180 * M_1_PIf;
	float azi = atan2f(-DdotU + DdotN * NdotU, -RHS * dot(direction, cross(up, normal))) * 180 * M_1_PIf;
	if (azi < 0.0f) azi += 360.0f;

	const float kpola[] = { 6.5f, 19.5f, 32.5f, 46.5f, 61.5f, 76.5f, 90.0f };
	const int knaz[] = { 1, 8, 12, 16, 20, 12, 4 };	// Number of patches per row

	int row = 0;
	while (pol > kpola[row]) // This is kfindrow
		row++;

	float inc = 360.0f / knaz[row];
	int kaccum = ((360.0f - 0.5f * inc) > azi) ? (int)floor((azi + 0.5f * inc) / inc) : 0; // This is kazn

	for (int r = 0; r < row; r++)
		kaccum += knaz[r];

	return kaccum;
}
