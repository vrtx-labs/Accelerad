#include "hip/hip_runtime.h"
/*
 *  perezlum.cu - program for Perez All-Weather Sky distribution on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>

struct Perez_coef
{
	float a, b, c, d, e;
};

struct Transform
{
	optix::Matrix<3,3> m;
};

/* Program variables */
rtDeclareVariable(float,      diffuse, , ); /* diffuse normalization */
rtDeclareVariable(float,      ground, , ); /* ground plane brightness */
rtDeclareVariable(Perez_coef, coef, , ); /* coefficients for the Perez model */
rtDeclareVariable(float3,     sun, , ); /* sun direction */
rtDeclareVariable(Transform,  transform, , ); /* transformation matrix */

// Calculate the All-weather Angular Sky Luminance Distribution value for the current ray direction.
// This function replicates the algorithm in perezlum.cal.
RT_CALLABLE_PROGRAM float3 skybright(const float3 direction, const float3 ignore)
{
	//rtPrintf("PerezLum Recieved (%f, %f, %f)\n", direction.x, direction.y, direction.z);

	const float3 dir = transform.m * direction;

	const float cosgamma = 0.999999f * optix::dot(dir, sun); // cosgamma = Dx*A8 + Dy*A9 + Dz*A10; // Adjusted to keep in range [-1,1]
	const float gamma = acosf(cosgamma); // gamma = Acos(cosgamma);		{ angle from sun to this point in sky }
	//float zt = acos(sun.z); // zt = Acos(A10);			{ angle from zenith to sun }
	//float eta = acos(dir.z); // eta = Acos(Dz);			{ angle from zenith to this point in sky }

	float dz = dir.z;
	if (dz < 0.01f) {
		dz = 0.01f;
	}

	const float intersky = diffuse * (1.0f + coef.a * expf( coef.b / dz ) ) * ( 1.0f + coef.c * expf(coef.d * gamma) + coef.e * cosgamma * cosgamma );
	//intersky = if( (Dz-0.01),  
	//		A1 * (1 + A3*Exp(A4/Dz) ) * ( 1 + A5*Exp(A6*gamma) + A7*cos(gamma)*cos(gamma) ),
	//		A1 * (1 + A3*Exp(A4/0.01) ) * ( 1 + A5*Exp(A6*gamma) + A7*cos(gamma)*cos(gamma) ) );

	const float a = powf(dir.z + 1.01f, 10.0f);
	const float b = powf(dir.z + 1.01f, -10.0f);

	const float skybright = (a * intersky + b * ground) / (a + b); // wmean(a, x, b, y) = (a*x+b*y)/(a+b);
	return make_float3(skybright);
}