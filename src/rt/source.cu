#include "hip/hip_runtime.h"
/*
 *  source.cu - program for source distribution on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_common.h"

using namespace optix;

struct Transform
{
	optix::Matrix<3,3> m;
};

/* Program variables */
rtDeclareVariable(int, data, , ); /* texture ID */
rtDeclareVariable(int, type, , ); /* type of data (true for float) */
rtDeclareVariable(float3, org, , ); /* texture minimum coordinates */
rtDeclareVariable(float3, siz, , ); /* texture coordinates extent */
rtDeclareVariable(Transform, transform, , ); /* transformation matrix */
rtDeclareVariable(int, transpose, , ) = 0; /* flag to transpose texture to swap phi and theta */
rtDeclareVariable(float, symmetry, , ) = 0.0f; /* radial symmetry angle in radians */
rtDeclareVariable(float, multiplier, , ) = 1.0f; /* multiplier for light source intensity */
rtDeclareVariable(float3, bounds, , ); /* dimensions of axis-aligned box or Z-aligned cylinder in meters */

// Calculate source distribution.
RT_METHOD float3 source(const float3& dir)
{
	float theta = acosf(dir.z);
	float phi = atan2f(-dir.y, -dir.x);
	phi += 2.0f * M_PIf * (phi < 0.0f);

	if (symmetry > 0.0f) {
		phi = fabsf(symmetry - fmodf(phi + symmetry, 2 * symmetry));
	}

	if (transpose) {
		float temp = phi;
		phi = theta;
		theta = temp;
	}

	/* Normalize to [0, 1] within range */
	theta = (180.0f * M_1_PIf * theta - org.x) / siz.x;
	phi = (180.0f * M_1_PIf * phi - org.y) / siz.y;

	/* Renormalize to remove edges */
	uint3 ne = rtTexSize(data);
	theta = (theta * (ne.x - 1) + 0.5f) / ne.x;
	phi = (phi * (ne.y - 1) + 0.5f) / ne.y;

	if (type)
		return make_float3(multiplier * rtTex2D<float>(data, theta, phi)); // this is corr from source.cal
	float4 tex = rtTex2D<float4>(data, theta, phi);
	return multiplier * make_float3(tex.y, tex.z, tex.w);
}

// Calculate source distribution.
RT_CALLABLE_PROGRAM float3 corr(const float3 direction, const float3 ignore)
{
	const float3 dir = transform.m * direction;
	return source(dir); // this is corr from source.cal
}

// Calculate source distribution with correction for flat sources.
RT_CALLABLE_PROGRAM float3 flatcorr(const float3 direction, const float3 normal)
{
	const float3 dir = transform.m * direction;
	const float rdot = dot(direction, normal);
	return source(dir) / fabsf(rdot); // this is flatcorr from source.cal
}

// Calculate source distribution with correction for emitting boxes.
RT_CALLABLE_PROGRAM float3 boxcorr(const float3 direction, const float3 ignore)
{
	const float3 dir = transform.m * direction;
	const float boxprojection = fabsf(dir.x) * bounds.y * bounds.z + fabsf(dir.y) * bounds.x * bounds.z + fabsf(dir.z) * bounds.x * bounds.y;
	return source(dir) / boxprojection; // this is boxcorr from source.cal
}

// Calculate source distribution with correction for emitting cylinders.
RT_CALLABLE_PROGRAM float3 cylcorr(const float3 direction, const float3 ignore)
{
	const float3 dir = transform.m * direction;
	const float cylprojection = bounds.x * bounds.y * sqrtf(fmaxf(1.0f - dir.z * dir.z, 0.0f)) + M_PIf / 4.0f * bounds.x * bounds.x * fabsf(dir.z);
	return source(dir) / cylprojection; // this is cylcorr from source.cal
}