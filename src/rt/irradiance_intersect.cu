#include "hip/hip_runtime.h"
/*
 *  irradiance_intersect.cu - intersection program for virtual Lambertian surface on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "optix_shader_common.h"

using namespace optix;

/* OptiX variables */
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

/* Attributes */
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, surface_id, attribute surface_id, );
rtDeclareVariable(int, mat_id, attribute mat_id, );

RT_PROGRAM void irradiance_intersect( int primIdx )
{
	/* There should always be an intersection at t=0. */
	if ( rtPotentialIntersection( 0.0f ) ) {
		geometric_normal = shading_normal = normalize( -ray.direction );
		texcoord = make_float3( 0.0f, 0.0f, 0.0f );
		surface_id = -1; // Not a real surface

		/* Lambert material is material 0 */
		mat_id = 0;
		rtReportIntersection(0);
	}
}

RT_PROGRAM void irradiance_bounds(int primIdx, float result[6])
{  
	optix::Aabb* aabb = (optix::Aabb*)result;

	/* The single instance covers the entire scene. */
	aabb->m_min = make_float3( -RT_DEFAULT_MAX );
	aabb->m_max = make_float3( RT_DEFAULT_MAX );
}

