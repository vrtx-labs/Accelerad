#include "hip/hip_runtime.h"
/*
 *  rpict.cu - entry point for image generation on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_ray.h"
#ifdef CONTRIB_DOUBLE
#include "optix_double.h"
#endif

using namespace optix;

/* Contex variables */
rtDeclareVariable(unsigned int,  frame, , ); /* Current frame number, starting from zero */
rtDeclareVariable(unsigned int,  camera, , ); /* Camera type (-vt) */
rtDeclareVariable(float3,        eye, , ); /* Eye position (-vp) */
rtDeclareVariable(float3,        U, , ); /* view.hvec */
rtDeclareVariable(float3,        V, , ); /* view.vvec */
rtDeclareVariable(float3,        W, , ); /* view.vdir */
rtDeclareVariable(float2,        fov, , ); /* Field of view (-vh, -vv) */
rtDeclareVariable(float2,        shift, , ); /* Camera shift (-vs, -vl) */
rtDeclareVariable(float2,        clip, , ); /* Fore and aft clipping planes (-vo, -va) */
rtDeclareVariable(float,         vdist, , ); /* Focal length */
rtDeclareVariable(float,         dstrpix, , ) = 0.0f; /* Pixel sample jitter (-pj) */
rtDeclareVariable(float,         mblur, , ) = 0.0f; /* Motion blur (-pm) */
rtDeclareVariable(float,         dblur, , ) = 0.0f; /* Depth-of-field blur (-pd) */

rtBuffer<float4, 2>              output_buffer;
#ifdef RAY_COUNT
rtBuffer<unsigned int, 2>        ray_count_buffer;
#endif
rtBuffer<RayParams, 2>           last_view_buffer;
//rtBuffer<unsigned int, 2>        rnd_seeds;
rtDeclareVariable(rtObject,      top_object, , );

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

//#define TIME_VIEW


// Pick the ray direction based on camera type as in image.c.
RT_PROGRAM void ray_generator()
{
#ifdef TIME_VIEW
	clock_t t0 = clock();
	output_buffer[launch_index] = make_float4( t0 );
#endif
	PerRayData_radiance prd;
	prd.result = prd.mirror = make_float3(0.0f);
	prd.distance = prd.mirror_distance = RAY_END;
	prd.weight = 1.0f;
	prd.depth = 0;
	prd.ambient_depth = 0;
	//prd.seed = rnd_seeds[launch_index];
#ifdef CONTRIB
	prd.rcoef = make_contrib3(1.0f); //Probably not necessary
#endif
#ifdef ANTIMATTER
	prd.mask = 0u;
	prd.inside = 0;
#endif
	setupPayload(prd);

	init_rand(&prd.state, launch_index.x + launch_dim.x * launch_index.y);

	float2 d = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
	d = 0.5f + dstrpix * ( 0.5f - d ); // this is pixjitter() from rpict.c
	d = shift + ( make_float2( launch_index ) + d ) / make_float2( launch_dim ) - 0.5f;
	float3 ray_origin = eye;
	float z = 1.0f;

	// This is adapted from viewray() in image.c.
  	if( camera == VT_PAR ) { /* parallel view */
		ray_origin += d.x*U + d.y*V;
		d = make_float2( 0.0f );
	} else if ( camera == VT_HEM ) { /* hemispherical fisheye */
		z = 1.0f - d.x*d.x * dot( U, U ) - d.y*d.y * dot( V, V );
		if (z < 0.0f)
			goto done;
		z = sqrtf(z);
	} else if ( camera == VT_CYL ) { /* cylindrical panorama */
		float dd = d.x * fov.x * ( M_PIf / 180.0f );
		z = cosf( dd );
		d.x = sinf( dd );
	} else if ( camera == VT_ANG ) { /* angular fisheye */
		d *= fov / 180.0f;
		float dd = length(d);
		if (dd > 1.0f)
			goto done;
		z = cosf( M_PIf * dd );
		d *= dd < FTINY ? M_PIf : sqrtf(1.0f - z*z) / dd;
	} else if ( camera == VT_PLS ) { /* planispheric fisheye */
		d *= make_float2(length(U), length(V));
		float dd = dot( d, d );
		z = ( 1.0f - dd ) / ( 1.0f + dd );
		d *= 1.0f + z;
	}

	do { // do-while for variable scoping
		float3 ray_direction = d.x*U + d.y*V + z*W;
		ray_origin += clip.x * ray_direction;
		ray_direction = normalize(ray_direction);

		// Zero or negative aft clipping distance indicates infinity
		prd.tmax = clip.y - clip.x;
		if (prd.tmax <= FTINY) {
			prd.tmax = RAY_END;
		}
		float distance = vdist;

		/* optional motion blur */
		if (mblur > FTINY) {
			RayParams next;
			next.aft = prd.tmax;
			next.origin = ray_origin;
			next.direction = ray_direction;
			next.distance = distance;

			if (frame) {
				RayParams prev = last_view_buffer[launch_index];
				z = mblur * (0.5f - hiprand_uniform(prd.state));

				prd.tmax = lerp(prd.tmax, prev.aft, z);
				ray_origin = lerp(ray_origin, prev.origin, z);
				ray_direction = normalize(lerp(ray_direction, prev.direction, z));
				distance = lerp(distance, prev.distance, z);
			}

			last_view_buffer[launch_index] = next;
		}

		/* optional depth-of-field */
		if (dblur > FTINY) {
			float adj = 1.0f;
			z = 0.0f;

			/* random point on disk */
			SDsquare2disk(d, hiprand_uniform(prd.state), hiprand_uniform(prd.state));
			d *= 0.5f * dblur;
			if ((camera == VT_PER) | (camera == VT_PAR)) {
				if (camera == VT_PER)
					adj /= dot(ray_direction, W);
			}
			else {			/* non-standard view case */
				z = M_PI_4f * dblur * (0.5f - hiprand_uniform(prd.state));
			}
			if ((camera != VT_ANG) & (camera != VT_PLS)) {
				if (camera != VT_CYL)
					d.x /= length(U);
				d.y /= length(V);
			}
			ray_origin += d.x * U + d.y * V + z * W;
			ray_direction = normalize(eye + adj * distance * ray_direction - ray_origin);
		}

		Ray ray = make_Ray(ray_origin, ray_direction, RADIANCE_RAY, 0.0f, prd.tmax);

		rtTrace(top_object, ray, prd);

		checkFinite(prd.result);
	} while (0);

done:
#ifdef TIME_VIEW
	clock_t t1 = clock();
 
	float expected_fps   = 1.0f;
	float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
	output_buffer[launch_index] = make_float4( pixel_time );
#else
	output_buffer[launch_index] = make_float4(prd.result, rayDistance(prd));
#endif
#ifdef RAY_COUNT
	ray_count_buffer[launch_index] = prd.ray_count;
#endif
}

RT_PROGRAM void exception()
{
#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
#ifdef TIME_VIEW
	clock_t t1 = clock();
 
	float expected_fps   = 1.0f;
	float pixel_time     = ( t1 - output_buffer[launch_index].x ) * time_view_scale * expected_fps;
	output_buffer[launch_index] = make_float4( pixel_time );
#else
	output_buffer[launch_index] = exceptionToFloat4(rtGetExceptionCode());
#endif
}
