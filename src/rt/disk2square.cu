#include "hip/hip_runtime.h"
/*
 *  disk2square.cu - program for Shirley-Chiu mapping on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3, normal, , );	/* Normal direction */
rtDeclareVariable(float3, up, , );		/* Up direction */
rtDeclareVariable(int, RHS, , ) = 1;	/* Coordinate system handedness: set to -1 for left-handed system */
rtDeclareVariable(int, SCdim, , ) = 1;	/* Side length of square */

// Calculate the Shirley-Chiu mapping based on disk2square.cal.
RT_CALLABLE_PROGRAM int scbin(const float3 direction)
{
	// Compute oriented axis angles
	const float inc_dz = -dot(direction, normal);
	const float inc_rx = -RHS * dot(direction, cross(up, normal));
	const float inc_ry = -dot(direction, up) - inc_dz * dot(normal, up);

	/* -1 if behind surface */
	if (inc_dz <= 0.0f) return -1;

	const float inc_den2 = inc_rx * inc_rx + inc_ry * inc_ry;
	const float inc_radf = inc_den2 > 1e-7f ? sqrtf((1 - inc_dz*inc_dz) / inc_den2) : 0.0f;

	/* Compute square position from disk coordinates */
	const float2 in_disk = make_float2(inc_rx, inc_ry) * inc_radf;
	const float in_disk_r = length(in_disk);
	float in_disk_phi = atan2f(in_disk.y, in_disk.x);
	if (in_disk_phi < -M_PI_4f)
		in_disk_phi += 2.0f * M_PIf;

	float2 out_square;
	switch ((int)floor((in_disk_phi + M_PI_4f) / M_PI_2f)) {
	case 0:
		out_square = make_float2(in_disk_r, in_disk_phi * in_disk_r / M_PI_4f);
		break;
	case 1:
		out_square = make_float2((M_PI_2f - in_disk_phi) * in_disk_r / M_PI_4f, in_disk_r);
		break;
	case 2:
		out_square = make_float2(-in_disk_r, (M_PIf - in_disk_phi) * in_disk_r / M_PI_4f);
		break;
	case 3:
		out_square = make_float2((in_disk_phi - 3 * M_PI_2f) * in_disk_r / M_PI_4f, -in_disk_r);
		break;
	case 4:
		out_square = make_float2(in_disk_r, -in_disk_r); /* Edge case */
		break;
	default:
		return -1; /* Something's wrong */
	}
	out_square = (out_square + 1.0f) / 2.0f;

	/* Compute final bin */
	return (int)floor(out_square.x * SCdim) * SCdim + (int)floor(out_square.y * SCdim);
}