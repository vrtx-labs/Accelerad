#include "hip/hip_runtime.h"
/*
 *  reinhartb.cu - program for Reinhart sky patch identification on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3, normal, , );	/* Normal direction */
rtDeclareVariable(float3, up, , );		/* Up direction */
rtDeclareVariable(int, RHS, , ) = 1;	/* Coordinate system handedness: set to -1 for left-handed system */
rtDeclareVariable(int, mf, , ) = 1;		/* Number of divisions per Tregenza patch */

// Calculate the Reinhart patch based on reinhartb.cal.
RT_CALLABLE_PROGRAM int rbin(const float3 direction)
{
	// Compute oriented axis angles
	const float inc_dz = -dot(direction, normal);
	const float inc_rx = -RHS * dot(direction, cross(up, normal));
	const float inc_ry = dot(direction, up) + inc_dz * dot(normal, up);

	if (inc_dz <= 0.0f) return -1;
	float alt = (inc_dz >= 1.0f) ? 90.0f : asinf(inc_dz) * 180 * M_1_PIf;
	float azi = atan2f(inc_rx, inc_ry) * 180 * M_1_PIf;
	if (azi < 0.0f) azi += 360.0f;

	const int tnaz[] = { 30, 30, 24, 24, 18, 12, 6 };	// Number of patches per row

	float alpha = 90.0f / (mf * 7 + 0.5f);		// Separation between rows in degrees
	int r_row = (int)floor(alt / alpha);
	int rnaz = (r_row > (7 * mf - 0.5f)) ? 1 : mf * tnaz[(int)floor((r_row + 0.5f) / mf)];
	float r_inc = 360.0f / rnaz;
	int raccum = (359.9999f > 0.5f * r_inc + azi) ? (int)floor((azi + 0.5f * r_inc) / r_inc) : 0; // This is r_azn

	for (int r = 0; r < r_row; r++) {
		raccum += (r > (7 * mf - 0.5f)) ? 1 : mf * tnaz[(int)floor((r + 0.5f) / mf)];
	}

	return raccum;
}