#include "hip/hip_runtime.h"
/*
 *  isotropsky.cu - program for isotropic sky distribution on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>

struct Transform
{
	optix::Matrix<3,3> m;
};

/* Program variables */
rtDeclareVariable(float,      radiance, , ); /* isotropic sky radiance */
rtDeclareVariable(Transform,  transform, , ); /* transformation matrix, ignored */

// Calculate the isotropic sky value for the current ray direction.
// This function replicates the algorithm in isotrop_sky.cal distributed with Daysim.
RT_CALLABLE_PROGRAM float3 skybright(const float3 ignore0, const float3 ignore1)
{
	return make_float3(radiance);
}
