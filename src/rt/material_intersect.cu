#include "hip/hip_runtime.h"
/*
 *  material_intersect.cu - hit programs for the material on GPUs.
 */

#include "accelerad_copyright.h"

#include "otypes.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_ray.h"
#include "optix_point_common.h"
#ifdef CONTRIB
#include "optix_shader_contrib.h"
#endif

using namespace optix;


/* Program variables */
rtDeclareVariable(unsigned int, backvis, , ) = 1u; /* backface visibility (bv) */

/* Context variables */
rtDeclareVariable(unsigned int, do_irrad, , ) = 0u;	/* Calculate irradiance (-i) */
rtDeclareVariable(unsigned int, frame, , ) = 0u;	/* Current frame number, starting from zero, for rvu only */

rtBuffer<MaterialData> material_data;	/* One entry per Radiance material. */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(PerRayData_point_cloud, prd_point_cloud, rtPayload, );

/* Attributes */
//rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, surface_id, attribute surface_id, );
rtDeclareVariable(int, mat_id, attribute mat_id, );

/* Context variables */
rtDeclareVariable(rtObject, top_object, , );


RT_PROGRAM void any_hit()
{
	if (mat_id < 0 || mat_id >= material_data.size()) {
		rtIgnoreIntersection();
	}
	else {
		const MaterialData mat = material_data[mat_id];

		// Backface visibility
		if (mat.type != MAT_CLIP && !backvis && dot(geometric_normal, ray.direction) > 0) {
			rtIgnoreIntersection();
		}

		// Illum material
		if (mat.type == MAT_ILLUM && mat.proxy < 0 && (ray.ray_type != SHADOW_RAY || surface_id != -prd_shadow.target - 1)) {
			rtIgnoreIntersection();
		}
	}
}

RT_PROGRAM void closest_hit_radiance()
{
	IntersectData data;
	data.mat = material_data[mat_id];
	data.ray_type = ray.ray_type;
	data.t = t_hit;
	data.ray_direction = ray.direction;
	data.hit = ray.origin + t_hit * ray.direction;

	data.world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	data.world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	data.surface_id = surface_id;

#ifdef ANTIMATTER
	bool continue_ray = false;
	if (data.mat.type == MAT_CLIP) {
		if (dot(data.world_geometric_normal, ray.direction) < 0.0f) {
			/* Entering a volume */
			prd.mask |= data.mat.mask;
		}
		else if ((prd.mask & data.mat.mask) && prd.inside > 0 && data.mat.proxy > -1) {
			/* Leaving a volume and rendering the alternate material */
			data.mat = material_data[data.mat.proxy]; // TODO this will produce odd results if the proxy material is transparent
		}
		else {
			/* Just leave the volume */
			prd.mask &= ~data.mat.mask;
		}
	}
	else if (prd.mask & data.mat.mask) {
		/* Entering or leaving the material while in antimatter. */
		prd.inside += dot(data.world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;
		continue_ray = true;
	}
#endif /* ANTIMATTER */

	while (data.mat.type == MAT_ILLUM) {
		if (data.mat.proxy < 0) return; // This should be disallowed by any hit program
		data.mat = material_data[data.mat.proxy];
	}
	if (prd.depth == 0 && do_irrad)
		if (data.mat.type == MAT_PLASTIC || data.mat.type == MAT_METAL || data.mat.type == MAT_TRANS) {
			data.mat = material_data[0];
		}
	int pid = data.mat.radiance_program_id;
	if (prd.depth == 0 && frame)
		pid = data.mat.diffuse_program_id;
#ifdef ANTIMATTER
	if (continue_ray)
		pid = RT_PROGRAM_ID_NULL;
#endif /* ANTIMATTER */

	/* Call the material's callable program. */
	if (pid != RT_PROGRAM_ID_NULL) {
		prd.distance = data.t; // in case it isn't set later
		prd = rtMarkedCallableProgramId<PerRayData_radiance(IntersectData const&, PerRayData_radiance)>(pid, "closest_hit_radiance_call_site")(data, prd);
	}
	else {
		Ray new_ray = make_Ray(ray.origin, ray.direction, RADIANCE_RAY, t_hit + ray_start(data.hit, ray.direction, data.world_geometric_normal, RAY_START), prd.tmax);
		rtTrace(top_object, new_ray, prd);
	}

#ifdef HIT_TYPE
	prd.hit_type = data.mat.type;
#endif
#ifdef HIT_COUNT
	prd.hit_count++;
#endif
#ifdef CONTRIB
	contribution(prd.rcoef, prd.result, ray.direction, data.mat.contrib_index, data.mat.contrib_function);
#endif
}

RT_PROGRAM void closest_hit_shadow()
{
	IntersectData data;
	data.mat = material_data[mat_id];
	data.ray_type = ray.ray_type;
	data.t = t_hit;
	data.ray_direction = ray.direction;
	data.hit = ray.origin + t_hit * ray.direction;

	data.world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	data.world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	data.surface_id = surface_id;

#ifdef ANTIMATTER
	bool continue_ray = false;
	if (data.mat.type == MAT_CLIP) {
		if (dot(data.world_geometric_normal, ray.direction) < 0.0f) {
			/* Entering a volume */
			prd_shadow.mask |= data.mat.mask;
		}
		else if ((prd_shadow.mask & data.mat.mask) && prd_shadow.inside > 0 && data.mat.proxy > -1) {
			/* Leaving a volume and rendering the alternate material */
			data.mat = material_data[data.mat.proxy]; // TODO this will produce odd results if the proxy material is transparent
		}
		else {
			/* Just leave the volume */
			prd_shadow.mask &= ~data.mat.mask;
		}
	}
	else if (prd_shadow.mask & data.mat.mask) {
		/* Entering or leaving the material while in antimatter. */
		prd_shadow.inside += dot(data.world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;
		continue_ray = true;
	}
#endif /* ANTIMATTER */

	if (surface_id != -prd_shadow.target - 1) { // Hit the wrong surface
		while (data.mat.type == MAT_ILLUM) {
			if (data.mat.proxy < 0) return; // This should be disallowed by any hit program
			data.mat = material_data[data.mat.proxy];
		}
	}
	int pid = data.mat.shadow_program_id;
#ifdef ANTIMATTER
	if (continue_ray)
		pid = RT_PROGRAM_ID_NULL;
#endif /* ANTIMATTER */

	if (pid != RT_PROGRAM_ID_NULL)
		prd_shadow = rtMarkedCallableProgramId<PerRayData_shadow(IntersectData const&, PerRayData_shadow)>(pid, "closest_hit_shadow_call_site")(data, prd_shadow);
	else {
		/* Continue the ray */
		Ray new_ray = make_Ray(ray.origin, ray.direction, SHADOW_RAY, t_hit + ray_start(data.hit, ray.direction, data.world_geometric_normal, RAY_START), RAY_END);
		rtTrace(top_object, new_ray, prd_shadow);
	}

	#ifdef CONTRIB
		contribution(prd_shadow.rcoef, prd_shadow.result, ray.direction, data.mat.contrib_index, data.mat.contrib_function);
	#endif
}

RT_PROGRAM void closest_hit_point_cloud()
{
	IntersectData data;
	data.mat = material_data[mat_id];
	data.ray_type = ray.ray_type;
	data.t = t_hit;
	data.ray_direction = ray.direction;
	data.hit = ray.origin + t_hit * ray.direction;

	data.world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	data.world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));

	data.surface_id = surface_id;

#ifdef ANTIMATTER
	bool continue_ray = false;
	if (data.mat.type == MAT_CLIP) {
		if (dot(data.world_geometric_normal, ray.direction) < 0.0f) {
			/* Entering a volume */
			prd_point_cloud.mask |= data.mat.mask;
		}
		else if ((prd_point_cloud.mask & data.mat.mask) && prd_point_cloud.inside > 0 && data.mat.proxy > -1) {
			/* Leaving a volume and rendering the alternate material */
			data.mat = material_data[data.mat.proxy]; // TODO this will produce odd results if the proxy material is transparent
		}
		else {
			/* Just leave the volume */
			prd_point_cloud.mask &= ~data.mat.mask;
		}
	}
	else if (prd_point_cloud.mask & data.mat.mask) {
		/* Entering or leaving the material while in antimatter. */
		prd_point_cloud.inside += dot(data.world_geometric_normal, ray.direction) < 0.0f ? 1 : -1;
		continue_ray = true;
	}
#endif /* ANTIMATTER */

	while (data.mat.type == MAT_ILLUM) {
		if (data.mat.proxy < 0) return; // This should be disallowed by any hit program
		data.mat = material_data[data.mat.proxy];
	}
	int pid = data.mat.point_cloud_program_id;
#ifdef ANTIMATTER
	if (continue_ray)
		pid = RT_PROGRAM_ID_NULL;
#endif /* ANTIMATTER */

	if (pid != RT_PROGRAM_ID_NULL)
		prd_point_cloud = rtMarkedCallableProgramId<PerRayData_point_cloud(IntersectData const&, PerRayData_point_cloud)>(pid, "closest_hit_point_cloud_call_site")(data, prd_point_cloud);
	else {
		/* Continue the ray */
		Ray new_ray = make_Ray(ray.origin, ray.direction, POINT_CLOUD_RAY, t_hit + ray_start(data.hit, ray.direction, data.world_geometric_normal, RAY_START), RAY_END);
		rtTrace(top_object, new_ray, prd_point_cloud);
	}
}
