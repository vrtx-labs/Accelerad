#include "hip/hip_runtime.h"
/*
 *  triangle_mesh.cu - intersection program for triangles on GPUs.
 */

#include "accelerad_copyright.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "optix_shader_common.h"

#define RTX

using namespace optix;

/* Program variables */
rtDeclareVariable(unsigned int, backvis, , ); /* backface visibility (bv) */

/* Instance variables */
rtDeclareVariable(int, sole_material, , ); /* sole material index to use for all objects, or -1 to use per-face material index */

/* Contex variables */
rtBuffer<float3> vertex_buffer;
rtBuffer<float3> normal_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<uint3>  vindex_buffer;    // position indices 
//rtBuffer<uint3>  nindex_buffer;    // normal indices
//rtBuffer<uint3>  tindex_buffer;    // texcoord indices
rtBuffer<unsigned int>    material_buffer; // per-face material index

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );

/* Attributes */
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, surface_id, attribute surface_id, );
rtDeclareVariable(int, mat_id, attribute mat_id, );


#ifdef RTX
RT_PROGRAM void mesh_attribute()
{
	const uint3 v_idx = vindex_buffer[rtGetPrimitiveIndex()];

	const float3 v0 = vertex_buffer[v_idx.x];
	const float3 v1 = vertex_buffer[v_idx.y];
	const float3 v2 = vertex_buffer[v_idx.z];
	const float3 Ng = optix::cross(v1 - v0, v2 - v0);

	geometric_normal = optix::normalize(Ng);

	const float2 barycentrics = rtGetTriangleBarycentrics();
	texcoord = make_float3(barycentrics.x, barycentrics.y, 0.0f);

	if (normal_buffer.size() == 0)
	{
		shading_normal = geometric_normal;
	}
	else
	{
		shading_normal = normal_buffer[v_idx.y] * barycentrics.x + normal_buffer[v_idx.z] * barycentrics.y
			+ normal_buffer[v_idx.x] * (1.0f - barycentrics.x - barycentrics.y);
	}

	if (texcoord_buffer.size() == 0)
	{
		texcoord = make_float3(0.0f, 0.0f, 0.0f);
	}
	else
	{
		const float2 t0 = texcoord_buffer[v_idx.x];
		const float2 t1 = texcoord_buffer[v_idx.y];
		const float2 t2 = texcoord_buffer[v_idx.z];
		texcoord = make_float3(t1*barycentrics.x + t2*barycentrics.y + t0*(1.0f - barycentrics.x - barycentrics.y));
	}

	surface_id = v_idx.x; // Not necessarily unique per triangle, but different for each surface

	int mat = sole_material;
	if (mat < 0) /* Use per-face material index */
		mat = material_buffer[rtGetPrimitiveIndex()];
	mat_id = mat;
}
#else
RT_PROGRAM void mesh_intersect(unsigned int primIdx)
{
	uint3 v_idx = vindex_buffer[primIdx];

	float3 p0 = vertex_buffer[ v_idx.x ];
	float3 p1 = vertex_buffer[ v_idx.y ];
	float3 p2 = vertex_buffer[ v_idx.z ];

	// Intersect ray with triangle
	float3 n;
	float  t, beta, gamma;
	if( intersect_triangle( ray, p0, p1, p2, n, t, beta, gamma ) && ( backvis || dot( n, ray.direction ) < 0) ) {

		int mat = sole_material;
		if ( mat < 0 ) /* Use per-face material index */
			mat = material_buffer[primIdx];

		if ( rtPotentialIntersection( t ) ) {

			//int3 n_idx = nindex_buffer[ primIdx ];
			geometric_normal = normalize( n );
			if ( normal_buffer.size() == 0 ) { //|| n_idx.x < 0 || n_idx.y < 0 || n_idx.z < 0 ) {
				shading_normal = geometric_normal;
			} else {
				float3 n0 = normal_buffer[ v_idx.x ];
				float3 n1 = normal_buffer[ v_idx.y ];
				float3 n2 = normal_buffer[ v_idx.z ];
				shading_normal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );
				if ( !isfinite( shading_normal ) )
					shading_normal = geometric_normal;
			}

			//int3 t_idx = tindex_buffer[ primIdx ];
			if ( texcoord_buffer.size() == 0 ) { //|| t_idx.x < 0 || t_idx.y < 0 || t_idx.z < 0 ) {
				texcoord = make_float3( 0.0f, 0.0f, 0.0f );
			} else {
				float2 t0 = texcoord_buffer[ v_idx.x ];
				float2 t1 = texcoord_buffer[ v_idx.y ];
				float2 t2 = texcoord_buffer[ v_idx.z ];
				texcoord = make_float3( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );
			}

			surface_id = v_idx.x; // Not necessarily unique per triangle, but different for each surface
			mat_id = mat;

			rtReportIntersection(0);
		}
	}
}

RT_PROGRAM void mesh_bounds(unsigned int primIdx, float result[6])
{  
	const uint3 v_idx = vindex_buffer[primIdx];

	const float3 v0   = vertex_buffer[ v_idx.x ];
	const float3 v1   = vertex_buffer[ v_idx.y ];
	const float3 v2   = vertex_buffer[ v_idx.z ];
	const float  area = length(cross(v1-v0, v2-v0));

	Aabb* aabb = (Aabb*)result;
  
	if(area > 0.0f && !isinf(area)) {
		aabb->m_min = fminf( fminf( v0, v1), v2 );
		aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
	} else {
		aabb->invalidate();
	}
}
#endif
