#include "hip/hip_runtime.h"
/*
 *  ambient_cloud_generator.cu - entry point for geometry sampling on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_ray.h"
#include "optix_point_common.h"

using namespace optix;

#ifdef DAYSIM_COMPATIBLE
#define threadIndex()	((launch_index.x + launch_dim.x * launch_index.y) / stride + segment_offset)
#else
#define threadIndex()	((launch_index.x + launch_dim.x * launch_index.y) / stride)
#endif

/* Program variables */
rtDeclareVariable(unsigned int,  stride, , ) = 1u; /* Spacing between used threads in warp. */

/* Contex variables */
//rtBuffer<PointDirection, 1>      cluster_buffer; /* input */
rtDeclareVariable(PointDirectionBuffer, cluster_buffer, , ); /* input */
rtBuffer<AmbientRecord, 1>       ambient_record_buffer; /* ambient record output */
#ifdef DAYSIM_COMPATIBLE
rtBuffer<DC, 2>                  ambient_dc_buffer; /* daylight coefficient output */
#endif
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(rtObject,      top_irrad, , );
rtDeclareVariable(unsigned int,  level, , ) = 0u;
#ifdef DAYSIM_COMPATIBLE
rtDeclareVariable(unsigned int,  segment_offset, , ) = 0u; /* Offset into data if computed with multiple segments */
#endif /* DAYSIM_COMPATIBLE */
rtDeclareVariable(unsigned int,  imm_irrad, , ) = 0u; /* Immediate irradiance (-I) */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
//rtDeclareVariable(unsigned int, launch_index, rtLaunchIndex, );
//rtDeclareVariable(unsigned int, launch_dim,   rtLaunchDim, );


RT_PROGRAM void ambient_cloud_camera()
{
	// Check stride
	if ((launch_index.x + launch_dim.x * launch_index.y) % stride)
		return;
	const unsigned int index = threadIndex();
	if (index >= cluster_buffer.size())
		return;

	PerRayData_ambient_record prd;
	init_rand(&prd.state, launch_index.x + launch_dim.x * (launch_index.y + launch_dim.y * level));
	prd.result.pos = prd.result.val = make_float3( 0.0f );
	prd.result.lvl = level;
	prd.result.weight = 1.0f;
	for ( int i = level; i--; )
		prd.result.weight *= AVGREFL; // Compute weight as in makeambient() from ambient.c
	prd.result.rad = make_float2( 0.0f );
	prd.result.udir = 0; // Initialize in case something goes wrong
#ifdef DAYSIM_COMPATIBLE
	prd.dc = make_uint3(0, 0, index - segment_offset);
	daysimSet(prd.dc, 0.0f);
#endif
#ifdef RAY_COUNT
	prd.result.ray_count = 1;
#endif
#ifdef HIT_COUNT
	prd.result.hit_count = 0;
#endif

	// Get the position and normal of the ambient record to be created
	PointDirection cluster = cluster_buffer[index];

	if ( dot( cluster.dir, cluster.dir ) > FTINY ) { // Check that this is a valid ray
		float3 ray_direction = -normalize( cluster.dir ); // Ray will face opposite the normal direction
		const float tmax = ray_start( cluster.pos, RAY_START );
		if (imm_irrad && !level) {
			Ray ray = make_Ray(cluster.pos, ray_direction, AMBIENT_RECORD_RAY, 0.0f, tmax); // For rtrace, the position is already offset
			rtTrace(top_irrad, ray, prd);
		}
		else {
			Ray ray = make_Ray(cluster.pos - ray_direction * tmax, ray_direction, AMBIENT_RECORD_RAY, 0.0f, 2.0f * tmax);
			rtTrace(top_object, ray, prd);
		}
	}

	checkFinite(prd.result.val);
	checkFinite(prd.result.gdir);

	ambient_record_buffer[index] = prd.result;
#ifdef DAYSIM_COMPATIBLE
	if (ambient_dc_buffer.size().x)
		daysimCopy(&ambient_dc_buffer[make_uint2(0, index)], prd.dc);
#endif
}

RT_PROGRAM void exception()
{
	// Check stride
	if ((launch_index.x + launch_dim.x * launch_index.y) % stride)
		return;
	const unsigned int index = threadIndex();
	if (index >= ambient_record_buffer.size())
		return;

#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
	ambient_record_buffer[index].lvl = level;
	ambient_record_buffer[index].val = exceptionToFloat3(rtGetExceptionCode());
	ambient_record_buffer[index].weight = -1.0f;
}
