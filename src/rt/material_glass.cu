#include "hip/hip_runtime.h"
/*
 *  material_glass.cu - hit programs for glass materials on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_ray.h"
#ifdef CONTRIB_DOUBLE
#include "optix_double.h"
#endif

using namespace optix;

/* Context variables */
rtDeclareVariable(rtObject,     top_object, , );


RT_CALLABLE_PROGRAM PerRayData_shadow closest_hit_glass_shadow(IntersectData const&data, PerRayData_shadow prd_shadow)
{
	float3 ffnormal = faceforward(data.world_shading_normal, -data.ray_direction, data.world_geometric_normal);
	const float3 snormal = faceforward(data.world_geometric_normal, -data.ray_direction, data.world_geometric_normal);

	float3 mcolor = data.mat.color;

	/* check transmission */
	const bool hastrans = fmaxf( mcolor ) > 1e-15f;
	if (!hastrans) {
		return prd_shadow;
	}
	mcolor = fmaxf(mcolor, make_float3(1e-15f)); // no color channel should be smaller than 1e-15

	/* get modifiers */
	// we'll skip this for now

	/* perturb normal */
	// if there's a bump map, we use that, else
	float pdot = -dot( data.ray_direction, ffnormal );
	if (pdot < 0.0f) {		/* fix orientation from raynormal in raytrace.c */
		ffnormal += 2.0f * pdot * data.ray_direction;
		pdot = -pdot;
	}

	/* angular transmission */
	float cos2 = sqrtf(1.0f + (pdot * pdot - 1.0f) / (data.mat.params.r_index * data.mat.params.r_index));
	//if (hastrans) {
	mcolor = make_float3( powf( mcolor.x, 1.0f / cos2 ), powf( mcolor.y, 1.0f / cos2 ), powf( mcolor.z, 1.0f / cos2 ) );
	//}

	/* compute reflection */
	float r1e = (pdot - data.mat.params.r_index * cos2) / (pdot + data.mat.params.r_index * cos2);
	r1e *= r1e;
	float r1m = (1.0f / pdot - data.mat.params.r_index / cos2) / (1.0f / pdot + data.mat.params.r_index / cos2);
	r1m *= r1m;

	/* compute transmission */
	//if (hastrans) {
		float3 trans = 0.5f * (1.0f-r1e) * (1.0f-r1e) * mcolor / (1.0f - r1e * r1e * mcolor * mcolor);
		trans       += 0.5f * (1.0f-r1m) * (1.0f-r1m) * mcolor / (1.0f - r1m * r1m * mcolor * mcolor);

		/* modify by pattern */
		//trans *= pcol;

		/* transmitted ray */
#ifdef CONTRIB
		prd_shadow.rcoef *= trans;
#endif
		Ray trans_ray = make_Ray(data.hit, data.ray_direction, SHADOW_RAY, ray_start(data.hit, data.ray_direction, snormal, RAY_START), RAY_END);
		rtTrace(top_object, trans_ray, prd_shadow);
		prd_shadow.result *= trans;
#ifdef DAYSIM_COMPATIBLE
		daysimScale(prd_shadow.dc, trans.x);
#endif
	//}
	return prd_shadow;
}


RT_CALLABLE_PROGRAM PerRayData_radiance closest_hit_glass_radiance(IntersectData const&data, PerRayData_radiance prd)
{
	float3 ffnormal = faceforward(data.world_shading_normal, -data.ray_direction, data.world_geometric_normal);
	const float3 snormal = faceforward(data.world_geometric_normal, -data.ray_direction, data.world_geometric_normal);

	PerRayData_radiance new_prd;
	float3 result = prd.mirror = make_float3(0.0f);
	float3 mcolor = data.mat.color;

	/* check transmission */
	const bool hastrans = fmaxf( mcolor ) > 1e-15f;
	if (hastrans) {
		mcolor = fmaxf( mcolor, make_float3( 1e-15f ) ); // no color channel should be smaller than 1e-15
	} // else we return if it's a shadow ray, which it isn't

	/* get modifiers */
	// we'll skip this for now

	/* perturb normal */
	float3 pert = snormal - ffnormal;
	int hastexture = dot(pert, pert) > FTINY * FTINY;
	float pdot = -dot(data.ray_direction, ffnormal);
	if (pdot < 0.0f) {		/* fix orientation from raynormal in raytrace.c */
		ffnormal += 2.0f * pdot * data.ray_direction;
		pdot = -pdot;
	}

	/* angular transmission */
	float cos2 = sqrtf(1.0f + (pdot * pdot - 1.0f) / (data.mat.params.r_index * data.mat.params.r_index));
	if (hastrans) {
		mcolor = make_float3( powf( mcolor.x, 1.0f / cos2 ), powf( mcolor.y, 1.0f / cos2 ), powf( mcolor.z, 1.0f / cos2 ) );
	}

	/* compute reflection */
	float r1e = (pdot - data.mat.params.r_index * cos2) / (pdot + data.mat.params.r_index * cos2);
	r1e *= r1e;
	float r1m = (1.0f / pdot - data.mat.params.r_index / cos2) / (1.0f / pdot + data.mat.params.r_index / cos2);
	r1m *= r1m;

	/* compute transmission */
	if (hastrans) {
		float3 trans = 0.5f * (1.0f - r1e) * (1.0f - r1e) * mcolor / (1.0f - r1e * r1e * mcolor * mcolor);
		trans       += 0.5f * (1.0f-r1m) * (1.0f-r1m) * mcolor / (1.0f - r1m * r1m * mcolor * mcolor);

		/* modify by pattern */
		//trans *= pcol;

		/* transmitted ray */
		if (rayorigin(new_prd, prd, trans, 0, 0)) {
			new_prd.result = make_float3(0.0f);
#ifdef DAYSIM_COMPATIBLE
			new_prd.dc = daysimNext(prd.dc);
#endif
			float3 R = data.ray_direction;

			if (!new_prd.ambient_depth && hastexture) {
				R = normalize(data.ray_direction + pert * (2.0f * (1.0f - data.mat.params.r_index)));
				if (isnan(R))
					R = data.ray_direction;
			}

			setupPayload(new_prd);
			Ray trans_ray = make_Ray(data.hit, R, RADIANCE_RAY, ray_start(data.hit, R, snormal, RAY_START), new_prd.tmax);
			rtTrace(top_object, trans_ray, new_prd);
			new_prd.result *= trans;
			result += new_prd.result;
			if (prd.ambient_depth || !hastexture)
				prd.distance = data.t + rayDistance(new_prd);
#ifdef DAYSIM_COMPATIBLE
			daysimAddScaled(prd.dc, new_prd.dc, trans.x);
#endif
			resolvePayload(prd, new_prd);
		}
	}
	// stop if it's a shadow ray, which it isn't

	/* compute reflectance */
	float3 refl = 0.5f * r1e * ( 1.0f + (1.0f-2.0f*r1e) * mcolor * mcolor ) / (1.0f - r1e * r1e * mcolor * mcolor );
	refl       += 0.5f * r1m * ( 1.0f + (1.0f-2.0f*r1m) * mcolor * mcolor ) / (1.0f - r1m * r1m * mcolor * mcolor );

	/* reflected ray */
	if (rayorigin(new_prd, prd, refl, 1, 0)) {
		new_prd.result = make_float3(0.0f);
#ifdef DAYSIM_COMPATIBLE
		new_prd.dc = daysimNext(prd.dc);
#endif
		setupPayload(new_prd);
		float3 R = reflect(data.ray_direction, ffnormal);
		Ray refl_ray = make_Ray(data.hit, R, RADIANCE_RAY, ray_start(data.hit, R, snormal, RAY_START), new_prd.tmax);
		rtTrace(top_object, refl_ray, new_prd);
		new_prd.result *= refl;
		prd.mirror = new_prd.result;
		result += new_prd.result;
		prd.mirror_distance = data.t;
		if (prd.ambient_depth || !hastexture)
			prd.mirror_distance += rayDistance(new_prd);
#ifdef DAYSIM_COMPATIBLE
		daysimAddScaled(prd.dc, new_prd.dc, refl.x);
#endif
		resolvePayload(prd, new_prd);
	}
  
	// pass the color back up the tree
	prd.result = result;
	return prd;
}
