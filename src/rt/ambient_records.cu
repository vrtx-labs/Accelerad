#include "hip/hip_runtime.h"
/*
 *  ambient_records.cu - intersection testing for irradiance caching on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>
#include "optix_shader_ray.h"
#include "optix_shader_ambient.h"

using namespace optix;

rtBuffer<AmbientRecord> ambient_records;
#ifdef DAYSIM_COMPATIBLE
rtBuffer<DC, 2> ambient_dc;
#endif

rtDeclareVariable(float,        ambacc, , ); /* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(float,        minarad, , ); /* minimum ambient radius */

rtDeclareVariable(int, record_id, attribute record_id_attribute, );
rtDeclareVariable(float3, w, attribute w_attribute, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_ambient, prd, rtPayload, );


RT_METHOD void sumambient();
RT_METHOD int plugaleak(const AmbientRecord* record, const float3& anorm, const float3& normal, float ang);


// Ignore the intersection so that the intersection program will continue to run for all overlapping recrods.
RT_PROGRAM void ambient_record_any_hit()
{
#ifdef HIT_COUNT
	prd.hit_count++;
#endif
	sumambient();
	rtIgnoreIntersection(); // Continue checking other intersections
}

// based on sumambient from ambient.c
RT_METHOD void sumambient()
{
	const AmbientRecord record = ambient_records[record_id];

	/* Ambient level test. */
	if ( record.lvl > prd.ambient_depth )
		return;
	if (record.lvl == prd.ambient_depth && record.weight < 0.9f * prd.weight)
		return;

	const float3 normal = ray.direction;

	/* Direction test using unperturbed normal */
	float d = dot(w, normal); // Ray direction is unperturbed surface normal
	if ( d <= 0.0f )		/* >= 90 degrees */
		return;
	if (d > 1.0f)
		d = 1.0f;

	float delta_r2 = 2.0f - 2.0f * d;	/* approx. radians^2 */
	const float minangle = 10.0f * M_PIf / 180.0f;
	float maxangle = minangle + ambacc;
	/* adjust maximum angle */
	if (prd.weight < 0.6f)
		maxangle = (maxangle - M_PI_2f) * powf(prd.weight, 0.13f) + M_PI_2f;
	if ( delta_r2 >= maxangle * maxangle )
		return;

	/* Modified ray behind test */
	float3 ck0 = prd.surface_point - record.pos;
	d = dot( ck0, w );
	if ( d < -minarad * ambacc - 0.001f )
		return;
	d /= record.rad.x;
	float delta_t2 = d * d;
	if ( delta_t2 >= ambacc * ambacc )
		return;

	/* Elliptical radii test based on Hessian */
	float3 u = decodedir( record.udir );
	float3 v = cross( w, u );
	float uu, vv;
	d = (uu = dot( ck0, u )) / record.rad.x;
	delta_t2 += d * d;
	d = (vv = dot( ck0, v )) / record.rad.y;
	delta_t2 += d * d;
	if ( delta_t2 >= ambacc * ambacc )
		return;

	/* Test for potential light leak */
	if (record.corral && plugaleak(&record, w, normal, atan2f(vv, uu)))
		return;

	/* Extrapolate value and compute final weight (hat function) */
	/* This is extambient from ambient.c */
	/* gradient due to translation */
	d = 1.0f + dot( ck0, record.gpos.x * u + record.gpos.y * v );

	/* gradient due to rotation */
	ck0 = cross( w, prd.surface_normal );
	d += dot( ck0, record.gdir.x * u + record.gdir.y * v );

	//if (d < min_d)			/* should not use if we can avoid it */
	//	d = min_d;
	if ( d <= 0.05f )
		return;

	float wt = (1.0f - sqrtf(delta_r2) / maxangle) * (1.0f - sqrtf(delta_t2) / ambacc);
	prd.wsum += wt;

	// This assignment to the prd would take place in the any-hit program if there were one
	prd.result += record.val * (d * wt);
#ifdef DAYSIM_COMPATIBLE
	if (ambient_dc.size().x && prd.dc.x)
		daysimAddScaled(prd.dc, &ambient_dc[make_uint2(0, primIdx)], d * wt);
#endif
}

// based on makeambient from ambient.c
RT_PROGRAM void ambient_miss()
{
	//if ( prd.wsum == 0.0f )
	//	rtThrow( RT_EXCEPTION_CUSTOM );
}

RT_PROGRAM void ambient_record_intersect( int primIdx )
{
	const AmbientRecord record = ambient_records[primIdx];

	/* Check for intersection with plane */
	const float3 disk_normal = decodedir(record.ndir);

	const float d = dot(disk_normal, ray.direction); // Ray direction is unperturbed surface normal
	if (d <= 0.0f)		/* >= 90 degrees */
		return;

	const float t = dot(disk_normal, record.pos - ray.origin) / d;

	if (rtPotentialIntersection(t)) {
		w = disk_normal;
		record_id = primIdx;
		rtReportIntersection(0); // There is only one material for ambient geometry group
	}
}

RT_PROGRAM void ambient_record_bounds (int primIdx, float result[6])
{
	optix::Aabb* aabb = (optix::Aabb*)result;
	const AmbientRecord record = ambient_records[primIdx];
	const float2 rad = record.rad * ambacc; // Acceleration structure becomes dirty when ambacc is increased.

	if( rad.x > FTINY && isfinite(rad.y) ) {
		const float3 w = decodedir( record.ndir );
		const float3 u = decodedir( record.udir );
		const float3 v = cross( w, u );
		const float3 dims = sqrtf( u*u*(rad.x*rad.x) + v*v*(rad.y*rad.y) );// + FTINY; // Expanding by FTINY seems to help prevent misses
		//const float3 dims = sqrtf( u*u*(rad.x*rad.x) + v*v*(rad.y*rad.y) + w*w*(rad.x*rad.x) );// + FTINY;
		aabb->m_min = record.pos - dims;
		aabb->m_max = record.pos + dims;
	} else {
		aabb->invalidate();
	}
}

/* Plug a potential leak where ambient cache value is occluded */
RT_METHOD int plugaleak(const AmbientRecord* record, const float3& anorm, const float3& normal, float ang)
{
	const float cost70sq = 0.1169778f;	/* cos(70deg)^2 */
	float2 t;

	ang += 2.0f * M_PIf * (ang < 0);			/* check direction flags */
	if ( !(record->corral>>(int)(ang * 16.0f * M_1_PIf) & 1) )
		return(0);
	/*
	 * Generate test ray, targeting 20 degrees above sample point plane
	 * along surface normal from cache position.  This should be high
	 * enough to miss local geometry we don't really care about.
	 */
	float3 vdif = record->pos - prd.surface_point;
	float normdot = dot(anorm, normal);
	float ndotd = dot(vdif, normal);
	float nadotd = dot( vdif, anorm );
	float a = normdot * normdot - cost70sq;
	float b = 2.0f * ( normdot * ndotd - nadotd * cost70sq );
	float c = ndotd * ndotd - dot( vdif, vdif ) * cost70sq;
	if ( quadratic( &t, a, b, c ) != 2 )
		return(1);			/* should rarely happen */
	//if ( t.y <= FTINY )
		return(0);			/* should fail behind test */

	/* Can't shoot rays from an intersection program. */
	//float3 rdir = vdif + anorm * t.y;	/* further dist. > plane */
	//Ray shadow_ray = make_Ray( prd.surface_point, normalize( rdir ), SHADOW_RAY, RAY_START, length( rdir ) );
	//PerRayData_shadow shadow_prd;
	//shadow_prd.result = make_float3( 1.0f );
#ifdef CONTRIB
	//shadow_prd.rcoef = make_contrib3(0.0f);
#endif
#ifdef ANTIMATTER
	//shadow_prd.mask = prd.mask;
	//shadow_prd.inside = prd.inside;
#endif
	//rtTrace( top_object, shadow_ray, shadow_prd );
	//return( dot( shadow_prd.result, shadow_prd.result ) < 1.0f );	/* check for occluder */
}
