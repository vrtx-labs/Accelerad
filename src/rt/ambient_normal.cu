#include "hip/hip_runtime.h"
/*
 *  ambient_normal.cu - hit programs for ambient sampling on GPUs.
 */

#include "accelerad_copyright.h"

#include "otypes.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "optix_shader_ray.h"
#include "optix_shader_ambient.h"
#include "optix_ambient_common.h"

using namespace optix;

#define threadIndex()	(launch_index.y / stride)
//#define threadIndex()	((launch_index.x + launch_dim.x * launch_index.y) / stride)
#define CORRAL
#define hessrow(i)	hess_row_buffer[make_uint2(i, threadIndex())]
#define gradrow(i)	grad_row_buffer[make_uint2(i, threadIndex())]
#ifdef AMB_SAVE_MEM
#define prevrow(i)	amb_samp_buffer[make_uint2(i, threadIndex())]
#define corral_u(i)	corral_u_buffer[make_uint2(i, threadIndex())]
#define corral_d(i)	corral_d_buffer[make_uint2(i, threadIndex())]
#else /* AMB_SAVE_MEM */
#ifdef DAYSIM_COMPATIBLE
#define ambsam(i,j)	amb_samp_buffer[make_uint3(i, j, threadIndex() + segment_offset)]
#else /* DAYSIM_COMPATIBLE */
#define ambsam(i,j)	amb_samp_buffer[make_uint3(i, j, threadIndex())]
#endif /* DAYSIM_COMPATIBLE */
#ifdef AMB_SUPER_SAMPLE
#define earr(i,j)	earr_buffer[make_uint3(i, j, threadIndex())]
#endif
#endif /* AMB_SAVE_MEM */

typedef struct {
	unsigned int	ns;		/* number of samples per axis */
	int	sampOK;		/* acquired full sample set? */
	float3	acoef;		/* division contribution coefficient */
	float3	acol;		/* accumulated color */
	float3	ux, uy;		/* tangent axis unit vectors */
} AMBHEMI;		/* ambient sample hemisphere */

typedef struct {
	float3 r_i, r_i1, e_i, rcp, rI2_eJ2;
	float I1, I2;
} FFTRI;		/* vectors and coefficients for Hessian calculation */

/* Context variables */
rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(unsigned int, stride, , ) = 1u; /* Spacing between used threads in warp. */
#ifdef DAYSIM_COMPATIBLE
rtDeclareVariable(unsigned int, segment_offset, , ) = 0u; /* Offset into data if computed with multiple segments */
#endif /* DAYSIM_COMPATIBLE */

//rtDeclareVariable(float,        specthresh, , ); /* This is the minimum fraction of reflection or transmission, under which no specular sampling is performed */
//rtDeclareVariable(float,        specjitter, , );

//rtDeclareVariable(float3,       ambval, , ); /* This is the final value used in place of an indirect light calculation */
//rtDeclareVariable(int,          ambvwt, , ); /* As new indirect irradiances are computed, they will modify the default ambient value in a moving average, with the specified weight assigned to the initial value given on the command and all other weights set to 1 */
//rtDeclareVariable(int,          ambounce, , ); /* Ambient bounces (ab) */
//rtDeclareVariable(int,          ambres, , ); /* Ambient resolution (ar) */
rtDeclareVariable(float,        ambacc, , ); /* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(int,          ambdiv, , ); /* Ambient divisions (ad) */
rtDeclareVariable(int,          ambssamp, , ); /* Ambient super-samples (as) */
rtDeclareVariable(float,        maxarad, , ); /* maximum ambient radius */
rtDeclareVariable(float,        minarad, , ); /* minimum ambient radius */
//rtDeclareVariable(float,        avsum, , ); /* computed ambient value sum (log) */
//rtDeclareVariable(unsigned int, navsum, , ); /* number of values in avsum */

rtBuffer<MaterialData> material_data;	/* One entry per Radiance material. */

/* Program variables */
rtBuffer<optix::Matrix<3, 3>, 2> hess_row_buffer;
rtBuffer<float3, 2>              grad_row_buffer;
#ifdef AMB_SAVE_MEM
rtBuffer<AmbientSample, 2>       amb_samp_buffer;
rtBuffer<float2, 2>              corral_u_buffer;
rtBuffer<float, 2>               corral_d_buffer;
#else /* AMB_SAVE_MEM */
rtBuffer<AmbientSample, 3>       amb_samp_buffer;
#ifdef AMB_SUPER_SAMPLE
rtBuffer<float, 3>               earr_buffer;
#endif
#endif /* AMB_SAVE_MEM */

/* OptiX variables */
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_ambient_record, prd, rtPayload, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );

/* Attributes */
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(int, mat_id, attribute mat_id, );


#ifdef CHECK_OVERLAP
RT_METHOD int check_overlap( const float3& normal, const float3& hit );
RT_METHOD int plugaleak( const AmbientRecord* record, const float3& anorm, const float3& normal, const float3& hit, float ang );
#endif
RT_METHOD int doambient( float3 *rcol, optix::Matrix<2,3> *uv, float2 *ra, float2 *pg, float2 *dg, unsigned int *crlp, const float3& normal, const float3& hit );
RT_METHOD int ambsample(AMBHEMI *hp, AmbientSample *ap, const unsigned int& i, const unsigned int& j, const unsigned int& n, const float3& normal, const float3& hit);
#ifdef AMB_SAVE_MEM
RT_METHOD int samp_hemi(AMBHEMI *hp, float3 *rcol, float wt, optix::Matrix<2, 3> *uv, float2 *ra, float2 *pg, float2 *dg, unsigned int *crlp, const float3& normal, const float3& hit);
#else /* AMB_SAVE_MEM */
#ifdef AMB_SUPER_SAMPLE
RT_METHOD void getambdiffs(AMBHEMI *hp);
RT_METHOD void ambsupersamp(AMBHEMI *hp, int cnt, const float3& normal, const float3& hit);
#endif /* AMB_SUPER_SAMPLE */
RT_METHOD int samp_hemi( AMBHEMI *hp, float3 *rcol, float wt, const float3& normal, const float3& hit );
RT_METHOD void ambHessian( AMBHEMI *hp, optix::Matrix<2,3> *uv, float2 *ra, float2 *pg, const float3& normal, const float3& hit );
RT_METHOD void ambdirgrad( AMBHEMI *hp, const float3& u, const float3& v, float2 *dg, const float3& normal, const float3& hit );
RT_METHOD unsigned int ambcorral( AMBHEMI *hp, optix::Matrix<2,3> *uv, const float2& r, const float3& hit );
#endif /* AMB_SAVE_MEM */
RT_METHOD float back_ambval( const AmbientSample *n1, const AmbientSample *n2, const AmbientSample *n3 );
RT_METHOD void comp_fftri( FFTRI *ftp, const AmbientSample *n0, const AmbientSample *n1, const float3& hit );
RT_METHOD optix::Matrix<3,3> compose_matrix( const float3& va, const float3& vb );
RT_METHOD optix::Matrix<3,3> comp_hessian( FFTRI *ftp, const float3& normal );
RT_METHOD float3 comp_gradient( FFTRI *ftp, const float3& normal );
RT_METHOD optix::Matrix<2,2> eigenvectors( optix::Matrix<2,3> *uv, float2 *ra, optix::Matrix<3,3> *hessian );
//RT_METHOD float2 multisamp2(float r);
//RT_METHOD int ilhash(int3 d);

RT_PROGRAM void any_hit_ambient()
{
	if (mat_id < 0 || mat_id >= material_data.size()) {
		rtIgnoreIntersection();
	}
	else {
		MaterialData mat = material_data[mat_id];

		// Check if an alternate material is used
		while ((mat.type == MAT_CLIP || mat.type == MAT_ILLUM) && mat.proxy > -1)
			mat = material_data[mat.proxy];

		// This only applies to normal materials
		if (mat.type != MAT_PLASTIC && mat.type != MAT_METAL && mat.type != MAT_TRANS)
			rtIgnoreIntersection();

		// Check that this material is included
		if (!mat.params.n.ambincl)
			rtIgnoreIntersection();
	}
}

RT_PROGRAM void closest_hit_ambient()
{
	float3 ffnormal = -ray.direction;
	float3 hit_point = ray.origin + t_hit * ray.direction;

	/* compute weight */
	//float weight = 1.0f;
	//for (int i = prd.ambient_depth; i-- > 0; ) //TODO start at i-1?
	//	weight *= AVGREFL;
	//if (prd.weight < 0.1f * weight)	/* heuristic override */
	//	weight = 1.25f * prd.weight;
	float3 acol = make_float3( AVGREFL );
#ifdef DAYSIM_COMPATIBLE
	daysimSet(prd.dc, 0.0f);
#endif
	optix::Matrix<2,3> uv;
	float2 pg = make_float2( 0.0f );
	float2 dg = make_float2( 0.0f );
	float2 rad = make_float2( 0.0f );
	unsigned int corral = 0u;

	/* compute ambient */
	int i = doambient( &acol, &uv, &rad, &pg, &dg, &corral, ffnormal, hit_point );
	if ( !i || rad.x <= FTINY )	/* no Hessian or zero radius */
		return;

	acol *= 1.0f / AVGREFL;		/* undo assumed reflectance */

	//if (rn != r->ron)
	//	extambient(acol, &amb, r->rop, rn);	/* texture */

	// pass the color back up the tree
	prd.result.pos = hit_point;
	prd.result.val = acol;
	prd.result.gpos = pg;
	prd.result.gdir = dg;
	prd.result.rad = rad;
	prd.result.ndir = encodedir( ffnormal );
	prd.result.udir = encodedir( uv.getRow(0) );
	prd.result.corral = corral;
	//prd.result.lvl = lvl;
	//prd.result.weight = weight;
#ifdef DAYSIM_COMPATIBLE
	daysimScale(prd.dc, 1.0f / AVGREFL); // TODO Scaling should be done before extambient if textured
#endif
}

#ifdef CHECK_OVERLAP /* We don't have need for this currently. */
// based on sumambient from ambient.c
RT_METHOD int check_overlap( const float3& normal, const float3& hit )
{
	// Check that there is a parent
	if ( !prd.parent )
		return( 0 );

	/* Direction test using unperturbed normal */
	float3 w = decodedir( prd.parent->ndir );
	float d = dot( w, normal );
	if ( d <= 0.0f )		/* >= 90 degrees */
		return( 0 );

	float delta_r2 = 2.0f - 2.0f * d;	/* approx. radians^2 */
	const float minangle = 10.0f * M_PIf / 180.0f;
	float maxangle = minangle + ambacc;
					/* adjust maximum angle */
	//if (at->alist != NULL && (at->alist->lvl <= al) & (r->rweight < 0.6))
	//	maxangle = (maxangle - PI/2.)*pow(r->rweight,0.13) + PI/2.;
	if ( delta_r2 >= maxangle * maxangle )
		return( 0 );

	/* Modified ray behind test */
	float3 ck0 = hit - prd.parent->pos;
	d = dot( ck0, w );
	if ( d < -minarad * ambacc - 0.001f )
		return( 0 );
	d /= prd.parent->rad.x;
	float delta_t2 = d * d;
	if ( delta_t2 >= ambacc * ambacc )
		return( 0 );
	
	/* Elliptical radii test based on Hessian */
	float3 u = decodedir( prd.parent->udir );
	float3 v = cross( w, u );
	float uu, vv;
	d = (uu = dot( ck0, u )) / prd.parent->rad.x;
	delta_t2 += d * d;
	d = (vv = dot( ck0, v )) / prd.parent->rad.y;
	delta_t2 += d * d;
	if ( delta_t2 >= ambacc * ambacc )
		return( 0 );
	
	/* Test for potential light leak */
	if ( prd.parent->corral && plugaleak( prd.parent, w, normal, hit, atan2f( vv, uu ) ) )
		return( 0 );
	return( 1 );
}

/* Plug a potential leak where ambient cache value is occluded */
RT_METHOD int plugaleak( const AmbientRecord* record, const float3& anorm, const float3& normal, const float3& hit, float ang )
{
	const float cost70sq = 0.1169778f;	/* cos(70deg)^2 */
	float2 t;

	ang += 2.0f * M_PIf * (ang < 0);			/* check direction flags */
	if ( !(record->corral>>(int)( ang * 16.0f * M_1_PIf ) & 1) )
		return(0);
	/*
	 * Generate test ray, targeting 20 degrees above sample point plane
	 * along surface normal from cache position.  This should be high
	 * enough to miss local geometry we don't really care about.
	 */
	float3 vdif = record->pos - hit;
	float normdot = dot( anorm, normal );
	float ndotd = dot( vdif, normal );
	float nadotd = dot( vdif, anorm );
	float a = normdot * normdot - cost70sq;
	float b = 2.0f * ( normdot * ndotd - nadotd * cost70sq );
	float c = ndotd * ndotd - dot( vdif, vdif ) * cost70sq;
	if ( quadratic( &t, a, b, c ) != 2 )
		return(1);			/* should rarely happen */
	if ( t.y <= FTINY )
		return(0);			/* should fail behind test */

	float3 rdir = vdif + anorm * t.y;	/* further dist. > plane */
	Ray shadow_ray = make_Ray( hit, normalize( rdir ), SHADOW_RAY, RAY_START, length( rdir ) );
	PerRayData_shadow shadow_prd;
	shadow_prd.target = 0;
	shadow_prd.result = make_float3( 1.0f );
#ifdef CONTRIB
	shadow_prd.rcoef = make_contrib3(0.0f);
#endif
#ifdef ANTIMATTER
	shadow_prd.mask = 0u; //TODO check if we are in an antimatter volume
	shadow_prd.inside = 0;
#endif
	rtTrace( top_object, shadow_ray, shadow_prd );
	return( dot( shadow_prd.result, shadow_prd.result ) < 1.0f );	/* check for occluder */
}
#endif /* CHECK_OVERLAP */

RT_METHOD int doambient( float3 *rcol, optix::Matrix<2,3> *uv, float2 *ra, float2 *pg, float2 *dg, unsigned int *crlp, const float3& normal, const float3& hit )
{
	const float wt = prd.result.weight;
	AMBHEMI hp;

	if (!samp_hemi(&hp, rcol, wt,
#ifdef AMB_SAVE_MEM
		uv, ra, pg, dg, crlp,
#endif
		normal, hit))
		return(0);

	/* clear return values */
	//if (u != NULL)
	//	*u = make_float3( 0.0f );
	//if (v != NULL)
	//	*v = make_float3( 0.0f );
	//if (ra != NULL)
	//	*ra = make_float2( 0.0f );
	//if (pg != NULL)
	//	*pg = make_float2( 0.0f );
	//if (dg != NULL)
	//	*dg = make_float2( 0.0f );
	//if (crlp != NULL)
	//	*crlp = 0u;
	//if (hp == NULL)			/* sampling falure? */
	//	return(0);

	if ((ra == NULL) & (pg == NULL) & (dg == NULL) || (hp.sampOK < 0) | (hp.ns < 6)) { /* Hessian not requested/possible */
		return(-1);		/* value-only return value */
	}
#ifndef AMB_SAVE_MEM
	float	d, K;
	if ((d = bright(*rcol)) > FTINY) {	/* normalize Y values */
		d = 0.99f * ( hp.ns * hp.ns ) / d;
		K = 0.01f;
	} else {			/* or fall back on geometric Hessian */
		K = 1.0f;
		pg = NULL;
		dg = NULL;
		crlp = NULL;
	}
				/* relative Y channel from here on... */
	for (unsigned int i = 0; i < hp.ns; i++)
		for (unsigned int j = 0; j < hp.ns; j++) {
			ambsam(i, j).v.y = bright(ambsam(i, j).v) * d + K;
		}

	//if (uv == NULL)			/* make sure we have axis pointers */
	//	uv = my_uv;
					/* compute radii & pos. gradient */
	ambHessian( &hp, uv, ra, pg, normal, hit );

	if (dg != NULL)			/* compute direction gradient */
		ambdirgrad( &hp, uv->getRow(0), uv->getRow(1), dg, normal, hit );

	if (ra != NULL) {		/* scale/clamp radii */
		if (pg != NULL) {
			if ( ra->x * (d = fabsf( pg->x ) ) > 1.0f )
				ra->x = 1.0f / d;
			if ( ra->y * (d = fabsf( pg->y ) ) > 1.0f )
				ra->y = 1.0f / d;
			if ( ra->x > ra->y )
				ra->x = ra->y;
		}
		if ( ra->x < minarad ) {
			ra->x = minarad;
			if ( ra->y < minarad )
				ra->y = minarad;
		}
		*ra *= 1.0f / sqrtf( wt );
		if ( ra->y > 2.0f * ra->x )
			ra->y = 2.0f * ra->x;
		if ( ra->y > maxarad ) {
			ra->y = maxarad;
			if ( ra->x > maxarad )
				ra->x = maxarad;
		}
#ifdef CORRAL
					/* flag encroached directions */
		if (crlp != NULL)
			*crlp = ambcorral( &hp, uv, *ra * ambacc, hit );
#endif /* CORRAL */
		if (pg != NULL) {	/* cap gradient if necessary */
			d = pg->x*pg->x * ra->x*ra->x + pg->y*pg->y * ra->y*ra->y;
			if ( d > 1.0f ) {
				*pg *= 1.0f / sqrtf(d);
			}
		}
	}
#endif /* AMB_SAVE_MEM */
	//free(hp);			/* clean up and return */
	return(1);
}

/* sample indirect hemisphere, based on samp_hemi in ambcomp.c */
RT_METHOD int samp_hemi(
	AMBHEMI *hp,
	float3 *rcol,
	float wt,
#ifdef AMB_SAVE_MEM
	optix::Matrix<2, 3> *uv,
	float2 *ra,
	float2 *pg,
	float2 *dg,
	unsigned int *crlp,
#endif /* AMB_SAVE_MEM */
	const float3& normal,
	const float3& hit
)
{
					/* insignificance check */
	if (bright(*rcol) <= FTINY)
		return 0;
					/* set number of divisions */
#ifndef AMB_PARALLEL
	float d;
	if (ambacc <= FTINY && wt > (d = 0.8f * fmaxf(*rcol) * wt / (ambdiv*minweight))) //TODO second wt should be radiance ray weight
		wt = d;			/* avoid ray termination */
#endif
	unsigned int n = sqrtf(ambdiv * wt) + 0.5f;
	unsigned int i = 1 + 5 * (ambacc > FTINY);	/* minimum number of samples */
	if (n < i)
		n = i;
					/* allocate sampling array */
	hp->ns = n;
	hp->acol = make_float3( 0.0f );
	hp->sampOK = 0;
					/* assign coefficient */
	hp->acoef = *rcol * (1.0f / (n*n));
					/* make tangent plane axes */
#ifdef AMB_PARALLEL
	hp->ux = getperpendicular(normal);
#else
	hp->ux = getperpendicular( normal, prd.state );
#endif
	hp->uy = cross( normal, hp->ux );

#ifdef AMB_SAVE_MEM
	AmbientSample current, prev;

	/* ambHessian from ambcomp.c */
	optix::Matrix<3,3> hessian;
	float3 gradient = make_float3( 0.0f );
	hessian.setRow( 0, gradient ); // Set zero matrix
	hessian.setRow( 1, gradient );
	hessian.setRow( 2, gradient );
	optix::Matrix<3,3> hessianY;
	float3 gradientY = make_float3( 0.0f );
	hessianY.setRow( 0, gradientY ); // Set zero matrix
	hessianY.setRow( 1, gradientY );
	hessianY.setRow( 2, gradientY );

	FFTRI fftr;
					/* be sure to assign unit vectors */
	uv->setRow( 0, hp->ux );
	uv->setRow( 1, hp->uy );

	/* ambdirgrad from ambcomp.c */
	float3 dgsum = make_float3( 0.0f );	/* sum values times -tan(theta) */

#ifdef CORRAL
	/* ambcorral from ambcomp.c */
	const float max_d = 1.0f / ( minarad * ambacc + 0.001f );
	const float ang_res = M_PI_2f / hp->ns;
	const float ang_step = ang_res / ( (int)( 16.0f * M_1_PIf * ang_res ) + 1.01f );
	float avg_d = 0.0f;
	unsigned int corral_count = 0u;
#endif /* CORRAL */

					/* sample divisions */
	for ( i = 0; i < hp->ns; i++ ) {
		optix::Matrix<3,3> hesscol;	/* compute first vertical edge */
		float3 gradcol;

	    for (unsigned int j = 0; j < hp->ns; j++ ) {
			hp->sampOK += ambsample(hp, &current, i, j, 0, normal, hit);
			current.v.y = bright( current.v ); /* relative Y channel from here on... */

			/* ambHessian from ambcomp.c */
			if ( i ) {
				if ( j ) {
					optix::Matrix<3,3> hessdia;	/* compute triangle contributions */
					float3 graddia;
					optix::Matrix<3,3> hesstmp;
					float3 gradtmp;

					float backg = back_ambval( &prevrow(j - 1), &prevrow(j), &prev );
								/* diagonal (inner) edge */
					comp_fftri(&fftr, &prevrow(j), &prev, hit);
					hessdia = comp_hessian( &fftr, normal );
					hessian += ( hesstmp = hessrow(j - 1) + hessdia - hesscol );
					hessianY += backg * hesstmp;
					graddia = comp_gradient( &fftr, normal );
					gradient += ( gradtmp = gradrow(j - 1) + graddia - gradcol );
					gradientY += backg * gradtmp;
								/* initialize edge in next row */
					comp_fftri( &fftr, &current, &prev, hit );
					hessrow(j - 1) = comp_hessian( &fftr, normal );
					gradrow(j - 1) = comp_gradient( &fftr, normal );
								/* new column edge & paired triangle */
					backg = back_ambval( &current, &prev, &prevrow(j) );
					comp_fftri( &fftr, &prevrow(j), &current, hit );
					hesscol = comp_hessian( &fftr, normal );
					hessian += ( hesstmp = hessrow(j - 1) - hessdia + hesscol );
					hessianY += backg * hesstmp;
					gradcol = comp_gradient( &fftr, normal );
					gradient += ( gradtmp = gradrow(j - 1) - graddia + gradcol );
					gradientY += backg * gradtmp;
					if ( i < hp->ns-1 ) {
						hessrow(j - 1) *= -1.0f;
						gradrow(j - 1) = -gradrow(j - 1);
					}

#ifdef CORRAL
					/* ambcorral from ambcomp.c */
					if ( ( i < hp->ns * 3 / 4 ) && ( i >= hp->ns>>2 ) )
						if ( ( j < hp->ns * 3 / 4 ) && ( j >= hp->ns>>2 ) )
							avg_d += current.d;
#endif /* CORRAL */
				} else {
					comp_fftri(&fftr, &prevrow(0), &current, hit);
					hesscol = comp_hessian( &fftr, normal );
					gradcol = comp_gradient( &fftr, normal );
				}
			} else if ( j ) {
					/* compute first row of edges */
				comp_fftri( &fftr, &prev, &current, hit );
				hessrow(j - 1) = comp_hessian(&fftr, normal);
				gradrow(j - 1) = comp_gradient(&fftr, normal);
			}

			/* ambdirgrad from ambcomp.c */
					/* use vector for azimuth + 90deg */
			const float3 vd = current.p - hit;
					/* brightness over cosine factor */
			const float gfact = current.v.y / dot( normal, vd );
					/* sine = proj_radius/vd_length */
			dgsum += vd * gfact;

			if (j)
				prevrow(j - 1) = prev;
			else
				prevrow(hp->ns - 1) = prev;
			prev = current;

#ifdef CORRAL
			/* ambcorral from ambcomp.c */
			if ( !i || !j || i == hp->ns - 1 || j == hp->ns - 1 ) {
				if ( ( current.d <= FTINY ) | ( current.d >= max_d ) )
					continue;	/* too far or too near */
				corral_u(corral_count) = *uv * vd;
				corral_d(corral_count++) = current.d * current.d;
			}
#endif /* CORRAL */
		}
	}
#else /* AMB_SAVE_MEM */
					/* sample divisions */
	for (i = hp->ns; i--; )
		for (unsigned int j = hp->ns; j--;)
			hp->sampOK += ambsample(hp, &ambsam(i, j), i, j, 0, normal, hit);
#endif /* AMB_SAVE_MEM */
	*rcol = hp->acol;

	if (!hp->sampOK) {		/* utter failure? */
		return( 0 );
	}
	if (hp->sampOK < hp->ns * hp->ns) {
		hp->sampOK *= -1;	/* soft failure */
		return( 1 );
	}

#ifdef AMB_SAVE_MEM
	/* doambient from ambcomp.c */
	if ((d = bright(*rcol)) > FTINY) {	/* normalize Y values */
		d = 0.99f * ( hp->ns * hp->ns ) / d;
		hessian = d * hessianY + 0.01f * hessian;
		gradient = d * gradientY + 0.01f * gradient;
	} else {
		pg = dg = NULL;
		crlp = NULL;
	}

	/* ambHessian from ambcomp.c */
	optix::Matrix<2,2> ab;
	if ( ra )			/* extract eigenvectors & radii */
		ab = eigenvectors( uv, ra, &hessian );

	/* ambHessian from ambcomp.c */
	if ( pg )
		*pg = *uv * gradient;

	/* ambdirgrad from ambcomp.c */
	if ( dg ) {
		optix::Matrix<2,2> rotate;
		rotate[0] = rotate[3] = 0.0f;
		rotate[1] = -1.0f;
		rotate[2] = 1.0f;
		*dg = rotate * *uv * dgsum / (hp->ns*hp->ns);
	}

	/* ambcorral from ambcomp.c */
	if ( ra ) {
		if ( pg ) {
			if ( ra->x * (d = fabsf( pg->x ) ) > 1.0f )
				ra->x = 1.0f / d;
			if ( ra->y * (d = fabsf( pg->y ) ) > 1.0f )
				ra->y = 1.0f / d;
			if ( ra->x > ra->y )
				ra->x = ra->y;
		}
		if ( ra->x < minarad ) {
			ra->x = minarad;
			if ( ra->y < minarad )
				ra->y = minarad;
		}
		*ra *= 1.0f / sqrtf( prd.result.weight );
		if ( ra->y > 2.0f * ra->x )
			ra->y = 2.0f * ra->x;
		if ( ra->y > maxarad ) {
			ra->y = maxarad;
			if ( ra->x > maxarad )
				ra->x = maxarad;
		}

#ifdef CORRAL
		if ( crlp ) {
			unsigned int flgs = 0u;
			const float2 r = *ra * ambacc;
			avg_d *= 4.0f / ( hp->ns * hp->ns );
			if ( ( hp->ns >= 12 ) && ( avg_d * r.x < 1.0f )	&& ( avg_d < max_d ) ) {
						/* else circle around perimeter */
				for ( i = 0; i < corral_count; i++ ) {
					float2 u = ab * corral_u(i);
					if ( ( r.x*r.x * u.x*u.x + r.y*r.y * u.y*u.y ) * corral_d(i) <= dot( u, u ) )
						continue;	/* occluder outside ellipse */
					float ang = atan2f( u.y, u.x );	/* else set direction flags */
					for ( float a1 = ang - ang_res; a1 <= ang + ang_res; a1 += ang_step )
						flgs |= 1L<<(int)( 16.0f * M_1_PIf * ( a1 + 2.0f * M_PIf * ( a1 < 0.0f ) ) );
				}
				*crlp = flgs;
			}
		}
#endif /* CORRAL */

		if ( pg ) {	/* cap gradient if necessary */
			d = pg->x*pg->x * ra->x*ra->x + pg->y*pg->y * ra->y*ra->y;
			if ( d > 1.0f )
				*pg *= 1.0f / sqrtf(d);
		}
	}
#else /* AMB_SAVE_MEM */
#ifdef AMB_SUPER_SAMPLE
	if (hp->sampOK < 64)
		return(1);		/* insufficient for super-sampling */
	n = ambssamp * wt + 0.5f;
	if (n > 8) {			/* perform super-sampling? */
		ambsupersamp(hp, n, normal, hit);
		*rcol = hp->acol;
	}
#endif
#endif /* AMB_SAVE_MEM */

	return( 1 );			/* all is well */
}

RT_METHOD int ambsample(AMBHEMI *hp, AmbientSample *ap, const unsigned int& i, const unsigned int& j, const unsigned int& n, const float3& normal, const float3& hit)
{
#ifdef AMB_PARALLEL
	if (!n) {
		if (ap->d == -1.0f) // An exception occurred
			rtThrow((int)(ap->v.x) | RT_RETHROWN_EXCEPTION);
		if (ap->d == 0.0f) // No exception, but bad data
			return(0);

		ap->v *= hp->acoef;	/* apply coefficient */
		hp->acol += ap->v;	/* add to our sum */
#ifdef DAYSIM_COMPATIBLE
		DaysimCoef sample_dc = make_uint3(0, i + hp->ns * j, prd.dc.z);
		sample_dc = daysimNext(sample_dc); // Skip ahead one
		daysimAddScaled(prd.dc, sample_dc, hp->acoef.x);
#endif
#ifdef RAY_COUNT
		prd.result.ray_count += ap->ray_count;
#endif
#ifdef HIT_COUNT
		prd.result.hit_count += ap->hit_count;
#endif
		return(1);
	}
#endif /* AMB_PARALLEL */
#if defined AMB_SUPER_SAMPLE || !defined AMB_PARALLEL
	PerRayData_radiance new_prd;
	float b2;
					/* generate hemispherical sample */
					/* ambient coefficient for weight */
	if (ambacc > FTINY)
		b2 = AVGREFL; // Reusing this variable
	else
		b2 = fminf(fmaxf(hp->acoef), 1.0f);
	new_prd.weight = prd.result.weight * b2;
	if (new_prd.weight < minweight) //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		return(0);
	//if (ambacc > FTINY) {
	//	rcoef *= h->acoef;
	//	rcoef *= 1.0f / AVGREFL; // This all seems unnecessary
	//}
	//hlist[0] = hp->rp->rno;
	//hlist[1] = j;
	//hlist[2] = i;
	//multisamp(spt, 2, urand(ilhash(hlist,3)+n));
	float2 spt = make_float2( hiprand_uniform( prd.state ), hiprand_uniform( prd.state ) );
	if (!n && i > 0 && i < hp->ns - 1 && j > 0 && j < hp->ns - 1) /* avoid coincident samples */
		spt = 0.1f + 0.8f * spt;
	SDsquare2disk( spt, (j+spt.y) / hp->ns, (i+spt.x) / hp->ns );
	float zd = sqrtf( 1.0f - dot( spt, spt ) );
	float3 rdir = normalize( spt.x*hp->ux + spt.y*hp->uy + zd*normal );
	//dimlist[ndims++] = AI(hp,i,j) + 90171;

	new_prd.depth = prd.result.lvl + 1;//prd.depth + 1;
	new_prd.ambient_depth = prd.result.lvl + 1;//prd.ambient_depth + 1;
	new_prd.tmax = RAY_END;
	//new_prd.seed = prd.seed;//lcg( prd.seed );
	new_prd.state = prd.state;
#ifdef CONTRIB
#ifdef CONTRIB_DOUBLE
	new_prd.rcoef = make_contrib3(prd.result.weight * hp->acoef); //TODO This is not exact, but it's probably not used
#else
	new_prd.rcoef = prd.result.weight * hp->acoef; //TODO This is not exact, but it's probably not used
#endif
#endif
#ifdef ANTIMATTER
	new_prd.mask = 0u; //TODO check if we are in an antimatter volume
	new_prd.inside = 0;
#endif
#ifdef DAYSIM_COMPATIBLE
	new_prd.dc = daysimNext(prd.dc);
#if defined AMB_PARALLEL && defined AMB_SUPER_SAMPLE
	new_prd.dc = daysimNext(new_prd.dc); // Skip ahead one
#endif /* AMB_PARALLEL && AMB_SUPER_SAMPLE */
#endif /* DAYSIM_COMPATIBLE */
	setupPayload(new_prd);
	Ray amb_ray = make_Ray(hit, rdir, RADIANCE_RAY, ray_start(hit, rdir, normal, RAY_START), new_prd.tmax);
	rtTrace(top_object, amb_ray, new_prd);
#ifdef RAY_COUNT
	prd.result.ray_count += new_prd.ray_count;
#endif
#ifdef HIT_COUNT
	prd.result.hit_count += new_prd.hit_count;
#endif

	//ndims--;
	if ( isnan( new_prd.result ) ) // TODO How does this happen?
		return(0);
	if ( new_prd.distance <= FTINY )
		return(0);		/* should never happen */
	new_prd.result *= hp->acoef;	/* apply coefficient */
	if (!n || new_prd.distance * ap->d < 1.0f )		/* new/closer distance? */
		ap->d = 1.0f / new_prd.distance;
	if (!n) {			/* record first vertex & value */
		if (new_prd.distance > 50.0f * maxarad + 1000.0f) // 10 * thescene.cusize + 1000
			new_prd.distance = 50.0f * maxarad + 1000.0f;
		ap->p = hit + rdir * new_prd.distance;
		ap->v = new_prd.result; // only one AmbientSample, otherwise would need +=
#ifdef DAYSIM_COMPATIBLE
		daysimAddScaled(prd.dc, new_prd.dc, hp->acoef.x);
#endif
#ifdef AMB_SUPER_SAMPLE
	} else {			/* else update recorded value */
		hp->acol -= ap->v;
		zd = 1.0f / (n+1);
		ap->v *= n * zd;
		ap->v += new_prd.result * zd;
#ifdef DAYSIM_COMPATIBLE
#ifdef AMB_PARALLEL
		DaysimCoef sample_dc = make_uint3(0, i + hp->ns * j, prd.dc.z);
		sample_dc = daysimNext(sample_dc); // Skip ahead one
		daysimAddScaled(prd.dc, sample_dc, -hp->acoef.x);
		daysimRunningAverage(sample_dc, new_prd.dc, n);
		daysimAddScaled(prd.dc, sample_dc, hp->acoef.x);
#endif /* AMB_PARALLEL */
		// TODO Daysim compatible solution if not parallel ambient calculation
#endif /* DAYSIM_COMPATIBLE */
#endif /* AMB_SUPER_SAMPLE */
	}
	hp->acol += ap->v;	/* add to our sum */
#endif /* AMB_SUPER_SAMPLE || !AMB_PARALLEL */
	return(1);
}

#ifdef AMB_SUPER_SAMPLE
/* Estimate variance based on ambient division differences */
RT_METHOD void getambdiffs(AMBHEMI *hp)
{
	const float normf = 1.0f / bright(hp->acoef);

	/* compute squared neighbor diffs */
	for (unsigned int i = 0u; i < hp->ns; i++)
		for (unsigned int j = 0u; j < hp->ns; j++) {
			earr(i, j) = 0.0f;
			float b = bright(ambsam(i, j).v);
			if (i) {		/* from above */
				float b1 = bright(ambsam(i - 1, j).v);
				float d2 = b - b1;
				d2 *= d2 * normf / (b + b1);
				earr(i, j) += d2;
				earr(i - 1, j) += d2;
			}
			if (!j) continue;
			/* from behind */
			float b1 = bright(ambsam(i, j - 1).v);
			float d2 = b - b1;
			d2 *= d2 * normf / (b + b1);
			earr(i, j) += d2;
			earr(i, j - 1) += d2;
			if (!i) continue;
			/* diagonal */
			b1 = bright(ambsam(i - 1, j - 1).v);
			d2 = b - b1;
			d2 *= d2 * normf / (b + b1);
			earr(i, j) += d2;
			earr(i - 1, j - 1) += d2;
		}

	/* correct for number of neighbors */
	earr(0, 0) *= 8.0f / 3.0f;
	earr(0, hp->ns - 1) *= 8.0f / 3.0f;
	earr(hp->ns - 1, 0) *= 8.0f / 3.0f;
	earr(hp->ns - 1, hp->ns - 1) *= 8.0f / 3.0f;
	for (unsigned int i = 1u; i < hp->ns - 1; i++) {
		earr(i, 0) *= 8.0f / 5.0f;
		earr(i, hp->ns - 1) *= 8.0f / 5.0f;
		earr(0, i) *= 8.0f / 5.0f;
		earr(hp->ns - 1, i) *= 8.0f / 5.0f;
	}
}

/* Perform super-sampling on hemisphere (introduces bias) */
RT_METHOD void ambsupersamp(AMBHEMI *hp, int cnt, const float3& normal, const float3& hit)
{
	getambdiffs(hp);
	float e2rem = 0.0f;

	/* accumulate estimated variances */
	for (unsigned int i = hp->ns; i--; )
		for (unsigned int j = hp->ns; j--; )
			e2rem += earr(i, j);

	/* perform super-sampling */
	for (unsigned int i = 0u; i < hp->ns; i++)
		for (unsigned int j = 0u; j < hp->ns; j++) {
			if (e2rem <= FTINY)
				return;	/* nothing left to do */
			const float ep = earr(i, j);
			const int nss = ep / e2rem * cnt + hiprand_uniform(prd.state);
			for (int n = 1; n <= nss && ambsample(hp, &ambsam(i, j), i, j, n, normal, hit); n++)
				if (!--cnt) return;
			e2rem -= ep;		/* update remainder */
		}
}
#endif /* AMB_SUPER_SAMPLE */

/* Return brightness of farthest ambient sample */
RT_METHOD float back_ambval( const AmbientSample *n1, const AmbientSample *n2, const AmbientSample *n3 )
{
	if (n1->d <= n2->d) {
		if (n1->d <= n3->d)
			return(n1->v.y);
		return(n3->v.y);
	}
	if (n2->d <= n3->d)
		return(n2->v.y);
	return(n3->v.y);
}

/* Compute vectors and coefficients for Hessian/gradient calcs */
RT_METHOD void comp_fftri( FFTRI *ftp, const AmbientSample *n0, const AmbientSample *n1, const float3& hit )
{
	ftp->r_i = n0->p - hit;
	ftp->r_i1 = n1->p - hit;
	ftp->e_i = n1->p - n0->p;
	ftp->rcp = cross( ftp->r_i, ftp->r_i1 );
	const float rdot_cp = 1.0f / dot( ftp->rcp, ftp->rcp );
	const float dot_e = dot( ftp->e_i, ftp->e_i );
	const float dot_er = dot( ftp->e_i, ftp->r_i );
	const float rdot_r = 1.0f / dot( ftp->r_i, ftp->r_i );
	const float rdot_r1 = 1.0f / dot( ftp->r_i1, ftp->r_i1 );
	ftp->I1 = acosf(clamp(dot(ftp->r_i, ftp->r_i1) * sqrtf(rdot_r * rdot_r1), -1.0f, 1.0f)) * sqrtf(rdot_cp);
	ftp->I2 = ( dot( ftp->e_i, ftp->r_i1 ) * rdot_r1 - dot_er * rdot_r + dot_e * ftp->I1 ) * 0.5f * rdot_cp;
	const float J2 =  ( 0.5f * ( rdot_r - rdot_r1 ) - dot_er * ftp->I2 ) / dot_e;
	ftp->rI2_eJ2 = ftp->I2 * ftp->r_i + J2 * ftp->e_i;
}

/* Compose 3x3 matrix from two vectors */
RT_METHOD optix::Matrix<3,3> compose_matrix( const float3& va, const float3& vb )
{
	optix::Matrix<3,3> mat;
	mat.setRow( 0, va * vb.x + vb * va.x );
	mat.setRow( 1, va * vb.y + vb * va.y );
	mat.setRow( 2, va * vb.z + vb * va.z );
	//mat += mat.transpose();
	return mat;
}

/* Compute partial 3x3 Hessian matrix for edge */
RT_METHOD optix::Matrix<3,3> comp_hessian( FFTRI *ftp, const float3& normal )
{
					/* compute intermediate coefficients */
	float d1 = 1.0f / dot( ftp->r_i, ftp->r_i );
	float d2 = 1.0f / dot( ftp->r_i1, ftp->r_i1 );
	float d3 = 1.0f / dot( ftp->e_i, ftp->e_i );
	float d4 = dot( ftp->e_i, ftp->r_i );
	const float I3 = ( dot( ftp->e_i, ftp->r_i1 ) * d2 * d2 - d4 * d1 * d1 + 3.0f / d3 * ftp->I2 ) / ( 4.0f * dot( ftp->rcp, ftp->rcp ) );
	const float J3 = 0.25f * d3 * ( d1 * d1 - d2 * d2 ) - d4 * d3 * I3;
	const float K3 = d3 * ( ftp->I2 - I3 / d1 - 2.0f * d4 * J3);
					/* intermediate matrices */
	const float3 ncp = cross( normal, ftp->e_i );
	const optix::Matrix<3,3> m1 = compose_matrix( ncp, ftp->rI2_eJ2 );
	const optix::Matrix<3,3> m2 = compose_matrix( ftp->r_i, ftp->r_i );
	const optix::Matrix<3,3> m3 = compose_matrix( ftp->e_i, ftp->e_i );
	const optix::Matrix<3,3> m4 = compose_matrix( ftp->r_i, ftp->e_i );
	d1 = dot( normal, ftp->rcp );
	d2 = -d1 * ftp->I2;
	d1 *= 2.0f;
					/* final matrix sum */
	optix::Matrix<3,3> hess = m1 + d1 * ( I3 * m2 + K3 * m3 + 2.0f * J3 * m4 );
	hess += d2 * Matrix<3,3>::identity();
	hess *= -M_1_PIf;
	return hess;
}

/* Compute partial displacement form factor gradient for edge */
RT_METHOD float3 comp_gradient( FFTRI *ftp, const float3& normal )
{
	const float f1 = 2.0f * dot( normal, ftp->rcp );
	const float3 ncp = cross( normal, ftp->e_i );
	return ( 0.5f * M_1_PIf ) * ( ftp->I1 * ncp + f1 * ftp->rI2_eJ2 );
}

/* Compute anisotropic radii and eigenvector directions */
RT_METHOD optix::Matrix<2,2> eigenvectors( optix::Matrix<2,3> *uv, float2 *ra, optix::Matrix<3,3> *hessian )
{
					/* project Hessian to sample plane */
	const optix::Matrix<2,2> hess2 = *uv * *hessian * uv->transpose();
					/* compute eigenvalue(s) */
	float2 evalue;
	const unsigned int i = quadratic( &evalue, 1.0f, -hess2[0] - hess2[3], hess2[0] * hess2[3] - hess2[1] * hess2[2] );
	//if (i == 1u)			/* double-root (circle) */
	//	evalue.y = evalue.x;
	if (!i || ((evalue.x = fabsf(evalue.x)) <= FTINY*FTINY) | ((evalue.y = fabsf(evalue.y)) <= FTINY*FTINY) ) {
		*ra = make_float2( maxarad );
		return optix::Matrix<2,2>::identity();
	}
	float slope1;
	if ( evalue.x > evalue.y ) {
		*ra = sqrtf( sqrtf ( 4.0f / evalue ) );
		slope1 = evalue.y;
	} else {
		*ra = make_float2( sqrtf( sqrtf ( 4.0f / evalue.y ) ), sqrtf( sqrtf ( 4.0f / evalue.x ) ) );
		slope1 = evalue.x;
	}
					/* compute unit eigenvectors */
	if ( fabsf( hess2[1] ) <= FTINY )
		return optix::Matrix<2,2>::identity();			/* uv OK as is */
	slope1 = ( slope1 - hess2[0] ) / hess2[1];
	const float xmag1 = sqrtf( 1.0f / ( 1.0f + slope1 * slope1 ) );
	optix::Matrix<2,2> ab;
	ab[0] = ab[3] = slope1 * xmag1;
	ab[1] = -xmag1;
	ab[2] = xmag1;
	*uv = ab * *uv;

	/* needed for ambcorral */
	return ab;
}

#ifndef AMB_SAVE_MEM
RT_METHOD void ambHessian( AMBHEMI *hp, optix::Matrix<2,3> *uv, float2 *ra, float2 *pg, const float3& normal, const float3& hit )
{
	optix::Matrix<3,3> hessian;
	float3 gradient = make_float3( 0.0f );
	hessian.setRow( 0, gradient ); // Set zero matrix
	hessian.setRow( 1, gradient );
	hessian.setRow( 2, gradient );
	FFTRI fftr;
					/* be sure to assign unit vectors */
	uv->setRow( 0, hp->ux );
	uv->setRow( 1, hp->uy );
			/* clock-wise vertex traversal from sample POV */
	//if (ra != NULL) {		/* initialize Hessian row buffer */
	//	hessrow = (FVECT (*)[3])malloc(sizeof(FVECT)*3*(hp->ns-1)); //TODO set memory size
	//	if (hessrow == NULL)
	//		error(SYSTEM, memerrmsg);
	//	memset(hessian, 0, sizeof(hessian));
	//} else if (pg == NULL)		/* bogus call? */
	//	return;
	//if (pg != NULL) {		/* initialize form factor row buffer */
	//	gradrow = (FVECT *)malloc(sizeof(FVECT)*(hp->ns-1));
	//	if (gradrow == NULL)
	//		error(SYSTEM, memerrmsg);
	//	memset(gradient, 0, sizeof(gradient));
	//}
					/* compute first row of edges */
	for (unsigned int j = 0; j < hp->ns-1; j++) {
		comp_fftri(&fftr, &ambsam(0, j), &ambsam(0, j + 1), hit);
		if (ra != NULL)
			hessrow(j) = comp_hessian( &fftr, normal );
		if (pg != NULL)
			gradrow(j) = comp_gradient( &fftr, normal );
	}
					/* sum each row of triangles */
	for (unsigned int i = 0; i < hp->ns - 1; i++) {
	    optix::Matrix<3,3> hesscol;	/* compute first vertical edge */
	    float3 gradcol;
		comp_fftri(&fftr, &ambsam(i, 0), &ambsam(i + 1, 0), hit);
		if (ra != NULL)
			hesscol = comp_hessian( &fftr, normal );
		if (pg != NULL)
			gradcol = comp_gradient( &fftr, normal );
		for (unsigned int j = 0; j < hp->ns - 1; j++) {
			optix::Matrix<3,3> hessdia;	/* compute triangle contributions */
			float3 graddia;
			float backg = back_ambval(&ambsam(i, j), &ambsam(i, j + 1), &ambsam(i + 1, j));
						/* diagonal (inner) edge */
			comp_fftri(&fftr, &ambsam(i, j + 1), &ambsam(i + 1, j), hit);
			if (ra != NULL) {
				hessdia = comp_hessian( &fftr, normal );
				//hesscol = -hesscol;
				hessian += backg * ( hessrow(j) + hessdia - hesscol );
			}
			if (pg != NULL) {
				graddia = comp_gradient( &fftr, normal );
				//gradcol = -gradcol;
				gradient += backg * ( gradrow(j) + graddia - gradcol );
			}
						/* initialize edge in next row */
			comp_fftri(&fftr, &ambsam(i + 1, j + 1), &ambsam(i + 1, j), hit);
			if (ra != NULL)
				hessrow(j) = comp_hessian( &fftr, normal );
			if (pg != NULL)
				gradrow(j) = comp_gradient( &fftr, normal );
						/* new column edge & paired triangle */
			backg = back_ambval(&ambsam(i + 1, j + 1), &ambsam(i + 1, j), &ambsam(i, j + 1));
			comp_fftri(&fftr, &ambsam(i, j + 1), &ambsam(i + 1, j + 1), hit);
			if (ra != NULL) {
				hesscol = comp_hessian( &fftr, normal );
				//hessdia = -hessdia;
				hessian += backg * ( hessrow(j) - hessdia + hesscol );
				if ( i < hp->ns-2 )
					hessrow(j) *= -1.0f;
			}
			if (pg != NULL) {
				gradcol = comp_gradient( &fftr, normal );
				//graddia = -graddia;
				gradient += backg * ( gradrow(j) - graddia + gradcol );
				if ( i < hp->ns-2 )
					gradrow(j) = -gradrow(j);
			}
	    }
	}
					/* release row buffers */
	//if (hessrow != NULL) free(hessrow);
	//if (gradrow != NULL) free(gradrow);
	
	if (ra != NULL)			/* extract eigenvectors & radii */
		eigenvectors( uv, ra, &hessian );
	if (pg != NULL) {		/* tangential position gradient */
		*pg = *uv * gradient;
	}
}

/* Compute direction gradient from a hemispherical sampling */
RT_METHOD void ambdirgrad( AMBHEMI *hp, const float3& u, const float3& v, float2 *dg, const float3& normal, const float3& hit )
{
	float2 dgsum = make_float2( 0.0f );	/* sum values times -tan(theta) */
	for (unsigned int i = 0; i < hp->ns; i++)
		for (unsigned int j = 0; j < hp->ns; j++) {
			const AmbientSample ap = ambsam(i, j);
					/* use vector for azimuth + 90deg */
			float3 vd = ap.p - hit;
					/* brightness over cosine factor */
			float gfact = dot(normal, vd);
			if (gfact < FTINY)
				gfact = FTINY;
			gfact = ap.v.y / gfact;
					/* sine = proj_radius/vd_length */
			dgsum.x -= dot( v, vd ) * gfact;
			dgsum.y += dot( u, vd ) * gfact;
		}
	*dg = dgsum / (hp->ns*hp->ns);
}

/* Compute potential light leak direction flags for cache value */
RT_METHOD unsigned int ambcorral( AMBHEMI *hp, optix::Matrix<2,3> *uv, const float2& r, const float3& hit )
{
	const float max_d = 1.0f / ( minarad * ambacc + 0.001f );
	const float ang_res = M_PI_2f / hp->ns;
	const float ang_step = ang_res / ( (int)( 16.0f * M_1_PIf * ang_res ) + ( 1.01f ) );
	float avg_d = 0.0f;
	unsigned int flgs = 0u;
					/* don't bother for a few samples */
	if (hp->ns < 8)
		return(0u);
					/* check distances overhead */
	for (unsigned int i = hp->ns * 3 / 4; i-- > hp->ns >> 2;)
		for (unsigned int j = hp->ns * 3 / 4; j-- > hp->ns >> 2;)
			avg_d += ambsam(i, j).d;
	avg_d *= 4.0f / ( hp->ns * hp->ns );
	if ( avg_d * r.x >= 1.0f )		/* ceiling too low for corral? */
		return(0u);
	if ( avg_d >= max_d )		/* insurance */
		return(0u);
					/* else circle around perimeter */
	for (unsigned int i = 0; i < hp->ns; i++)
		for (unsigned int j = 0; j < hp->ns; j += !i | (i == hp->ns - 1) ? 1 : hp->ns - 1) {
			const AmbientSample ap = ambsam(i, j);
			if ( ( ap.d <= FTINY ) | ( ap.d >= max_d ) )
				continue;	/* too far or too near */
			const float2 u = *uv * ( ap.p - hit );
			if ( ( r.x*r.x * u.x*u.x + r.y*r.y * u.y*u.y ) * ap.d*ap.d <= u.x*u.x + u.y*u.y )
				continue;	/* occluder outside ellipse */
			const float ang = atan2f( u.y, u.x );	/* else set direction flags */
			for ( float a1 = ang - ang_res; a1 <= ang + ang_res; a1 += ang_step )
				flgs |= 1L<<(int)( 16.0f * M_1_PIf * ( a1 + 2.0f * M_PIf * ( a1 < 0.0f ) ) );
	    }
	return(flgs);
}
#endif /* AMB_SAVE_MEM */

/* convert 1-dimensional sample to 2 dimensions, based on multisamp.c */
//RT_METHOD float2 multisamp2(float r)	/* 1-dimensional sample [0,1) */
//{
//	int	j;
//	register int	k;
//	int2	ti;
//	float	s;
//
//	ti = make_int2( 0 );
//	j = 8;
//	while (j--) {
//		k = s = r*(1<<2);
//		r = s - k;
//		ti += ti + make_int2( ((k>>2) & 1), ((k>>1) & 1) );
//	}
//	ti += make_int2( frandom() );
//	ti *= 1.0f/256.0f;
//}

/* hash a set of integer values */
//RT_METHOD int ilhash(int3 d)
//{
//	register int  hval;
//
//	hval = 0;
//	hval ^= d.x * 73771;
//	hval ^= d.y * 96289;
//	hval ^= d.z * 103699;
//	return(hval & 0x7fffffff);
//}
