#include "hip/hip_runtime.h"
/*
 *  rpict_cloud_generator.cu - entry point for geometry sampling for image generation on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_common.h"
#include "optix_point_common.h"

#define RING_BUFFER_SIZE	8

using namespace optix;

/* Contex variables */
rtDeclareVariable(unsigned int,  camera, , ); /* Camera type (-vt) */
rtDeclareVariable(float3,        eye, , ); /* Eye position (-vp) */
rtDeclareVariable(float3,        U, , ); /* view.hvec */
rtDeclareVariable(float3,        V, , ); /* view.vvec */
rtDeclareVariable(float3,        W, , ); /* view.vdir */
rtDeclareVariable(float2,        fov, , ); /* Field of view (-vh, -vv) */
rtDeclareVariable(float2,        shift, , ); /* Camera shift (-vs, -vl) */
rtDeclareVariable(float2,        clip, , ); /* Fore and aft clipping planes (-vo, -va) */
rtDeclareVariable(float,         dstrpix, , ); /* Pixel sample jitter (-pj) */

rtBuffer<PointDirection, 3>      seed_buffer;
rtDeclareVariable(rtObject,      top_object, , );

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

RT_METHOD float3 uniform_solid_angle( float2 in )
{
	float2 d = 2.0f * in - 1.0f; // map input from [0, 1] to [-1, 1]
	float z = d.y > 0.0f ? 1.0f : -1.0f;
	if ( ( fabsf( d.x ) < FTINY ) && ( fabsf( d.y ) < FTINY ) )
		return make_float3( 0.0f, 0.0f, z );

	d.y = 2.0f * d.y - z;
	float s, x, y;
	if ( fabsf( d.x ) > fabsf( d.y ) ) {
		float angle = M_PI_4f * d.y / d.x;
		s = d.x;
		x = cosf( angle );
		y = sinf( angle );
	} else {
		float angle = M_PI_4f * d.x / d.y;
		s = d.y;
		x = sinf( angle );
		y = cosf( angle );
	}
	float r = s * sqrtf( 2.0f - s * s );
	return make_float3( r * x, r * y, z - z * s * s );
}

RT_PROGRAM void point_cloud_camera()
{
	PerRayData_point_cloud prd;

	// Init random state
	init_rand(&prd.state, launch_index.x + launch_dim.x * launch_index.y);

	prd.index = make_uint3(launch_index, 0u);
	prd.seeds = seed_buffer.size().z;
	unsigned int loop = 2u * prd.seeds; // Prevent infinite looping

	float3 point_ring[RING_BUFFER_SIZE];
	float3 dir_ring[RING_BUFFER_SIZE];
	unsigned int ring_start = 0, ring_end = 0, ring_full = 0;

	Ray ray;
	ray.origin = eye;
	ray.ray_type = POINT_CLOUD_RAY;
	ray.tmin = 0.0f;

	float2 d = make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
	d = 0.5f + dstrpix * ( 0.5f - d ); // this is pixjitter() from rpict.c

	// Set initial ray direction
	if ( camera ) { // using the camera viewport
		d = shift + ( make_float2( launch_index ) + d ) / make_float2( launch_dim ) - 0.5f;
		float z = 1.0f;

		// This is adapted from viewray() in image.c.
  		if( camera == VT_PAR ) { /* parallel view */
			ray.origin += d.x*U + d.y*V;
			d = make_float2( 0.0f );
		} else if ( camera == VT_HEM ) { /* hemispherical fisheye */
			z = 1.0f - d.x*d.x * dot( U, U ) - d.y*d.y * dot( V, V );
			if (z < 0.0f)
				goto clearout;
			z = sqrtf(z);
		} else if ( camera == VT_CYL ) { /* cylindrical panorama */
			float dd = d.x * fov.x * ( M_PIf / 180.0f );
			z = cosf( dd );
			d.x = sinf( dd );
		} else if ( camera == VT_ANG ) { /* angular fisheye */
			d *= fov / 180.0f;
			float dd = length(d);
			if (dd > 1.0f)
				goto clearout;
			z = cosf( M_PIf * dd );
			d *= dd < FTINY ? M_PIf : sqrtf(1.0f - z*z) / dd;
		} else if ( camera == VT_PLS ) { /* planispheric fisheye */
			d *= make_float2(length(U), length(V));
			float dd = dot( d, d );
			z = ( 1.0f - dd ) / ( 1.0f + dd );
			d *= 1.0f + z;
		}

		ray.direction = d.x*U + d.y*V + z*W;
		ray.direction += clip.x * ray.direction;
		ray.direction = normalize(ray.direction);

		// Zero or negative aft clipping distance indicates infinity
		ray.tmax = clip.y - clip.x;
		if (ray.tmax <= FTINY) {
			ray.tmax = RAY_END;
		}
	} else { // using a sphere with equal solid angle divisions
		d = ( make_float2( launch_index ) + d ) / make_float2( launch_dim );// - 0.5f;

		// Get the position and normal of the first ray
		ray.direction = uniform_solid_angle(d);
		ray.tmax = RAY_END;
	}

	while (prd.index.z < prd.seeds && loop--) {
		prd.forward = prd.reverse = make_float3(0.0f);
#ifdef ANTIMATTER
		prd.mask = 0u;
		prd.inside = 0;
#endif

		// Trace the current ray
		rtTrace(top_object, ray, prd);

		// Add next forward ray to ring buffer
		if (isfinite(prd.point) && dot(prd.forward, prd.forward) > FTINY) { // NaN values will be false
			point_ring[ring_end] = prd.point;
			dir_ring[ring_end] = prd.forward;
			ring_end = (ring_end + 1) % RING_BUFFER_SIZE;
			ring_full = ring_start == ring_end;
		}

		// Add next reverse ray to ring buffer
		if (!ring_full && isfinite(prd.point) && dot(prd.reverse, prd.reverse) > FTINY) { // NaN values will be false
			point_ring[ring_end] = prd.point;
			dir_ring[ring_end] = prd.reverse;
			ring_end = (ring_end + 1) % RING_BUFFER_SIZE;
			ring_full = ring_start == ring_end;
		}

		if (!ring_full && ring_start == ring_end)
			break;

		// Prepare for next ray
		ray.origin = point_ring[ring_start];
		ray.direction = dir_ring[ring_start];
		ring_start = (ring_start + 1) % RING_BUFFER_SIZE;
		ring_full = 0;
		ray.tmin = ray_start(ray.origin, RAY_START);
		ray.tmax = RAY_END;
	}

clearout:
	// If outdoors, there are no bounces, but we need to prevent junk data
	while (prd.index.z < prd.seeds) {
		clear(seed_buffer[prd.index]);
		prd.index.z++;
	}
}

RT_PROGRAM void exception()
{
#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
	uint3 index = make_uint3(launch_index, seed_buffer.size().z - 1u); // record error to last segment
	seed_buffer[index].pos = exceptionToFloat3(rtGetExceptionCode());
	seed_buffer[index].dir = make_float3( 0.0f );
#ifdef AMBIENT_CELL
	seed_buffer[index].cell = make_uint2(0);
#endif
}
