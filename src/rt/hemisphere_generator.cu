#include "hip/hip_runtime.h"
/*
 *  hemisphere_generator.cu - entry point for geometry sampling after the first bounce on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_common.h"
#include "optix_point_common.h"

using namespace optix;

/* Contex variables */
//rtBuffer<PointDirection, 1>      cluster_buffer; /* input */
rtDeclareVariable(PointDirectionBuffer, cluster_buffer, , ); /* input */
rtBuffer<PointDirection, 3>      seed_buffer; /* output */
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  segment_offset, , ) = 0u; /* Offset into data if computed with multiple segments */

/* OptiX variables */
rtDeclareVariable(uint3, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint3, launch_dim,   rtLaunchDim, );

RT_PROGRAM void hemisphere_camera()
{
	PerRayData_point_cloud prd;
	clear(seed_buffer[launch_index]);

	PointDirection eye = cluster_buffer[launch_index.z + segment_offset];

	// Check for valid input
	if ( isfinite( eye.pos ) && isfinite( eye.dir ) && dot( eye.dir, eye.dir ) > FTINY ) { // NaN values will be false
		// Init random state
		init_rand(&prd.state, launch_index.x + launch_dim.x * (launch_index.y + launch_dim.y * launch_index.z));

		// Make axes
		float3 uz = normalize(eye.dir);
		float3 ux = getperpendicular(uz, prd.state);
		float3 uy = cross(uz, ux);
						/* avoid coincident samples */
		float2 spt = 0.1f + 0.8f * make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
		SDsquare2disk(spt, (launch_index.y + spt.y) / launch_dim.y, (launch_index.x + spt.x) / launch_dim.x);
		float zd = sqrtf(1.0f - dot(spt, spt));
		float3 rdir = normalize(spt.x * ux + spt.y * uy + zd * uz);

		prd.index = launch_index;
		prd.seeds = launch_index.z + 1;
#ifdef ANTIMATTER
		prd.mask = 0u;
		prd.inside = 0;
#endif

		// Trace the current ray
		Ray ray = make_Ray(eye.pos, rdir, POINT_CLOUD_RAY, ray_start( eye.pos, rdir, uz, RAY_START ), RAY_END);
		rtTrace(top_object, ray, prd);
	}
}

RT_PROGRAM void exception()
{
#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
	seed_buffer[launch_index].pos = exceptionToFloat3(rtGetExceptionCode());
	seed_buffer[launch_index].dir = make_float3( 0.0f );
#ifdef AMBIENT_CELL
	seed_buffer[launch_index].cell = make_uint2(0);
#endif
}
