#include "hip/hip_runtime.h"
/*
 *  ambient_sample_generator.cu - entry point for ambient sampling on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_ray.h"
#include "optix_ambient_common.h"
#include "optix_point_common.h"
#ifdef CONTRIB_DOUBLE
#include "optix_double.h"
#endif

#ifdef AMB_PARALLEL

using namespace optix;

/* Contex variables */
//rtBuffer<PointDirection, 1>     cluster_buffer; /* input */
rtDeclareVariable(PointDirectionBuffer, cluster_buffer, , ); /* input */
rtBuffer<AmbientSample, 3>      amb_samp_buffer; /* ambient sample output */
rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(unsigned int, level, , ) = 0u;
#ifdef DAYSIM_COMPATIBLE
rtDeclareVariable(unsigned int, segment_offset, , ) = 0u; /* Offset into data if computed with multiple segments */
#endif /* DAYSIM_COMPATIBLE */

//rtDeclareVariable(float,        ambacc, , ); /* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(float,        maxarad, , ); /* maximum ambient radius */

/* OptiX variables */
rtDeclareVariable(uint3, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint3, launch_dim,   rtLaunchDim, );


RT_PROGRAM void ambient_sample_camera()
{
	uint3 index = launch_index;
#ifdef DAYSIM_COMPATIBLE
	index.z += segment_offset;
#endif /* DAYSIM_COMPATIBLE */
	PointDirection cluster = cluster_buffer[index.z];

	PerRayData_radiance prd;
	init_rand(&prd.state, launch_index.x + launch_dim.x * (launch_index.y + launch_dim.y * (launch_index.z + launch_dim.z * level)));
	float b2;
					/* generate hemispherical sample */
					/* ambient coefficient for weight */
//	if (ambacc > FTINY)
		b2 = AVGREFL; // Reusing this variable
//	else
//		b2 = fmaxf(hp->acoef); //TODO

	prd.weight = b2;
	for ( int i = level; i--; )
		prd.weight *= AVGREFL; // Compute weight as in makeambient() from ambient.c

	if (prd.weight < minweight) { //if (rayorigin(&ar, AMBIENT, r, ar.rcoef) < 0)
		amb_samp_buffer[index].d = 0.0f;
		return;
	}
	//if (ambacc > FTINY) {
	//	rcoef *= h->acoef;
	//	rcoef *= 1.0f / AVGREFL; // This all seems unnecessary
	//}
	//hlist[0] = hp->rp->rno;
	//hlist[1] = j;
	//hlist[2] = i;
	//multisamp(spt, 2, urand(ilhash(hlist,3)+n));
	float3 ux = getperpendicular(cluster.dir); // Can't be random, must be same for all threads for this point
	float3 uy = cross(cluster.dir, ux);
					/* avoid coincident samples */
	float2 spt = 0.1f + 0.8f * make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
	SDsquare2disk(spt, (launch_index.y + spt.y) / launch_dim.y, (launch_index.x + spt.x) / launch_dim.x);
	float zd = sqrtf(1.0f - dot(spt, spt));
	float3 rdir = normalize(spt.x * ux + spt.y * uy + zd * cluster.dir);
	//dimlist[ndims++] = AI(hp,i,j) + 90171;

	prd.depth = level + 1;//prd.depth + 1;
	prd.ambient_depth = level + 1;//prd.ambient_depth + 1;
	prd.tmax = RAY_END;
#ifdef CONTRIB
	prd.rcoef = make_contrib3(prd.weight); //Probably not necessary
#endif
#ifdef ANTIMATTER
	prd.mask = 0u; //TODO this assumes we are not inside an antimatter volume
	prd.inside = 0;
#endif
#ifdef DAYSIM_COMPATIBLE
	prd.dc = make_uint3(0, launch_index.x + launch_dim.x * launch_index.y, launch_index.z);
	prd.dc = daysimNext(prd.dc); // Skip ahead one
	daysimSet(prd.dc, 0.0f);
#endif
	setupPayload(prd);
	Ray ray = make_Ray(cluster.pos, rdir, RADIANCE_RAY, ray_start(cluster.pos, rdir, cluster.dir, RAY_START), prd.tmax);
	rtTrace(top_object, ray, prd);
#ifdef RAY_COUNT
	amb_samp_buffer[index].ray_count = prd.ray_count;
#endif
#ifdef HIT_COUNT
	amb_samp_buffer[index].hit_count = prd.hit_count;
#endif

	//ndims--;
	checkFinite(prd.result);
	if (prd.distance <= FTINY) {
		amb_samp_buffer[index].d = 0.0f;
		return;
	}

	//if ( new_prd.distance * ap->d < 1.0f )		/* new/closer distance? */ //TODO where did this value come from?
		amb_samp_buffer[index].d = 1.0f / prd.distance;
	//if (!n) {			/* record first vertex & value */
		if (prd.distance > 50.0f * maxarad + 1000.0f) // 10 * thescene.cusize + 1000
			prd.distance = 50.0f * maxarad + 1000.0f;
		amb_samp_buffer[index].p = cluster.pos + rdir * prd.distance;
		amb_samp_buffer[index].v = prd.result; // only one AmbientSample, otherwise would need +=
	//} else {			/* else update recorded value */
	//	hp->acol -= ap->v;
	//	zd = 1.0f / (float)(n+1);
	//	prd.result *= zd;
	//	zd *= (float)n;
	//	ambient_sample_buffer[index].v *= zd;
	//	ambient_sample_buffer[index].v += new_prd.result;
	//}
}

RT_PROGRAM void exception()
{
#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
	uint3 index = launch_index;
#ifdef DAYSIM_COMPATIBLE
	index.z += segment_offset;
#endif /* DAYSIM_COMPATIBLE */
	amb_samp_buffer[index].d = -1.0f;
	amb_samp_buffer[index].v = amb_samp_buffer[index].p = exceptionToFloat3(rtGetExceptionCode());
#ifdef RAY_COUNT
	amb_samp_buffer[index].ray_count = 0;
#endif
#ifdef HIT_COUNT
	amb_samp_buffer[index].hit_count = 0;
#endif
}

#endif /* AMB_PARALLEL */
