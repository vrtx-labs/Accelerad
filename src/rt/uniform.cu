#include "hip/hip_runtime.h"
/*
 *  uniform.cu - program for uniform sampling of sky and other surfaces on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>

rtDeclareVariable(float3, normal, , );	/* Normal direction */

// Return 0 if the surface is hit from the front, -1 otherwise.
RT_CALLABLE_PROGRAM int front(const float3 direction)
{
	return optix::dot(direction, normal) < 0 ? 0 : -1;
}