#include "hip/hip_runtime.h"
/*
 *  material_light.cu - hit programs for light materials on GPUs.
 */

#include "accelerad_copyright.h"

#include "otypes.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_ray.h"


using namespace optix;

/* Context variables */
rtDeclareVariable(int,          directvis, , );		/* Boolean switch for light source visibility (dv) */


RT_METHOD int spotout(const IntersectData &data);

/* wrongsource *
 *
 * This source is the wrong source (ie. overcounted) if we are
 * aimed to a different source than the one we hit and the one
 * we hit is not an illum that should be passed.
 */

#define  wrongsource(prd_shadow, data)	(data.surface_id != -prd_shadow.target - 1)// && \
//				(data.mat.type != MAT_ILLUM || illumblock(m,r)))

/* distglow *
 *
 * A distant glow is an object that sometimes acts as a light source,
 * but is too far away from the test point to be one in this case.
 * (Glows with negative radii should NEVER participate in illumination.)
 */
#define  distglow(data)	(data.mat.type == MAT_GLOW && \
				data.mat.params.l.maxrad >= -FTINY && \
				data.t > data.mat.params.l.maxrad)

/* badcomponent *
 *
 * We must avoid counting light sources in the ambient calculation,
 * since the direct component is handled separately.  Therefore, any
 * ambient ray which hits an active light source must be discarded.
 * The same is true for stray specular samples, since the specular
 * contribution from light sources is calculated separately.
 */
#define  badcomponent(prd, data)   ((prd.ambient_depth > 0 && \
				!(dot(data.world_shading_normal, data.ray_direction) > 0.0f || \
		/* not 100% correct */	distglow(data))))

/* srcignore *
 *
 * The -dv flag is normally on for sources to be visible. Not for shadow rays.
 */
#define  srcignore(prd, data)	!(directvis || (distglow(data) && !prd.depth))

RT_CALLABLE_PROGRAM PerRayData_shadow closest_hit_light_shadow(IntersectData const&data, PerRayData_shadow prd_shadow)
{
	if (wrongsource(prd_shadow, data) || dot(data.world_shading_normal, data.ray_direction) > 0.0f || spotout(data)) {
		prd_shadow.result = make_float3(0.0f);
#ifdef CONTRIB
		prd_shadow.rcoef = make_contrib3(0.0f);
#endif
	}
	else if (data.mat.params.l.function > RT_PROGRAM_ID_NULL)
		prd_shadow.result = data.mat.color * ((rtCallableProgramId<float3(const float3, const float3)>)data.mat.params.l.function)(data.ray_direction, data.world_shading_normal);
	else
		prd_shadow.result = data.mat.color;
	return prd_shadow;
}

RT_CALLABLE_PROGRAM PerRayData_radiance closest_hit_light_radiance(IntersectData const&data, PerRayData_radiance prd)
{
	// no contribution to ambient calculation
	if (badcomponent(prd, data) || srcignore(prd, data)) {
		prd.result = make_float3(0.0f);
#ifdef CONTRIB
		prd.rcoef = make_contrib3(0.0f);
#endif
	}
	else if (dot(data.world_shading_normal, data.ray_direction) > 0.0f || spotout(data))
		prd.result = make_float3(0.0f);
	else if (data.mat.params.l.function > RT_PROGRAM_ID_NULL)
		prd.result = data.mat.color * ((rtCallableProgramId<float3(const float3, const float3)>)data.mat.params.l.function)(data.ray_direction, data.world_shading_normal);
	else
		prd.result = data.mat.color;
	prd.mirror = make_float3(0.0f);
	return prd;
}

RT_METHOD int spotout(const IntersectData &data)
{
	if (data.mat.type != MAT_SPOT)
		return(0); /* Not a spotlight */
	if (data.mat.params.l.flen < -FTINY) {		/* distant source */
		const float3 ray_origin = data.hit - data.t * data.ray_direction;
		const float3 vd = data.mat.params.l.aim - ray_origin;
		float d = dot(data.ray_direction, vd);
		/*			wrong side?
		if (d <= FTINY)
			return(1);	*/
		d = dot( vd, vd ) - d * d;
		return (M_PIf * d > data.mat.params.l.siz); /* If true then out */
	}
					/* local source */
	return (data.mat.params.l.siz < 2.0f * M_PIf * (1.0f + dot(data.mat.params.l.aim, data.ray_direction)));	/* If true then out */
}
