#include "hip/hip_runtime.h"
/*
 *  rtrace.cu - entry point for individual ray tracing on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_ray.h"
#ifdef CONTRIB_DOUBLE
#include "optix_double.h"
#endif

using namespace optix;

/* Contex variables */
rtBuffer<RayData, 2>             ray_buffer;
#ifdef DAYSIM_COMPATIBLE
rtBuffer<DC, 3>                  dc_buffer;
#endif
//rtBuffer<unsigned int, 2>        rnd_seeds;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(rtObject,      top_irrad, , );
rtDeclareVariable(unsigned int,  imm_irrad, , ) = 0u; /* Immediate irradiance (-I) */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

//#define TIME_VIEW


RT_PROGRAM void ray_generator()
{
#ifdef TIME_VIEW
	clock_t t0 = clock();
	ray_buffer[launch_index].val = make_float3( t0 );
#endif
	PerRayData_radiance prd;
	init_rand(&prd.state, launch_index.x + launch_dim.x * launch_index.y);
	prd.result = prd.mirror = make_float3(0.0f);
	prd.distance = prd.mirror_distance = RAY_END;
	prd.weight = 1.0f;
	prd.depth = 0;
	prd.ambient_depth = 0;
	//prd.seed = rnd_seeds[launch_index];
#ifdef CONTRIB
	prd.rcoef = make_contrib3(1.0f); //Probably not necessary
#endif
#ifdef ANTIMATTER
	prd.mask = 0u;
	prd.inside = 0;
#endif
#ifdef DAYSIM_COMPATIBLE
	prd.dc = make_uint3(0, launch_index.x, launch_index.y);
#endif
	setupPayload(prd);

	const float tmin = ray_start( ray_buffer[launch_index].origin, RAY_START );
	if ( imm_irrad ) {
		prd.tmax = 2.0f * tmin;
		Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, RADIANCE_RAY, 0.0f, prd.tmax);
		rtTrace(top_irrad, ray, prd);
	} else {
		// Zero or negative aft clipping distance indicates infinity
		prd.tmax = ray_buffer[launch_index].max;
		if (prd.tmax <= FTINY) {
			prd.tmax = RAY_END;
		}

		Ray ray = make_Ray(ray_buffer[launch_index].origin, ray_buffer[launch_index].dir, RADIANCE_RAY, tmin, prd.tmax);
		rtTrace(top_object, ray, prd);
	}

	checkFinite(prd.result);

#ifdef TIME_VIEW
	clock_t t1 = clock();
 
	float expected_fps   = 1.0f;
	float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
	ray_buffer[launch_index].val = make_float3( pixel_time );
#else
	ray_buffer[launch_index].val = prd.result;
#endif
	ray_buffer[launch_index].length = prd.distance;
	ray_buffer[launch_index].mirror = prd.mirror;
	ray_buffer[launch_index].mirrored_length = prd.mirror_distance;
	//ray_buffer[launch_index].hit = ray_buffer[launch_index].origin + prd.distance * ray_buffer[launch_index].dir;
	ray_buffer[launch_index].weight = prd.weight;
	//ray_buffer[launch_index].t = prd.distance;
#ifdef RAY_COUNT
	ray_buffer[launch_index].ray_count = prd.ray_count;
#endif
#ifdef DAYSIM_COMPATIBLE
	if (dc_buffer.size().x)
		daysimCopy(&dc_buffer[prd.dc], prd.dc);
#endif
}

RT_PROGRAM void exception()
{
#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
#ifdef TIME_VIEW
	clock_t t1 = clock();
 
	float expected_fps   = 1.0f;
	float ray_time       = ( t1 - ray_buffer[launch_index].val.x ) * time_view_scale * expected_fps;
	ray_buffer[index].val = make_float3( ray_time );
#else
	ray_buffer[launch_index].val = exceptionToFloat3(rtGetExceptionCode());
	ray_buffer[launch_index].weight = -1.0f;
#endif
}
