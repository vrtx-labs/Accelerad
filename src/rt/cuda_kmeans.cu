#include "hip/hip_runtime.h"
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
/*   File:         cuda_kmeans.cu  (CUDA version)                            */
/*   Description:  Implementation of simple k-means clustering algorithm     */
/*                 This program takes an array of N data objects, each with  */
/*                 M coordinates and performs a k-means clustering given a   */
/*                 user-provided value of the number of clusters (K). The    */
/*                 clustering results are saved in 2 arrays:                 */
/*                 1. a returned array of size [K][N] indicating the center  */
/*                    coordinates of K clusters                              */
/*                 2. membership[N] stores the cluster center ids, each      */
/*                    corresponding to the cluster a data object is assigned */
/*                                                                           */
/*   Author:  Wei-keng Liao                                                  */
/*            ECE Department, Northwestern University                        */
/*            email: wkliao@ece.northwestern.edu                             */
/*   Copyright, 2005, Wei-keng Liao                                          */
/*                                                                           */
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

// Copyright (c) 2005 Wei-keng Liao
// Copyright (c) 2011 Serban Giuroiu
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
// THE SOFTWARE.

// -----------------------------------------------------------------------------

#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"
#ifdef RANDOM_SEEDS
#include "random.h"
#endif /* RANDOM_SEEDS */

#ifdef CAP_REGISTERS_PER_THREAD
#include "accelerad.h"
/* This is the maximum number of registers used by any cuda kernel in this in this file,
found by using the flag "-Xptxas -v" to compile in nvcc. This should be updated when
changes are made to the kernels. */
#ifdef RTX
#define REGISTERS_PER_THREAD	40	/* Registers per thread under CUDA 10.0 */
#else
#define REGISTERS_PER_THREAD	26	/* Registers per thread under CUDA 7.5 */
#endif
#endif

#ifdef __cplusplus
extern "C" {
#endif

static inline int nextPowerOfTwo(int n) {
	n--;

	n = n >>  1 | n;
	n = n >>  2 | n;
	n = n >>  4 | n;
	n = n >>  8 | n;
	n = n >> 16 | n;
//  n = n >> 32 | n;    //  For 64-bit ints

	return ++n;
}

#ifdef IC_WEIGHT
/*----< ic_error >-----------------------------------------------------------*/
/* error metric from Wang et al. "An Efficient GPU-based Approach for        */
/* Interactive Global Illumination" Eq. 2                                    */
/* added by Nathaniel Jones 1/23/2014                                        */
__host__ __device__ inline static
float ic_error(int    numCoords,
			   int    numObjs,
			   int    numClusters,
			   float *objects,     // [numCoords][numObjs]
			   float *clusters,    // [numCoords][numClusters]
			   int    objectId,
			   int    clusterId,
			   float  alpha)
{
	int i;
	float ans=0.0f, ans1;

	for (i = 0; i < 3; i++) {
		ans1 = objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId];
		ans += ans1 * ans1;
	}

	ans1=1.0f;
	for ( ; i < numCoords; i++) {
		ans1 -= objects[numObjs * i + objectId] * clusters[numClusters * i + clusterId];
	}
	if (ans1 < 0.0f)
		ans1 = 0.0f;
	return alpha * sqrtf(ans) + sqrtf(2.0f*ans1);
}
#else /* IC_WEIGHT */
/*----< euclid_dist_2() >----------------------------------------------------*/
/* square of Euclid distance between two multi-dimensional points            */
__host__ __device__ inline static
float euclid_dist_2(int    numCoords,
					int    numObjs,
					int    numClusters,
					float *objects,     // [numCoords][numObjs]
					float *clusters,    // [numCoords][numClusters]
					int    objectId,
					int    clusterId)
{
	int i;
	float ans=0.0;

	for (i = 0; i < numCoords; i++) {
		ans += (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]) *
			   (objects[numObjs * i + objectId] - clusters[numClusters * i + clusterId]);
	}

	return(ans);
}
#endif /* IC_WEIGHT */

/*----< find_nearest_cluster() >---------------------------------------------*/
__global__ static
void find_nearest_cluster(int numCoords,
						  int numObjs,
						  int numClusters,
#ifdef IC_WEIGHT
						  float alpha,
#endif
						  float *objects,           //  [numCoords][numObjs]
						  float *deviceClusters,    //  [numCoords][numClusters]
						  int *membership,          //  [numObjs]
#ifdef RETURN_DISTANCE
						  float *distance,          //  [numObjs]
#endif
						  unsigned int *intermediates)
{
	extern __shared__ COUNTER sharedMemory[];

	//  The type chosen for membershipChanged must be large enough to support
	//  reductions! There are blockDim.x elements, one for each thread in the
	//  block. See numThreadsPerClusterBlock in cuda_kmeans().
	COUNTER *membershipChanged = (COUNTER *)sharedMemory;
#if BLOCK_SHARED_MEM_OPTIMIZATION
	float *clusters = (float *)(sharedMemory + blockDim.x);
#else
	float *clusters = deviceClusters;
#endif

	membershipChanged[threadIdx.x] = 0;

#if BLOCK_SHARED_MEM_OPTIMIZATION
	//  BEWARE: We can overrun our shared memory here if there are too many
	//  clusters or too many coordinates! For reference, a Tesla C1060 has 16
	//  KiB of shared memory per block, and a GeForce GTX 480 has 48 KiB of
	//  shared memory per block.
	for (int i = threadIdx.x; i < numClusters; i += blockDim.x) {
		for (int j = 0; j < numCoords; j++) {
			clusters[numClusters * j + i] = deviceClusters[numClusters * j + i];
		}
	}
	__syncthreads();
#endif

	int objectId = blockDim.x * blockIdx.x + threadIdx.x;

	if (objectId < numObjs) {
		int   index, i;
		float dist, min_dist;

		/* find the cluster id that has min distance to object */
		index    = 0;
#ifdef IC_WEIGHT
		min_dist = ic_error(numCoords, numObjs, numClusters, objects, clusters, objectId, 0, alpha);
#else
		min_dist = euclid_dist_2(numCoords, numObjs, numClusters, objects, clusters, objectId, 0);
#endif

		for (i=1; i<numClusters; i++) {
#ifdef IC_WEIGHT
			dist = ic_error(numCoords, numObjs, numClusters, objects, clusters, objectId, i, alpha);
#else
			dist = euclid_dist_2(numCoords, numObjs, numClusters, objects, clusters, objectId, i);
#endif
			/* no need square root */
			if (isnan(min_dist) || dist < min_dist) { /* find the min and its array index */
				min_dist = dist;
				index    = i;
			}
		}

		if (membership[objectId] != index) {
			membershipChanged[threadIdx.x] = 1;
		}

		/* assign the membership to object objectId */
		membership[objectId] = index;
#ifdef RETURN_DISTANCE
		distance[objectId] = min_dist;
#endif

		__syncthreads();    //  For membershipChanged[]

		//  blockDim.x *must* be a power of two!
		for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
			if (threadIdx.x < s) {
				membershipChanged[threadIdx.x] += membershipChanged[threadIdx.x + s];
			}
			__syncthreads();
		}

		if (threadIdx.x == 0) {
			intermediates[blockIdx.x] = membershipChanged[0];
		}
	}
}

__global__ static
void compute_delta(unsigned int *deviceIntermediates,
				   unsigned int numIntermediates)    //  The actual number of intermediates
{
	//  The number of elements in this array should be equal to
	//  numIntermediates2, the number of threads launched. It *must* be a power
	//  of two!
	extern __shared__ unsigned int intermediates[];

	//  Copy global intermediate values into shared memory.
	int objectId = blockDim.x * blockIdx.x + threadIdx.x;
	intermediates[threadIdx.x] = (objectId < numIntermediates) ? deviceIntermediates[objectId] : 0;

	__syncthreads();

	//  blockDim.x *must* be a power of two!
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
		if (threadIdx.x < s) {
			intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		deviceIntermediates[blockDim.x * blockIdx.x] = intermediates[0];
	}
}

/*----< cuda_kmeans() >-------------------------------------------------------*/
//
//  ----------------------------------------
//  DATA LAYOUT
//
//  objects         [numObjs][numCoords]
//  clusters        [numClusters][numCoords]
//  dimObjects      [numCoords][numObjs]
//  dimClusters     [numCoords][numClusters]
//  newClusters     [numCoords][numClusters]
//  deviceObjects   [numCoords][numObjs]
//  deviceClusters  [numCoords][numClusters]
//  ----------------------------------------
//
/* return an array of cluster centers of size [numClusters][numCoords]       */
float** CCALL cuda_kmeans(float **objects,      /* in: [numObjs][numCoords] */
						  int     numCoords,    /* no. features */
						  int     numObjs,      /* no. objects */
						  int     numClusters,  /* no. clusters */
						  int     max_iterations,	/* maximum k-means iterations */
						  float   threshold,    /* % objects change membership */
#ifdef IC_WEIGHT
						  float   weight,       /* relative weighting of position */
#endif
#ifdef RANDOM_SEEDS
						  int     randomSeeds,  /* use randomly selected cluster centers (boolean) */
#endif
						  int    *membership,   /* out: [numObjs] */
#ifdef RETURN_DISTANCE
						  float  *distance,     /* out: [numObjs] */
#endif
						  int    *loop_iterations)
{
	int      i, j, index, step, loop=0;
	int     *newClusterSize; /* [numClusters]: no. objects assigned in each
								new cluster */
	float    delta;          /* % of objects change their clusters */
	float  **dimObjects;
	float  **clusters;       /* out: [numClusters][numCoords] */
	float  **dimClusters;
	float  **newClusters;    /* [numCoords][numClusters] */

	float *deviceObjects;
	float *deviceClusters;
	int *deviceMembership;
#ifdef RETURN_DISTANCE
	float *deviceDistance;
#endif
	unsigned int *deviceIntermediates;

	//  Copy objects given in [numObjs][numCoords] layout to new
	//  [numCoords][numObjs] layout
	malloc2D(dimObjects, numCoords, numObjs, float);
	for (i = 0; i < numCoords; i++) {
		for (j = 0; j < numObjs; j++) {
			dimObjects[i][j] = objects[j][i];
		}
	}

	/* pick first numClusters elements of objects[] as initial cluster centers*/
	malloc2D(dimClusters, numCoords, numClusters, float);
	//step = numObjs / numClusters;
	//for (i = 0; i < numCoords; i++) {
	//	for (j = 0; j < numClusters; j++) {
	//		dimClusters[i][j] = dimObjects[i][j * step];
	//	}
	//}
#ifdef RANDOM_SEEDS
	if (randomSeeds)
		for (j = 0; j < numClusters; j++) {
			step = (int)((j + frandom()) * numObjs / numClusters);
			for (i = 0; i < numCoords; i++) {
				dimClusters[i][j] = dimObjects[i][step];
			}
		}
	else
#endif /* RANDOM_SEEDS */
	for (i = 0; i < numCoords; i++) {
		for (j = 0; j < numClusters; j++) {
			dimClusters[i][j] = dimObjects[i][j];
		}
	}

	/* initialize membership[] */
	//for (i=0; i<numObjs; i++) membership[i] = -1;

	/* need to initialize newClusterSize and newClusters[0] to all 0 */
	newClusterSize = (int*) calloc(numClusters, sizeof(int));
	assert(newClusterSize != NULL);

	malloc2D(newClusters, numCoords, numClusters, float);
	memset(newClusters[0], 0, numCoords * numClusters * sizeof(float));

	hipDeviceProp_t deviceProp;
	int deviceNum;
	hipGetDevice(&deviceNum);
	hipGetDeviceProperties(&deviceProp, deviceNum);

	//  To support reduction, numThreadsPerClusterBlock *must* be a power of
	//  two, and it *must* be no larger than the number of bits that will
	//  fit into an unsigned char, the type used to keep track of membership
	//  changes in the kernel.
#ifdef CAP_REGISTERS_PER_THREAD
	const unsigned int numRegistersPerClusterBlock = deviceProp.regsPerBlock;
	unsigned int numThreadsPerClusterBlock = deviceProp.maxThreadsPerBlock;
	while (numRegistersPerClusterBlock / numThreadsPerClusterBlock < REGISTERS_PER_THREAD)
		numThreadsPerClusterBlock >>= 1;
#else
	const unsigned int numThreadsPerClusterBlock = deviceProp.maxThreadsPerBlock;//128;
#endif
	const unsigned int numClusterBlocks = (numObjs - 1) / numThreadsPerClusterBlock + 1;
#if BLOCK_SHARED_MEM_OPTIMIZATION
	const size_t clusterBlockSharedDataSize = numThreadsPerClusterBlock * sizeof(COUNTER) + numClusters * numCoords * sizeof(float);

	if (clusterBlockSharedDataSize > deviceProp.sharedMemPerBlock) {
		err("Your CUDA hardware has insufficient block shared memory %llu (%llu needed). "
			"You need to recompile with BLOCK_SHARED_MEM_OPTIMIZATION=0.",
			deviceProp.sharedMemPerBlock, clusterBlockSharedDataSize);
	}
#else
	const size_t clusterBlockSharedDataSize = numThreadsPerClusterBlock * sizeof(COUNTER);
#endif

	const unsigned int numReductionBlocks = (numClusterBlocks - 1) / numThreadsPerClusterBlock + 1;
	const unsigned int numReductionThreads = nextPowerOfTwo(numClusterBlocks / numReductionBlocks); // per block
	//const unsigned int numReductionThreads = nextPowerOfTwo(numClusterBlocks);
	const unsigned int reductionBlockSharedDataSize = numReductionThreads * sizeof(unsigned int);

	checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(float)));
	checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(float)));
	checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
#ifdef RETURN_DISTANCE
	checkCuda(hipMalloc(&deviceDistance, numObjs*sizeof(float)));
#endif
	checkCuda(hipMalloc(&deviceIntermediates, numReductionBlocks*numReductionThreads*sizeof(unsigned int)));

	checkCuda(hipMemcpy(deviceObjects, dimObjects[0], numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice));
	//checkCuda(hipMemcpy(deviceMembership, membership, numObjs*sizeof(int), hipMemcpyHostToDevice));
	checkCuda(hipMemset(deviceMembership, -1, numObjs*sizeof(int)));

	const unsigned int reducedLength = (numReductionBlocks - 1) * numReductionThreads + 1;
	int* reducedSums;
	if (numReductionBlocks > 1u)
		reducedSums = (int*)malloc(reducedLength * sizeof(int));

	do {
		checkCuda(hipMemcpy(deviceClusters, dimClusters[0], numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice));

		find_nearest_cluster <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
			(numCoords, numObjs, numClusters,
#ifdef IC_WEIGHT
			weight,
#endif
			deviceObjects, deviceClusters, deviceMembership,
#ifdef RETURN_DISTANCE
			deviceDistance,
#endif
			deviceIntermediates);

		hipDeviceSynchronize(); checkLastCudaError();

		compute_delta <<< numReductionBlocks, numReductionThreads, reductionBlockSharedDataSize >>>
			(deviceIntermediates, numClusterBlocks);

		hipDeviceSynchronize(); checkLastCudaError();

		if (numReductionBlocks == 1u) {
			int d;
			checkCuda(hipMemcpy(&d, deviceIntermediates, sizeof(int), hipMemcpyDeviceToHost));
			delta = (float)d;
		} else {
			checkCuda(hipMemcpy(reducedSums, deviceIntermediates, reducedLength * sizeof(int), hipMemcpyDeviceToHost));
			unsigned int reducedSum = 0u;
			for (i=0; i<numReductionBlocks; i++)
				reducedSum += reducedSums[i * numReductionThreads];
			delta = (float)reducedSum;
			//int d;
			//long reductionSum = 0L;
			//for (i=0; i<numReductionBlocks; i++) {
			//	checkCuda(hipMemcpy(&d, deviceIntermediates + i * numReductionThreads, sizeof(int), hipMemcpyDeviceToHost));
			//	reductionSum += d;
			//}
			//delta = (float)reductionSum;
		}

		checkCuda(hipMemcpy(membership, deviceMembership, numObjs*sizeof(int), hipMemcpyDeviceToHost));

		for (i=0; i<numObjs; i++) {
			/* find the array index of nestest cluster center */
			index = membership[i];

			/* update new cluster centers : sum of objects located within */
			newClusterSize[index]++;
			for (j=0; j<numCoords; j++)
				newClusters[j][index] += objects[i][j];
		}

		//  TODO: Flip the nesting order
		//  TODO: Change layout of newClusters to [numClusters][numCoords]
		/* average the sum and replace old cluster centers with newClusters */
		for (i=0; i<numClusters; i++) {
			for (j=0; j<numCoords; j++) {
				if (newClusterSize[i] > 0)
					dimClusters[j][i] = newClusters[j][i] / newClusterSize[i];
				newClusters[j][i] = 0.0;   /* set back to 0 */
			}
			newClusterSize[i] = 0;   /* set back to 0 */
		}

		delta /= numObjs;
	} while (++loop < max_iterations && delta > threshold);

	*loop_iterations = loop;

	/* allocate a 2D space for returning variable clusters[] (coordinates
	   of cluster centers) */
	malloc2D(clusters, numClusters, numCoords, float);
	for (i = 0; i < numClusters; i++) {
		for (j = 0; j < numCoords; j++) {
			clusters[i][j] = dimClusters[j][i];
		}
	}

#ifdef RETURN_DISTANCE
	checkCuda(hipMemcpy(distance, deviceDistance, numObjs*sizeof(float), hipMemcpyDeviceToHost));
#endif

	checkCuda(hipFree(deviceObjects));
	checkCuda(hipFree(deviceClusters));
	checkCuda(hipFree(deviceMembership));
	checkCuda(hipFree(deviceDistance));
#ifdef RETURN_DISTANCE
	checkCuda(hipFree(deviceIntermediates));
#endif

	free(dimObjects[0]);
	free(dimObjects);
	free(dimClusters[0]);
	free(dimClusters);
	free(newClusters[0]);
	free(newClusters);
	free(newClusterSize);
	if (numReductionBlocks > 1u)
		free(reducedSums);

	return clusters;
}

#ifdef __cplusplus
}
#endif
