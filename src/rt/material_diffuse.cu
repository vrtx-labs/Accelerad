#include "hip/hip_runtime.h"
/*
 *  material_diffuse.cu - hit program for diffuse-only reflection on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "optix_shader_ray.h"

using namespace optix;

#define  TRANSMISSION

/* specularity flags */
#define  SP_REFL	01		/* has reflected specular component */
#define  SP_TRAN	02		/* has transmitted specular */
#define  SP_PURE	04		/* purely specular (zero roughness) */
#define  SP_FLAT	010		/* flat reflecting surface */
#define  SP_RBLT	020		/* reflection below sample threshold */
#define  SP_TBLT	040		/* transmission below threshold */

typedef struct {
	unsigned int specfl;		/* specularity flags, defined above */
	float3 mcolor;		/* color of this material */
	float3 scolor;		/* color of specular component */
	//float3 vrefl;		/* vector in direction of reflected ray */
	float3 prdir;		/* vector in transmitted direction */
	float3 normal;
	float3 hit;
	float  alpha2;		/* roughness squared */
	float  rdiff, rspec;	/* reflected specular, diffuse */
	float  trans;		/* transmissivity */
	float  tdiff, tspec;	/* transmitted specular, diffuse */
	float3 pnorm;		/* perturbed surface normal */
	float  pdot;		/* perturbed dot product */
}  NORMDAT;		/* normal material data */

/* Context variables */
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_ambient, , );

rtDeclareVariable(float3, ambval, , );	/* This is the final value used in place of an indirect light calculation */
rtDeclareVariable(int, ambvwt, , );	/* As new indirect irradiances are computed, they will modify the default ambient value in a moving average, with the specified weight assigned to the initial value given on the command and all other weights set to 1 */
rtDeclareVariable(int, ambounce, , );	/* Ambient bounces (ab) */
//rtDeclareVariable(int,          ambres, , );	/* Ambient resolution (ar) */
rtDeclareVariable(float, ambacc, , );	/* Ambient accuracy (aa). This value will approximately equal the error from indirect illuminance interpolation */
rtDeclareVariable(int, ambdiv, , );	/* Ambient divisions (ad) */
rtDeclareVariable(int, ambdiv_final, , ); /* Number of ambient divisions for final-pass fill (ag) */
//rtDeclareVariable(int, ambssamp, , );	/* Ambient super-samples (as) */
rtDeclareVariable(float, avsum, , );		/* computed ambient value sum (log) */
rtDeclareVariable(unsigned int, navsum, , );	/* number of values in avsum */


RT_METHOD float3 multambient(float3 aval, const float3& normal, const float3& pnormal, const float3& hit, const unsigned int& ambincl, PerRayData_radiance &prd);
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, PerRayData_radiance &prd, DaysimCoef dc);
#else
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, PerRayData_radiance &prd);
#endif


RT_CALLABLE_PROGRAM PerRayData_radiance closest_hit_diffuse_radiance(IntersectData const&data, PerRayData_radiance prd)
{
	NORMDAT nd;

	/* check for back side */
	nd.pnorm = faceforward(data.world_shading_normal, -data.ray_direction, data.world_geometric_normal);
	nd.normal = faceforward(data.world_geometric_normal, -data.ray_direction, data.world_geometric_normal);

	float3 result = make_float3(0.0f);
	nd.hit = data.hit;
	nd.mcolor = data.mat.color;
	nd.scolor = make_float3(0.0f);
	nd.rspec = data.mat.params.n.spec;
	nd.alpha2 = data.mat.params.n.rough * data.mat.params.n.rough;
	nd.specfl = 0u; /* specularity flags */

	/* get roughness */
	if (nd.alpha2 <= FTINY) {
		nd.specfl |= SP_PURE; // label this as a purely specular reflection
	}

	/* perturb normal */
	float3 pert = nd.normal - nd.pnorm;
	int hastexture = dot(pert, pert) > FTINY * FTINY;
	nd.pdot = -dot(data.ray_direction, nd.pnorm);
	if (nd.pdot < 0.0f) {		/* fix orientation from raynormal in raytrace.c */
		nd.pnorm += 2.0f * nd.pdot * data.ray_direction;
		nd.pdot = -nd.pdot;
	}
	if (nd.pdot < 0.001f)
		nd.pdot = 0.001f;			/* non-zero for dirnorm() */

	// if it's a face or a ring label as flat (currently we only support triangles, so everything is flat)
	nd.specfl |= SP_FLAT;

	/* modify material color */
	//nd.mcolor *= rtTex3D(rtTextureId id, texcoord.x, texcoord.y, texcoord.z).xyz;

	/* compute Fresnel approx. */
	float fest = 0.0f;
	if (nd.specfl & SP_PURE && nd.rspec >= FRESTHRESH) {
		fest = FRESNE(nd.pdot);
		nd.rspec += fest * (1.0f - nd.rspec);
	}

	/* compute transmission */
	nd.tdiff = nd.tspec = nd.trans = 0.0f; // because it's opaque

	/* diffuse reflection */
	nd.rdiff = 1.0f - nd.trans - nd.rspec;

	if (!(nd.specfl & SP_PURE && nd.rdiff <= FTINY && nd.tdiff <= FTINY)) { /* not 100% pure specular */
		/* ambient from this side */
		if (nd.rdiff > FTINY) {
			float3 aval = nd.mcolor * nd.rdiff;	/* modified by material color */
			if (nd.specfl & SP_RBLT)	/* add in specular as well? */
				aval += nd.scolor;
			result += multambient(aval, nd.normal, nd.pnorm, nd.hit, data.mat.params.n.ambincl, prd);	/* add to returned color */
		}

#ifdef TRANSMISSION
		/* ambient from other side */
		if (nd.tdiff > FTINY) {
			float3 aval = nd.mcolor;	/* modified by material color */
			if (nd.specfl & SP_TBLT)
				aval *= nd.trans;
			else
				aval *= nd.tdiff;
			result += multambient(aval, -nd.normal, -nd.pnorm, nd.hit, data.mat.params.n.ambincl, prd);	/* add to returned color */
		}
#endif /* TRANSMISSION */
	}

	prd.distance = data.t;

	// pass the color back up the tree
	prd.result = result;

	return prd;
}


// Compute the ambient component and multiply by the coefficient.
RT_METHOD float3 multambient(float3 aval, const float3& normal, const float3& pnormal, const float3& hit, const unsigned int& ambincl, PerRayData_radiance &prd)
{
	float 	d;

	/* ambient calculation */
	if (ambdiv > 0 && prd.ambient_depth < ambounce && ambincl) {
		float3 acol = aval;
	#ifdef DAYSIM_COMPATIBLE
		DaysimCoef dc = daysimNext(prd.dc);
		daysimSet(dc, 0.0f);
		d = doambient(&acol, normal, pnormal, hit, prd, dc);
		if (d > FTINY)
			daysimAdd(prd.dc, dc);
	#else
		d = doambient(&acol, normal, pnormal, hit, prd);
	#endif
		if (d > FTINY)
			return acol;
	}
					/* return global value */
	if ((ambvwt <= 0) || (navsum == 0)) {
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * ambval.x);
#endif
		return aval * ambval;
	}
	float l = bright(ambval);			/* average in computations */
	if (l > FTINY) {
		d = (logf(l)*(float)ambvwt + avsum) / (float)(ambvwt + navsum);
		d = expf(d) / l;
		aval *= ambval;	/* apply color of ambval */
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * ambval.x * d);
#endif
	}
	else {
		d = expf(avsum / (float)navsum);
#ifdef DAYSIM_COMPATIBLE
		daysimAdd(prd.dc, aval.x * d);
#endif
	}
	return aval * d;
}


/* sample indirect hemisphere, based on samp_hemi in ambcomp.c */
#ifdef DAYSIM_COMPATIBLE
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, PerRayData_radiance &prd, DaysimCoef dc)
#else
RT_METHOD int doambient(float3 *rcol, const float3& normal, const float3& pnormal, const float3& hit, PerRayData_radiance &prd)
#endif
{
	float	d;
	float wt = prd.weight;

	/* set number of divisions */
	if (ambacc <= FTINY && wt > (d = 0.8f * fmaxf(*rcol) * wt / (ambdiv_final * minweight)))
		wt = d;			/* avoid ray termination */
	float3 acol = make_float3(0.0f);
	float3 acoef = *rcol;

	/* Setup from ambsample in ambcomp.c */
	PerRayData_radiance new_prd;
	if (!rayorigin(new_prd, prd, acoef, 1, 1))
		return(0);

#ifdef DAYSIM_COMPATIBLE
	new_prd.dc = daysimNext(dc);
#endif

	/* End ambsample setup */

	/* make tangent plane axes */
	float3 ux = getperpendicular(pnormal, prd.state);
	float3 uy = cross(pnormal, ux);

	/* ambsample in ambcomp.c */
	float2 spt = make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
	SDsquare2disk(spt, spt.y, spt.x);
	float zd = sqrtf(1.0f - dot(spt, spt));
	float3 direction = normalize(spt.x*ux + spt.y*uy + zd*pnormal);
	if (dot(direction, normal) <= 0) /* Prevent light leaks */
		return(0);

	setupPayload(new_prd);
	Ray amb_ray = make_Ray(hit, direction, RADIANCE_RAY, ray_start(hit, direction, normal, RAY_START), new_prd.tmax);
	rtTrace(top_object, amb_ray, new_prd);
	resolvePayload(prd, new_prd);

	if (isnan(new_prd.result)) // TODO How does this happen?
		return(0);
	if (new_prd.distance <= FTINY)
		return(0);		/* should never happen */
	acol += new_prd.result * acoef;	/* add to our sum */
#ifdef DAYSIM_COMPATIBLE
	daysimAddScaled(dc, new_prd.dc, acoef.x);
#endif
	*rcol = acol;
	return(1);			/* all is well */
}
