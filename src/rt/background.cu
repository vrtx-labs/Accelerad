#include "hip/hip_runtime.h"
/*
 *  background.cu - miss program for ray tracing on GPUs.
 */

#include "accelerad_copyright.h"

#ifdef HIT_TYPE
#include "otypes.h"	/* For definition of OBJ_SOURCE */
#endif

#include <optix_world.h>
#include "optix_shader_ray.h"
#ifdef CONTRIB
#include "optix_shader_contrib.h"
#endif

using namespace optix;

/* Context variables */
rtBuffer<DistantLight> lights;
//rtBuffer<rtCallableProgramId<float(const float3)> > functions;
//rtDeclareVariable(rtCallableProgramId<float(float3)>, func, , );
//rtDeclareVariable(rtCallableProgramX<float(float3)>, func, , );
rtDeclareVariable(int, directvis, , );		/* Boolean switch for light source visibility (dv) */

/* OptiX variables */
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


#ifdef DAYSIM_COMPATIBLE
RT_METHOD unsigned int daysimComputePatch(const float3 dir);
#endif

RT_PROGRAM void miss()
{
	prd_radiance.result = prd_radiance.mirror = make_float3(0.0f);
	prd_radiance.distance = prd_radiance.mirror_distance = prd_radiance.tmax;
	if (prd_radiance.tmax < RAY_END) // ray length was truncated
		return;

	const float3 H = optix::normalize(ray.direction);

	// compute direct lighting
	int foundsrc = -1, glowsrc = -1;
	unsigned int num_lights = lights.size();
	for (int i = 0; i < num_lights; ++i) {
		DistantLight light = lights[i];

		// no contribution to ambient calculation
		if (prd_radiance.ambient_depth && light.casts_shadow) // badcomponent() in source.c
			continue; // TODO also no contribution from specular

		// get the angle bwetween the light direction and the view
		float3 L = optix::normalize(light.pos);
		float lDh = optix::dot( L, H );
		float solid_angle = 2.0f * M_PIf * (1.0f - lDh);

		// Check to see if ray is within solid angle of source
		if (solid_angle <= light.solid_angle) {
			// Use first hit
			if (light.casts_shadow) {
				foundsrc = i;
				break;
			}
			// If it's a glow or transparent illum, just remember it
			if (glowsrc == -1) glowsrc = i;
		}
	}

	// Do we need fallback?
	if (foundsrc == -1) {
		if (glowsrc == -1) return;
		foundsrc = glowsrc;
	}

	DistantLight light = lights[foundsrc];
	if (!directvis && light.casts_shadow) { // srcignore() in source.c
		prd_radiance.result = make_float3(0.0f);
	}
	else {
		float3 color = light.color;
		if (light.function != RT_PROGRAM_ID_NULL)
			color *= ((rtCallableProgramId<float3(const float3, const float3)>)light.function)(H, -H);
		prd_radiance.result = color;
#ifdef DAYSIM_COMPATIBLE
		if (daylightCoefficients >= 2) {
			daysimAddCoef(prd_radiance.dc, daysimComputePatch(ray.direction), color.x);
		}
#endif /* DAYSIM_COMPATIBLE */
	}
#ifdef CONTRIB
	contribution(prd_radiance.rcoef, prd_radiance.result, H, light.contrib_index, light.contrib_function);
#endif /* CONTRIB */

#ifdef HIT_TYPE
	prd_radiance.hit_type = OBJ_SOURCE;
#endif
}

RT_PROGRAM void miss_shadow()
{
	prd_shadow.result = make_float3(0.0f);

	const float3 H = optix::normalize(ray.direction);

	// compute direct lighting
	if ( prd_shadow.target >= 0 && prd_shadow.target < lights.size() ) {
		DistantLight light = lights[prd_shadow.target];
		if (light.casts_shadow) {

			// get the angle bwetween the light direction and the view
			float3 L = optix::normalize(light.pos);
			float lDh = optix::dot( L, H );
			float solid_angle = 2.0f * M_PIf * (1.0f - lDh);

			if (solid_angle <= light.solid_angle) {
				float3 color = light.color;
				if (light.function != RT_PROGRAM_ID_NULL)
					color *= ((rtCallableProgramId<float3(const float3, const float3)>)light.function)(H, -H);
				prd_shadow.result = color;
#ifdef DAYSIM_COMPATIBLE
				if (daylightCoefficients >= 2) {
					// TODO This assumes that all sources are sun positions in numerical order
					// TODO If files are merged, add 148 to prd_shadow.target
					daysimAddCoef(prd_shadow.dc, prd_shadow.target, color.x);
				}
#endif /* DAYSIM_COMPATIBLE */
#ifdef CONTRIB
				contribution(prd_shadow.rcoef, color, H, light.contrib_index, light.contrib_function);
#endif /* CONTRIB */
			}
		}
	}
}

#ifdef DAYSIM_COMPATIBLE
/*
* Computes the sky/ground patch hit by a ray in direction (dx,dy,dz)
* according to the Tregenza sky division.
*/
RT_METHOD unsigned int daysimComputePatch(const float3 dir)
{
	if (dir.z > 0.0f) { // sky
		const unsigned int number[8] = { 0, 30, 60, 84, 108, 126, 138, 144 };
		const float ring_division[8] = { 30.0f, 30.0f, 24.0f, 24.0f, 18.0f, 12.0f, 6.0f, 0.0f };
		int ringnumber = (int)(asinf(dir.z) * 15.0f * M_1_PIf);
		// origin of the number "15":
		// according to Tregenza, the celestial hemisphere is divided into 7 bands and
		// the zenith patch. The bands range from:
		//												altitude center
		// Band 1		0 to 12 Deg			30 patches	6
		// Band 2		12 to 24 Deg		30 patches	18
		// Band 3		24 to 36 Deg		24 patches	30
		// Band 4		36 to 48 Deg		24 patches	42
		// Band 5		48 to 60 Deg		18 patches	54
		// Band 6		60 to 72 Deg		12 patches	66
		// Band 7		72 to 84 Deg		 6 patches	78
		// Band 8		84 to 90 Deg		 1 patch 	90
		// since the zenith patch is only takes 6Deg instead of 12, the arc length
		// between 0 and 90 Deg (equlas o and Pi/2) is divided into 7.5 units:
		// Therefore, 7.5 units = (int) asin(z=1)/(Pi/2)
		//				1 unit = asin(z)*(2*7.5)/Pi)
		//				1 unit = asin(z)*(15)/Pi)
		// Note that (int) always rounds to the next lowest integer
		return number[ringnumber] + ring_division[ringnumber] * (atan2f(dir.y, dir.x) * 0.5f * M_1_PIf + (dir.y >= 0.0f ? 0.0f : 1.0f));
	}
	// ground
	if (dir.z >= -0.17365f)
		return 145;
	if (dir.z >= -0.5f)
		return 146;
	return 147;
}
#endif /* DAYSIM_COMPATIBLE */
