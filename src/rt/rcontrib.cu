#include "hip/hip_runtime.h"
/*
 *  rcontrib.cu - entry point for contribution coefficient calculation on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_ray.h"
#ifdef CONTRIB_DOUBLE
#include "optix_double.h"
#endif

using namespace optix;

/* Contex variables */
rtBuffer<float3, 2>              origin_buffer;
rtBuffer<float3, 2>              direction_buffer;
rtBuffer<contrib4, 3>            contrib_buffer;
#ifdef RAY_COUNT
rtBuffer<unsigned int, 2>        ray_count_buffer;
#endif
rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_irrad, , );
rtDeclareVariable(unsigned int, imm_irrad, , ) = 0u; /* Immediate irradiance (-I) */
rtDeclareVariable(unsigned int, lim_dist, , ) = 0u; /* Limit ray distance (-ld) */
rtDeclareVariable(unsigned int, contrib_segment, , ) = 0u; /* Start row for large outputs */

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );


RT_PROGRAM void ray_generator()
{
	const uint2 index = make_uint2(launch_index.x, launch_index.y + contrib_segment);
	PerRayData_radiance prd;
	init_rand(&prd.state, index.x + launch_dim.x * index.y);
	prd.result = prd.mirror = make_float3(0.0f); // Probably not necessary
	prd.distance = prd.mirror_distance = RAY_END;
	prd.weight = 1.0f;
	prd.depth = 0;
	prd.ambient_depth = 0;
	//prd.seed = rnd_seeds[launch_index];
#ifdef CONTRIB
	prd.rcoef = make_contrib3(1.0f);
#endif
#ifdef ANTIMATTER
	prd.mask = 0u;
	prd.inside = 0;
#endif
	setupPayload(prd);

	/* Zero the output */
	for (int i = 0; i < contrib_buffer.size().x; i++)
		contrib_buffer[make_uint3(i, launch_index.x, launch_index.y)] = make_contrib4(0.0f);

	float3 org = origin_buffer[index];
	float3 dir = direction_buffer[index];

	if (dot(dir, dir) > 0.0f) {
		const float tmin = ray_start(org, imm_irrad ? RAY_START : FTINY); // RAY_START is too large for rfluxmtx calls
		if (imm_irrad) {
			dir = -normalize(dir);
			prd.tmax = 2.0f * tmin;
			Ray ray = make_Ray(org - dir * tmin, dir, RADIANCE_RAY, 0.0f, prd.tmax);
			rtTrace(top_irrad, ray, prd);
		}
		else {
			prd.tmax = lim_dist ? length(dir) : RAY_END;
			Ray ray = make_Ray(org, normalize(dir), RADIANCE_RAY, tmin, prd.tmax);
			rtTrace(top_object, ray, prd);
		}
	}

	checkFinite(prd.result); // Probably not necessary

#ifdef RAY_COUNT
	ray_count_buffer[launch_index] = prd.ray_count;
#endif
}

RT_PROGRAM void exception()
{
#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
#ifdef CONTRIB_DOUBLE
	contrib_buffer[make_uint3(0, launch_index.x, launch_index.y)] = make_contrib4(rtGetExceptionCode(), 0.0f, 0.0f, -1.0f);
#else
	contrib_buffer[make_uint3(0, launch_index.x, launch_index.y)] = exceptionToFloat4(rtGetExceptionCode());
#endif
}
