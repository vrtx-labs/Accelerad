#include "hip/hip_runtime.h"
/*
 *  gencumulativesky.cu - program for gencumulativesky distribution on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>

using namespace optix;

struct Transform
{
	optix::Matrix<3, 3> m;
};

/* Program variables */
rtDeclareVariable(int, data, , ); /* texture ID */
rtDeclareVariable(Transform, transform, , ); /* transformation matrix */

// This function replicates the output .cal file from GenCumulativeSky.
RT_CALLABLE_PROGRAM float3 gencumsky(const float3 direction, const float3 ignore)
{
	const float3 dir = transform.m * direction;

	float alt = asinf(dir.z) * 180 * M_1_PIf;
	if (alt < 0.0f) return make_float3(0.0f);
	float az = atan2f(dir.x, dir.y) * 180 * M_1_PIf;
	if (az < 0.0f) az += 360.0f;
	
	int x = 0;
	if (alt < 12.0f)
		x = (int)(0.5f + az / 12.0f) % 30;
	else if (alt < 24.0f)
		x = (int)(0.5f + az / 12.0f) % 30 + 30;
	else if (alt < 36.0f)
		x = (int)(0.5f + az / 15.0f) % 24 + 60;
	else if (alt < 48.0f)
		x = (int)(0.5f + az / 15.0f) % 24 + 84;
	else if (alt < 60.0f)
		x = (int)(0.5f + az / 20.0f) % 18 + 108;
	else if (alt < 72.0f)
		x = (int)(0.5f + az / 30.0f) % 12 + 126;
	else if (alt < 84.0f)
		x = (int)(0.5f + az / 60.0f) % 6 + 138;
	else
		x = 144;

	return make_float3(rtTex1D<float>(data, x));
}
