#include "hip/hip_runtime.h"
/*
 *  utah.cu - program for Utah sky distribution on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_common.h"

struct Transform
{
	optix::Matrix<3,3> m;
};

/* Program variables */
rtDeclareVariable(unsigned int, monochrome, , ) = 0u; /* output brightness only */
rtDeclareVariable(float, turbidity, , ); /* turbidity */
rtDeclareVariable(float3, sun, , ); /* sun direction */
rtDeclareVariable(Transform, transform, , ); /* transformation matrix */


RT_METHOD float perez(const float& theta, const float& gamma, const float& a, const float& b, const float& c, const float& d, const float& e)
{
	const float cosGamma = cosf(gamma);
	//return (1.0f + a * expf(b / cosf(theta))) * (1.0f + c * expf(d * gamma) + e * cosGamma * cosGamma);
	return (1.0f + a * expf(b / (0.004f + abs(cosf(theta))))) * (1.0f + c * expf(d * gamma) + e * cosGamma * cosGamma);
}

// From Preetham, Shirley, Smits, "A Practical, Analytical Model for Daylight"
// Converted to Radiance by Mark J.Stock, mstock@umich.edu
// This function replicates the algorithm in utah.cal.
RT_CALLABLE_PROGRAM float3 skybr(const float3 direction, const float3 ignore)
{
	const float3 dir = transform.m * direction;

	const float cosgamma = 0.999999f * optix::dot(dir, sun); // cosgamma = Dx*A8 + Dy*A9 + Dz*A10; // Adjusted to keep in range [-1,1]
	const float gamma = acosf(cosgamma); // gamma = Acos(cosgamma);		{ angle from sun to this point in sky }
	const float theta = acosf(dir.z); // theta = Acos(Dz);		{ angle from zenith to this point in sky }
	const float thetas = acosf(sun.z); // thetas = Acos(A4);		{ angle from zenith to sun }
	const float turb = optix::clamp(turbidity, 1.2f, 6.0f); // turb = bound(1.2, A1, 6.);	{ clamp the bounds of turbidity }

	/* zenith brightness, chromaticity */
	float yyz = (4.0453f * turb - 4.971f) * tanf((0.4444f - turb / 120.0f) * (3.1415927f - 2.0f * thetas)) - 0.2155f * turb + 2.4192f;
	if (yyz <= 0.0f)
		yyz = 0.0f;
	const float xz = 0.25886f + 0.00394f * turb + thetas * (0.06052f - 0.03202f * turb * (1.0f - 0.065272f * turb) + thetas * (-0.21196f + 0.06377f * turb * (1.0f - 0.058805f * turb) + thetas * (0.11693f - 0.02903f * turb * (1.0f - 0.057182f * turb))));
	const float yz = 0.26688f + 0.00516f * turb + thetas * (0.0667f - 0.04153f * turb * (1.0f - 0.07633f * turb) + thetas * (-0.26756f + 0.0897f * turb * (1.0f - 0.068004f * turb) + thetas * (0.15346f - 0.04214f * turb * (1.0f - 0.065259f * turb))));

	/* distribution coefficients for luminance, chromaticity; functions of turbidity */
	const float ayy = 0.1787f * turb - 1.463f;
	const float byy = -0.3554f * turb + 0.4275f;
	const float cyy = -0.0227f * turb + 5.3251f;
	const float dyy = 0.1206f * turb - 2.5771f;
	const float eyy = -0.067f * turb + 0.3703f;

	const float ax = -0.0193f * turb - 0.2593f;
	const float bx = -0.0665f * turb + 0.0008f;
	const float cx = -0.0004f * turb + 0.2125f;
	const float dx = -0.0641f * turb - 0.8989f;
	const float ex = -0.0033f * turb + 0.0452f;

	const float ay = -0.0167f * turb - 0.2608f;
	const float by = -0.095f * turb + 0.0092f;
	const float cy = -0.0079f * turb + 0.2102f;
	const float dy = -0.0441f * turb - 1.6537f;
	const float ey = -0.0109f * turb + 0.0529f;

	/* point values for luminance, chromaticity */
	float yyp = yyz * perez(theta, gamma, ayy, byy, cyy, dyy, eyy) / perez(0.0f, thetas, ayy, byy, cyy, dyy, eyy);
	const float xp = xz * perez(theta, gamma, ax, bx, cx, dx, ex) / perez(0.0f, thetas, ax, bx, cx, dx, ex);
	const float yp = yz * perez(theta, gamma, ay, by, cy, dy, ey) / perez(0.0f, thetas, ay, by, cy, dy, ey);

	/* hack to allow stars to shine through haze at dusk and dawn */
	if (sun.z <= 0.05f)
		yyp *= expf(20.0f * (sun.z - 0.05f));

	/* output brightness */
	float3 skybr = make_float3(yyp);
	if (monochrome)
		return skybr;

	/* output radiance */

	/* first, tristimulus values(are these CIE XYZ ? ) */
	skybr.x *= xp / yp;
	if (xp + yp < 1.0f)
		skybr.z *= (1.0f - xp - yp) / yp;
	else
		skybr.z = 0.0f;

	/* convert using CIE M^-1 matrix from http://www.brucelindbloom.com/Eqn_RGB_XYZ_Matrix.html */
	const float xyz2rgb[9] = {
		2.3706743f, -0.9000405f, -0.4706338f,
		-0.513885f, 1.4253036f, 0.0885814f,
		0.0052982f, -0.0146949f, 1.0093968f };
	return (optix::Matrix<3, 3>)xyz2rgb * skybr;
}
