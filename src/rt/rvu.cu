#include "hip/hip_runtime.h"
/*
 *  rvu.cu - entry point for progressive rendering on GPUs.
 */

#include "accelerad_copyright.h"

#include <optix_world.h>
#include "optix_shader_ray.h"
#ifdef CONTRIB_DOUBLE
#include "optix_double.h"
#endif

#define GAMMA  2.2f
#define LUMINOUS_EFFICACY 179.0f
#define VT_ODS		'o'		/* omni-directional stereo */

using namespace optix;

#define angle(a, b)	acosf(clamp(dot(a, b), -1.0f, 1.0f))

/* Contex variables */
rtDeclareVariable(unsigned int, frame, , ); /* Current frame number, starting from zero */
rtDeclareVariable(unsigned int, camera, , ); /* Camera type (-vt) */
rtDeclareVariable(float3, eye, , ); /* Eye position (-vp) */
rtDeclareVariable(float3, U, , ); /* view.hvec */
rtDeclareVariable(float3, V, , ); /* view.vvec */
rtDeclareVariable(float3, W, , ); /* view.vdir */
rtDeclareVariable(float2, fov, , ); /* Field of view (-vh, -vv) */
rtDeclareVariable(float2, shift, , ); /* Camera shift (-vs, -vl) */
rtDeclareVariable(float2, clip, , ); /* Fore and aft clipping planes (-vo, -va) */
rtDeclareVariable(float, vdist, , ); /* Focal length */
rtDeclareVariable(float, dstrpix, , ); /* Pixel sample jitter (-pj) */
rtDeclareVariable(unsigned int, do_irrad, , ); /* Calculate irradiance (-i) */
rtDeclareVariable(unsigned int, do_lum, , ); /* Calculate luminance or illuminance */
#ifdef VT_ODS
rtDeclareVariable(float, ipd, , ) = 0.07f; /* inter-pupillary distance (this is between 0.055m and 0.07m on most humans) */
rtDeclareVariable(float3, gaze, , ); /* gaze direction (may be different from W) */
#endif

rtDeclareVariable(int2, task_position, , ); /* Position of task area (-T) */
rtDeclareVariable(float, task_angle, , ) = 0.0f; /* Opening angle of task area in radians (-T) */
rtDeclareVariable(int2, high_position, , ); /* Position of contrast high luminance area (-C) */
rtDeclareVariable(float, high_angle, , ) = 0.0f; /* Opening angle of contrast high luminance area (-C) */
rtDeclareVariable(int2, low_position, , ); /* Position of contrast high luminance area (-C) */
rtDeclareVariable(float, low_angle, , ) = 0.0f; /* Opening angle of contrast high luminance area (-C) */

rtDeclareVariable(float, exposure, , ) = 1.0f; /* Current exposure (-pe) */
rtDeclareVariable(unsigned int, greyscale, , ) = 0u; /* Convert to monocrhome (-b) */
rtDeclareVariable(int, tonemap, , ) = RT_TEXTURE_ID_NULL; /* texture ID */
rtDeclareVariable(float, fc_scale, , ) = 1000.0f; /* Maximum of scale for falsecolor images, zero for regular tonemapping (-s) */
rtDeclareVariable(int, fc_log, , ) = 0; /* Number of decades for log scale, zero for standard scale (-log) */
rtDeclareVariable(int, fc_base, , ) = 10; /* Base for log scale (-base) */
rtDeclareVariable(float, fc_mask, , ) = 0.0f; /* Minimum value to display in falsecolor images (-m) */
rtDeclareVariable(unsigned int, flags, , ) = 0; /* Flags for areas to highlight in image */

rtBuffer<unsigned int, 2>        color_buffer; /* Output RGBA colors */
rtBuffer<Metrics, 2>             metrics_buffer; /* Output metrics */
rtBuffer<float3, 2>              direct_buffer; /* GPU storage for direct component */
rtBuffer<float3, 2>              diffuse_buffer; /* GPU storage for diffuse component */
#ifdef RAY_COUNT
rtBuffer<unsigned int, 2>        ray_count_buffer;
#endif
//rtBuffer<RayParams, 2>           last_view_buffer;
//rtBuffer<unsigned int, 2>        rnd_seeds;
rtDeclareVariable(rtObject, top_object, , );

/* OptiX variables */
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim, rtLaunchDim, );


RT_METHOD float3 getViewDirection(float2 d);
RT_METHOD int splane_normal(const float3 &e1, const float3 &e2, float3 &n);
RT_METHOD float getSolidAngle();
RT_METHOD float getPositionIndex(const float3 &dir, const float3 &forward);
RT_METHOD int inTask(const int2 &position, const float &angle, const float3 &ray_direction);
RT_METHOD void tint(unsigned int &color, const unsigned int component);


// Pick the ray direction based on camera type as in image.c.
RT_PROGRAM void ray_generator()
{
	PerRayData_radiance prd;
	init_rand(&prd.state, launch_index.x + launch_dim.x * (launch_index.y + launch_dim.y * frame));

	float2 d = make_float2(hiprand_uniform(prd.state), hiprand_uniform(prd.state));
	d = 0.5f + dstrpix * (0.5f - d); // this is pixjitter() from rpict.c
	d = shift + (make_float2(launch_index) + d) / make_float2(launch_dim) - 0.5f;
	float3 ray_origin = eye;
	if (camera == VT_PAR) /* parallel view */
		ray_origin += d.x*U + d.y*V;
	float3 ray_direction = getViewDirection(d);
#ifdef VT_ODS
	if (camera == VT_ODS) {
		float dy = ipd * (d.y < 0 ? 0.5f : -0.5f);
		float az = d.x * fov.x * (M_PIf / 180.0f);
		ray_origin.x += cosf(az) * dy;
		ray_origin.y += sinf(az) * dy;
	}
#endif

	if (dot(ray_direction, ray_direction) > 0) {
		ray_origin += clip.x * ray_direction;
		ray_direction = normalize(ray_direction);

		// Zero or negative aft clipping distance indicates infinity
		prd.tmax = clip.y - clip.x;
		if (prd.tmax <= FTINY) {
			prd.tmax = RAY_END;
			if (frame)
				prd.tmax *= 0.9999f; // Do not let diffuse rays sample the sky
		}

		Ray ray = make_Ray(ray_origin, ray_direction, RADIANCE_RAY, 0.0f, prd.tmax);

		prd.result = make_float3(0.0f);
		prd.weight = 1.0f;
		prd.depth = 0;
		prd.ambient_depth = 0;
		//prd.seed = rnd_seeds[launch_index];
#ifdef CONTRIB
		prd.rcoef = make_contrib3(1.0f); //Probably not necessary
#endif
#ifdef ANTIMATTER
		prd.mask = 0u;
		prd.inside = 0;
#endif
		setupPayload(prd);

		rtTrace(top_object, ray, prd);

		checkFinite(prd.result);
	}
	else
		prd.result = make_float3(0.0f);

	float3 accum;
	if (frame)
		accum = direct_buffer[launch_index] + (diffuse_buffer[launch_index] = ((frame - 1.0f) / frame) * diffuse_buffer[launch_index] + (1.0f / frame) * prd.result);
	else
		accum = direct_buffer[launch_index] = prd.result;

	/* Tone map */
	const float efficacy = do_lum ? LUMINOUS_EFFICACY : 1.0f;
	const float luminance = bright(accum) * efficacy;
	if (tonemap == RT_TEXTURE_ID_NULL) { // Natural tone mapping
		//accum *= exposure / fc_scale;
		if (greyscale)
			accum = make_float3(luminance * exposure / fc_scale);
		else
			accum *= efficacy * exposure / fc_scale;
		if (fc_log > 0)
			accum = make_float3(logf(accum.x), logf(accum.y), logf(accum.z)) / (logf(fc_base) * fc_log) + 1.0f;
	}
	else { // False color tone mapping
		if (luminance < fc_mask)
			accum = make_float3(0.0f);
		else if (fc_log > 0)
			accum = make_float3(rtTex1D<float4>(tonemap, logf(luminance / fc_scale) / (logf(fc_base) * fc_log) + 1.0f));
		else
			accum = make_float3(rtTex1D<float4>(tonemap, luminance / fc_scale));
	}
	accum = clamp(accum * 256.0f, 0.0f, 255.0f);

	/* Save pixel color */
	color_buffer[launch_index] = 0xff000000 |
		((int)(256.0f * powf((accum.x + 0.5f) / 256.0f, 1.0f / GAMMA)) & 0xff) << 16 |
		((int)(256.0f * powf((accum.y + 0.5f) / 256.0f, 1.0f / GAMMA)) & 0xff) << 8 |
		((int)(256.0f * powf((accum.z + 0.5f) / 256.0f, 1.0f / GAMMA)) & 0xff);

	/* Calculate metrics */
	Metrics metrics;
	metrics.omega = getSolidAngle(); //TODO what if negative or bad angle?
	metrics.ev = metrics.dgp = 0.0f;
	if (do_irrad) {
		metrics.avlum = luminance; /* In this case it is illuminance. */
	}
	else {
		metrics.avlum = luminance * metrics.omega;
#ifdef VT_ODS
		float3 gaze_dir = dot(gaze, gaze) < FTINY ? W : gaze;
#else
		float3 gaze_dir = W;
#endif
		const float WdotD = dot(gaze_dir, ray_direction);
		if (WdotD > 0.0f) {
			metrics.ev = luminance * metrics.omega * WdotD;
			if (do_lum) {
				float guth = getPositionIndex(ray_direction, gaze_dir);
				metrics.dgp = (metric)luminance * luminance * metrics.omega / (guth * guth);
			}
		}
	}

	/* Calculate contributions to task areas */
	metrics.flags = 0;
	if (task_angle > 0.0f)
		metrics.flags |= inTask(task_position, task_angle, ray_direction) & 0x1;
	if (high_angle > 0.0f)
		metrics.flags |= (inTask(high_position, high_angle, ray_direction) & 0x1) << 1;
	if (low_angle > 0.0f)
		metrics.flags |= (inTask(low_position, low_angle, ray_direction) & 0x1) << 2;

	if (flags & metrics.flags & 0x1) tint(color_buffer[launch_index], 2);
	if (flags & metrics.flags & 0x2) tint(color_buffer[launch_index], 1);
	if (flags & metrics.flags & 0x4) tint(color_buffer[launch_index], 0);

	metrics_buffer[launch_index] = metrics;

#ifdef RAY_COUNT
	if (frame)
		ray_count_buffer[launch_index] += prd.ray_count;
	else
		ray_count_buffer[launch_index] = prd.ray_count;
#endif
}

/* From viewray() in image.c */
RT_METHOD float3 getViewDirection(float2 d)
{
	float z = 1.0f;

	if (camera == VT_PAR) { /* parallel view */
		d = make_float2(0.0f);
	}
	else if (camera == VT_HEM) { /* hemispherical fisheye */
		z = 1.0f - d.x*d.x * dot(U, U) - d.y*d.y * dot(V, V);
		if (z < 0.0f)
			return make_float3(0.0f);
		z = sqrtf(z);
	}
	else if (camera == VT_CYL) { /* cylindrical panorama */
		float dd = d.x * fov.x * (M_PIf / 180.0f);
		z = cosf(dd);
		d.x = sinf(dd);
	}
	else if (camera == VT_ANG) { /* angular fisheye */
		d *= fov / 180.0f;
		float dd = length(d);
		if (dd > 1.0f)
			return make_float3(0.0f);
		z = cosf(M_PIf * dd);
		d *= dd < FTINY ? M_PIf : sqrtf(1.0f - z*z) / dd;
	}
	else if (camera == VT_PLS) { /* planispheric fisheye */
		d *= make_float2(length(U), length(V));
		float dd = dot(d, d);
		z = (1.0f - dd) / (1.0f + dd);
		d *= 1.0f + z;
	}
#ifdef VT_ODS
	else if (camera == VT_ODS) { /* omni-directional stereo */
		d.y *= 2.0f;
		d.y += d.y < 0 ? 0.5f : -0.5f;
		d *= fov * (M_PIf / 180.0f); // d.x = azimuth, d.y = altitude
		z = cosf(d.x) * cosf(d.y);
		d.x = sinf(d.x) * cosf(d.y);
		d.y = sinf(d.y);
	}
#endif

	return d.x*U + d.y*V + z*W;
}

/* From splane_normal in pictool.c */
RT_METHOD int splane_normal(const float3 &e1, const float3 &e2, float3 &n)
{
	n = cross(e1, e2 - e1);
	if (dot(n, n) == 0.0f)
		return 0;
	n = normalize(n);
	return 1;
}

/* From pict_get_sangle in pictool.c */
RT_METHOD float getSolidAngle()
{
	const float2 min = shift + make_float2(launch_index) / make_float2(launch_dim) - 0.5f;
	const float2 max = shift + (make_float2(launch_index) + 1.0f) / make_float2(launch_dim) - 0.5f;
	const float3 minmin = getViewDirection(min);
	const float3 minmax = getViewDirection(make_float2(min.x, max.y));
	const float3 maxmin = getViewDirection(make_float2(max.x, min.y));
	const float3 maxmax = getViewDirection(max);

	float3 n[4] = { make_float3(0.0f), make_float3(0.0f), make_float3(0.0f), make_float3(0.0f) };

	int i = splane_normal(minmin, minmax, n[0]);
	i &= splane_normal(minmax, maxmax, n[1]);
	i &= splane_normal(maxmax, maxmin, n[2]);
	i &= splane_normal(maxmin, minmin, n[3]);

	if (!i)
		return 0.0f;
	float ang = 0.0f;
	for (i = 0; i < 4; i++) {
		ang += M_PIf - fabsf(angle(n[i], n[(i + 1) % 4]));
	}
	ang = ang - 2.0f * M_PIf;
	if ((ang > (2.0f * M_PIf)) || ang < 0) {
		//fprintf(stderr, "Normal error in pict_get_sangle %f %d %d\n", ang, x, y);
		return 0.0f;
	}
	return ang;
}

/* From get_posindex in evalglare.c */
RT_METHOD float getPositionIndex(const float3 &dir, const float3 &forward)
{
	float3 up = normalize(V); // TODO Not necessarily
	float3 hv = cross(forward, up);
	float phi = angle(cross(forward, hv), dir) - M_PI_2f;
	float teta = M_PI_2f - angle(hv, dir);
	float sigma = angle(forward, dir);
	hv = normalize(normalize(dir) / cosf(sigma) - forward);
	float tau = angle(up, hv);
	tau *= 180.0f / M_PIf;
	sigma *= 180.0f / M_PIf;

	if (phi == 0.0f)
		phi = FTINY;
	if (sigma <= 0)
		sigma = -sigma;
	if (teta == 0.0f)
		teta = FTINY;

	float posindex = expf((35.2f - 0.31889f * tau - 1.22f * expf(-2.0f * tau / 9.0f)) / 1000.0f * sigma + (21.0f + 0.26667f * tau - 0.002963f * tau * tau) / 100000.0f * sigma * sigma);

	/* below line of sight, using Iwata model */
	if (phi < 0.0f) {
		float fact = 0.8f;
		float d = 1.0f / tanf(phi);
		float s = tanf(teta) / tanf(phi);
		float r = sqrtf((s * s + 1.0f) / (d * d));
		if (r > 0.6f)
			fact = 1.2f;
		if (r > 3.0f)
			r = 3.0f;

		posindex = 1.0f + fact * r;
	}
	if (posindex > 16.0f)
		posindex = 16.0f;

	return posindex;
}

/* From get_task_lum() in evalglare.c */
RT_METHOD int inTask(const int2 &position, const float &angle, const float3 &ray_direction)
{
	float2 d = shift + make_float2(position) / make_float2(launch_dim) - 0.5f;
	float3 task_dir = getViewDirection(d);
	float r_actual = angle(task_dir, ray_direction);
	return r_actual <= angle;
}

/* Tint the color with emphasis on the component */
RT_METHOD void tint(unsigned int &color, const unsigned int component)
{
	unsigned int c = color;
	c += (0xff - (c & 0xff)) / (component == 2 ? 2 : 8);
	c += ((0xff - ((c >> 8) & 0xff)) / (component == 1 ? 2 : 8)) << 8;
	c += ((0xff - ((c >> 16) & 0xff)) / (component ? 8 : 2)) << 16;
	color = c;
}

RT_PROGRAM void exception()
{
#ifdef PRINT_OPTIX
	rtPrintExceptionDetails();
#endif
	color_buffer[launch_index] = 0xffffffff;
	if (!frame)
		direct_buffer[launch_index] = make_float3(0.0f);
	Metrics metrics;
	metrics.omega = -1.0f;
	metrics.ev = rtGetExceptionCode();
	metrics.avlum = 0.0f;
	metrics.dgp = 0.0f;
	metrics_buffer[launch_index] = metrics;
}
